#include "hip/hip_runtime.h"
//# t_cuda_complex.cu
//# Copyright (C) 2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include <lofar_config.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <complex>

#include <GPUProc/complex.h>

hipError_t addWithCuda( std::complex<float> * output_complex, const std::complex<float> * input_complex, size_t size);

__global__ void addKernel( void *in_ptr, const void *out_ptr)
{
    int i = threadIdx.x;
    // Cast to complex

    LOFAR::Cobalt::gpu::complex<float>*in = (LOFAR::Cobalt::gpu::complex<float>*) in_ptr;
    LOFAR::Cobalt::gpu::complex<float>*out = (LOFAR::Cobalt::gpu::complex<float>*) out_ptr;

    //do some computations, We are not testing the correctness of the implementation here.
    out[i] = in[i] + in[i];
    out[i] -= in[i];
    out[i] = out[i];
    out[i] *= 10.0;
}

using namespace std;
int main()
{
    const int arraySize = 5;
    // insert some values
    const complex<float> complex_in[5] = { complex<float>(1.0,1.0),
                                           complex<float>(1,-1),
                                           complex<float>(-1,1),
                                           complex<float>(-1,-1),
                                           complex<float>(4,-4)};
    complex<float> complex_out[5] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(complex_out, complex_in,arraySize);
    if (cudaStatus == hipErrorNoDevice) {
        return 3;
    }
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    const complex<float> complex_target[5] = {complex<float>(10,10),
        complex<float>(10, -10),
        complex<float>(-10,10),
        complex<float>(-10,-10),
        complex<float>(40,-40)};
      

    // validate that the output of the kernel is correct!
    if (complex_out[0] == complex_target[0] &&
        complex_out[1] == complex_target[1] &&
        complex_out[2] == complex_target[2] &&
        complex_out[3] == complex_target[3] &&
        complex_out[4] == complex_target[4]
      )
      {
        return 0;
      }
    else //print the output data and return -1
    {
      cout << "The complex values returned from the device were incorrect:" << endl;
      cout << "complex numbers, expected - received: {";
      for (int idx =0; idx < 5 ;++idx)
      {
        cout << complex_target[idx] << " - " << complex_out[idx] ;
        if (complex_target[idx]  != complex_out[idx])
          cout << "<<<";
        cout << endl;
      }
      cout << " }" << endl;
      return -1;
    }
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(std::complex<float>* output_complex,
                        const std::complex<float>* input_complex,
                        size_t size)
{
    std::complex<float> *dev_in = 0;
    std::complex<float> *dev_out = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }

    // allocate the complex buffers
    cudaStatus = hipMalloc((void**)&dev_in, size * sizeof(std::complex<float>));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_out, size * sizeof(std::complex<float>));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_in, input_complex, size * sizeof(std::complex<float>), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>((void *) dev_in, (const void *) dev_out);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(output_complex, dev_out, size * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(dev_in);
    hipFree(dev_out);
    
    return cudaStatus;
}

