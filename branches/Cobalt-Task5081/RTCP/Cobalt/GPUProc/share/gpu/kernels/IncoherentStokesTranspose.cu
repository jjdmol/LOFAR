#include "hip/hip_runtime.h"
//# IncoherentStokesTranspose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include <stdio.h>

//#define NAIVE
#define SHARED_MEM

#if !(TILE_SIZE == 16)
#error Precondition violated: TILE_SIZE == 16
#endif

#if !(NR_CHANNELS >= 1)
#error Precondition violated: NR_CHANNELS >= 1
#endif

#if !(NR_POLARIZATIONS == 2)
#error Precondition violated: NR_POLARIZATIONS == 2
#endif

#if !(NR_SAMPLES_PER_CHANNEL > 0 && NR_SAMPLES_PER_CHANNEL % 16 == 0)
#error Precondition violated: NR_SAMPLES_PER_CHANNEL > 0 && NR_SAMPLES_PER_CHANNEL % 16 == 0
#endif

#if !(NR_STATIONS >= 1)
#error Precondition violated: NR_STATIONS >= 1
#endif

// 3-D input data array of band-pass corrected data. Note that, actually, the
// data is 4-D (<tt>[station][channel][time][pol]</tt>), but the 4th dimension
// has been squashed into a single float4 (i.e., two complex polarizations).
typedef float4 (*InputDataType)[NR_STATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];

// 4-D output data array of band-pass corrected data that can be fed into an
// inverse FFT (<tt>[station][pol][time][channel]</tt>).
typedef float2 (*OutputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS];


// Performs data transposition from the output of the beamformer kernel to a
// data order suitable for an inverse FFT. Parallelisation is performed over the
// TABs and number of samples (time).
//
// We have 4 dimensions, but CUDA thread blocks can be up to three. Mangle the
// TAB and sample dimension in to dim 0 (x).
//
// The kernel needs to determine for each thread whether to read and separately
// whether to write back a sample, because the number of TABs may not divide by
// the 16x16 thread arrangement (even though we have a 1D thread block).
//
// \param[out] OutputDataType 4D output array of samples. For each TAB and pol,
// a spectrum per time step of complex floats.
// \param[in] ComplexVoltagesType 3D input array of samples (last dim (pol) is
// implicit). For each channel, the TABs per time step of two complex floats.
//
// Pre-processor input symbols (some are tied to the execution configuration)
// Symbol                  | Valid Values            | Description
// ----------------------- | ----------------------- | -----------
// NR_CHANNELS             | >= 1                    | number of frequency channels per subband
// NR_POLARIZATIONS        | 2                       | number of polarizations
// NR_SAMPLES_PER_CHANNEL  | multiple of 16 and > 0  | number of input samples per channel
// NR_STATIONS             | >= 1                    | number of Tied Array Beams to create
//
// Execution configuration:
//
// - LocalWorkSize = 1 dimensional; (256, 1, 1) is in use. 
//   Multiples of (32, 1, 1) may work too.
// - GlobalWorkSize = 3 dimensional:
//   + inner dim (x): always 1 block
//   + middle dim (y): 16 TABs can be processed in a block.
//     Number of blocks required, rounded-up. eg for 17 tabs we need 2 blocks
//   + outer dim (z): 16 samples per channel can be processed in a block.
//     Number of blocks required (fits exactly). 32 channels is 2 blocks
extern "C"
__global__ void transpose(OutputDataType output,
                          const InputDataType input)
{
  unsigned time, channel;

#if defined(NAIVE)
  // Naive approach: directly reading from and writing to global memory.
  for (int station = 0; station < NR_STATIONS; station++) {
    time = blockIdx.x * blockDim.x + threadIdx.x;
    channel = blockIdx.y * blockDim.y + threadIdx.y;
    if (time < NR_SAMPLES_PER_CHANNEL && channel < NR_CHANNELS) {
      float4 sample = (*input)[station][channel][time];
      (*output)[station][0][time][channel] = make_float2(sample.x, sample.y);
      (*output)[station][1][time][channel] = make_float2(sample.z, sample.w);
    }
  }
#elif defined(SHARED_MEM)
  // Use shared memory to do a block transpose. Both reads and writes to global
  // memory can then be made coalesced.
  __shared__ float4 tmp[TILE_SIZE][TILE_SIZE + 1];

  for (int station = 0; station < NR_STATIONS; station++) {
    time = blockIdx.x * blockDim.x + threadIdx.x;
    channel = blockIdx.y * blockDim.y + threadIdx.y;
    // Inside our data cube?
    if (channel < NR_CHANNELS && time < NR_SAMPLES_PER_CHANNEL) {
      // Read data
      tmp[threadIdx.y][threadIdx.x] =  (*input)[station][channel][time];
    }
    __syncthreads();

    time = blockIdx.x * blockDim.x + threadIdx.y;
    channel = blockIdx.y * blockDim.y + threadIdx.x;
    // Inside our data cube?
    if (channel < NR_CHANNELS && time < NR_SAMPLES_PER_CHANNEL) {
      // Write data
      float4 sample = tmp[threadIdx.x][threadIdx.y];
      (*output)[station][0][time][channel] = make_float2(sample.x, sample.y);
      (*output)[station][1][time][channel] = make_float2(sample.z, sample.w);
    }
    __syncthreads();

  }
#endif

}
