#include "hip/hip_runtime.h"
//# CoherentStokes.cu: Calculate the Stokes parameters
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#if !(TIME_INTEGRATION_FACTOR >= 1)
#error Precondition violated: TIME_INTEGRATION_FACTOR >= 1
#endif

#if !(NR_CHANNELS >= 1)
#error Precondition violated: NR_CHANNELS >= 1
#endif

#if !(NR_COHERENT_STOKES == 1 || NR_COHERENT_STOKES == 4)
#error Precondition violated: NR_COHERENT_STOKES == 1 || NR_COHERENT_STOKES == 4
#endif

#if !(COMPLEX_VOLTAGES == 0 || NR_COHERENT_STOKES == 4)
#error Precondition violated: COMPLEX_VOLTAGES == 0 || NR_COHERENT_STOKES == 4
#endif

#if !(NR_POLARIZATIONS == 2)
#error Precondition violated: NR_POLARIZATIONS == 2
#endif

#if !(NR_SAMPLES_PER_CHANNEL > 0 && NR_SAMPLES_PER_CHANNEL % TIME_INTEGRATION_FACTOR == 0)
#error Precondition violated: NR_SAMPLES_PER_CHANNEL > 0 && NR_SAMPLES_PER_CHANNEL % TIME_INTEGRATION_FACTOR == 0
#endif

#if !(NR_TABS >= 1)
#error Precondition violated: NR_TABS >= 1
#endif

//4D input array of complex samples. For each tab and polarization there are
//time lines with data for each channel
typedef float2 (*InputDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS]; 

//4D output array of stokes values. Each sample contains 1 or 4 stokes
//paramters. For each tab, there are NR_COHERENT_STOKES timeseries of channels
typedef float (*OutputDataType)[NR_TABS][NR_COHERENT_STOKES][NR_SAMPLES_PER_CHANNEL / TIME_INTEGRATION_FACTOR][NR_CHANNELS];

/*!
 * Computes the Stokes I or IQUV, or outputs the 4 complex voltages (XrXiYrYi).
 * http://www.astron.nl/~romein/papers/EuroPar-11/EuroPar-11.pdf 
 * In case of Stokes:
 * \code
 * I = X * conj(X) + Y * conj(Y)
 * Q = X * conj(X) - Y * conj(Y)
 * U = 2 * real(X * con(Y))
 * V = 2 * imag(X * con(Y))
 * \endcode
 * This reduces to (validated on paper by Wouter and John):
 * \code
 * Px = real(X) * real(X) + imag(X) * imag(X)
 * Py = real(Y) * real(Y) + imag(Y) * imag(Y)
 * I = Px + Py
 * Q = Px - Py
 * U = 2 * (real(X) * real(Y) + imag(X) * imag(Y))
 * V = 2 * (imag(X) * real(Y) - real(X) * imag(Y))
 * \endcode
 * 
 * The kernel's first parallel dimension is on the channels; the second
 * dimension is in time; the third on the tabs.  The thread block size based on
 * these factors could be larger then the hardmare max.  Therefore<tt>
 * NR_CHANNELS * timeParallelFactor * NR_TABS</tt> should not exceed the
 * hardware maximum of threads per block (1024 on an NVIDIA K10).
 *
 * \param[out] output
 *             4D output array of stokes values. Each sample contains 1 or 4
 *             stokes paramters. For each tab, there are \c NR_COHERENT_STOKES
 *             time series of channels. The dimensions are: \c NR_TABS, \c
 *             NR_COHERENT_STOKES,
 *             <tt>(NR_SAMPLES_PER_CHANNEL / TIME_INTEGRATION_FACTOR)</tt>, \c
 *             NR_CHANNELS.
 * \param[in]  input
 *             4D input array of complex samples. For each tab and polarization
 *             there are time lines with data for each channel. The dimensions
 *             are: \c NR_TABS, \c NR_POLARIZATIONS, \c NR_SAMPLES_PER_CHANNEL,
 *             \c NR_CHANNELS
 * \param[in]  timeIntegrationFactor
 *             immediate value that indicates in how many sub-ranges the sample
 *             range (time) is split for independent processing. Must be >= 1 and:
 *             NR_SAMPLES_PER_CHANNEL / timeParallelFactor must be a multiple of
 *             timeIntegrationFactor.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values  | Description
 * ----------------------- | ------------- | -----------
 * TIME_INTEGRATION_FACTOR | >= 1          | amount of samples to integrate to a single output sample
 * NR_CHANNELS             | >= 1          | number of frequency channels per subband
 * NR_COHERENT_STOKES      | 1 or 4        | number of stokes paramters to create
 * COMPLEX_VOLTAGES        | 1 or 0        | whether we compute complex voltages or coherent stokes
 * NR_POLARIZATIONS        | 2             | number of polarizations
 * NR_SAMPLES_PER_CHANNEL  | multiple of TIME_INTEGRATION_FACTOR | number of input samples per channel
 * NR_TABS                 | >= 1          | number of tabs to create
 * 
 * Execution configuration:
 * - LocalWorkSize = 3 dimensional; (\c NR_CHANNELS, \c timeParallelFactor, \c
 *                   NR_TABS).
 * - GlobalWorkSize = 3 dimensional; depends on the size of \c NR_TABS, \c
 *                   timeParallelFactor, \c NR_CHANNELS.
 */
extern "C" __global__ void coherentStokes(OutputDataType output,
                                          const InputDataType input,
                                          unsigned timeParallelFactor)
{
  unsigned channel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned time_idx    = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned tab_idx     = blockIdx.z * blockDim.z + threadIdx.z;    

  // We support all sizes of TABs, channels, and integrations:
  // skip current thread if not needed.
  if ( tab_idx >= NR_TABS)
    return;
  if ( channel_idx >= NR_CHANNELS)
    return;
  // unneeded time_idx threads already skip the for loop below.

  //# Process samples by reading TIME_INTEGRATION_FACTOR samples and writing one
  //# set of Stokes. For parallelism over time, split the sample range in
  //# timeParallelFactor sub-ranges and process these independently.
  //# For complex voltages, we don't compute anything; it's only a transpose.
  //#
  //# TODO: This kernel must be rewritten as if it is a transpose to get efficient global mem read and write accesses.
  //#       This reqs shmem. Note that combining shmem barriers with the conditional returns above is problematic.
  //# TODO: For very large TIME_INTEGRATION_FACTOR (e.g. 1024), we may need parallel reduction to have enough parallelization. TBD.
  unsigned read_idx  = time_idx * (NR_SAMPLES_PER_CHANNEL / timeParallelFactor);
  unsigned write_idx = read_idx / TIME_INTEGRATION_FACTOR;
  for ( ; read_idx < (time_idx + 1) * (NR_SAMPLES_PER_CHANNEL / timeParallelFactor) &&
          read_idx < NR_SAMPLES_PER_CHANNEL; write_idx++)
  {
    //# Integrate all values in the current stride
#   if COMPLEX_VOLTAGES == 1
      float4 stokes = { 0.0f, 0.0f, 0.0f, 0.0f };
#   else
      float stokesI = 0.0f;
#     if NR_COHERENT_STOKES == 4
        float stokesQ = 0.0f;
        float halfStokesU = 0.0f;
        float halfStokesV = 0.0f;
#     endif
#   endif
    
    //# Do the integration
    for (unsigned stride_read_idx_end = read_idx + TIME_INTEGRATION_FACTOR;
         read_idx < stride_read_idx_end; read_idx++)
    {
      float2 X = (*input)[tab_idx][0][read_idx][channel_idx];
      float2 Y = (*input)[tab_idx][1][read_idx][channel_idx];

#     if COMPLEX_VOLTAGES == 1
        stokes.x += X.x;
        stokes.y += X.y;
        stokes.z += Y.x;
        stokes.w += Y.y;
#     else
        float powerX = X.x * X.x + X.y * X.y;
        float powerY = Y.x * Y.x + Y.y * Y.y;
        stokesI += powerX + powerY;
#       if NR_COHERENT_STOKES == 4
          stokesQ += powerX - powerY;
          halfStokesU += X.x * Y.x + X.y * Y.y;
          halfStokesV += X.y * Y.x - X.x * Y.y;
#       endif
#     endif
    }

#   if COMPLEX_VOLTAGES == 1
      (*output)[tab_idx][0][write_idx][channel_idx] = stokes.x;
      (*output)[tab_idx][1][write_idx][channel_idx] = stokes.y;
      (*output)[tab_idx][2][write_idx][channel_idx] = stokes.z;
      (*output)[tab_idx][3][write_idx][channel_idx] = stokes.w;
#   else
      (*output)[tab_idx][0][write_idx][channel_idx] = stokesI;
#     if NR_COHERENT_STOKES == 4
        (*output)[tab_idx][1][write_idx][channel_idx] = stokesQ;
        (*output)[tab_idx][2][write_idx][channel_idx] = 2.0f * halfStokesU;
        (*output)[tab_idx][3][write_idx][channel_idx] = 2.0f * halfStokesV;
#     endif  
#   endif
  }
}
