#include "hip/hip_runtime.h"
//# DelayAndBandPass.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/** @file
 * This file contains an Cuda implementation of the GPU kernel for the delay
 * and bandpass correction.
 *
 * Usually, this kernel will be run after the polyphase filter kernel FIR.cl. In
 * that case, the input data for this kernel is already in floating point format
 * (@c NR_CHANNELS > 1). However, if this kernel is the first in row, then the
 * input data is still in integer format (@c NR_CHANNELS == 1), and this kernel
 * needs to do the integer-to-float conversion.
 *
 * @attention The following pre-processor variables must be supplied when
 * compiling this program. Please take the pre-conditions for these variables
 * into account:
 * - @c NR_CHANNELS: 1 or a multiple of 16
 * - if @c NR_CHANNELS == 1 (input data is in integer format):
 *   - @c NR_BITS_PER_SAMPLE: 8 or 16
 *   - @c NR_SAMPLES_PER_SUBBAND: a multiple of 16
 * - if @c NR_CHANNELS > 1 (input data is in floating point format):
 *   - @c NR_SAMPLES_PER_CHANNEL: a multiple of 16
 * - @c NR_POLARIZATIONS: 2
 * - @c SUBBAND_WIDTH: a multiple of @c NR_CHANNELS
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "complex.cuh"

#if NR_CHANNELS == 1
#undef BANDPASS_CORRECTION  // TODO: Should this be an assert: this result in unexpected behaviour
#endif

typedef LOFAR::Cobalt::gpu::complex<float> complexfloat;
typedef LOFAR::Cobalt::gpu::complex<short> complexshort;
typedef LOFAR::Cobalt::gpu::complex<char> complexchar;
typedef  complexfloat (* OutputDataType)[NR_STATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_POLARIZATIONS];
#if NR_CHANNELS == 1
#if NR_BITS_PER_SAMPLE == 16
typedef  complexshort (* InputDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
#elif NR_BITS_PER_SAMPLE == 8
typedef  complexchar (* InputDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
#else
#error unsupport NR_BITS_PER_SAMPLE
#endif
#else
typedef  complexfloat (* InputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS];
#endif
typedef  const float (* DelaysType)[NR_BEAMS][NR_STATIONS][COMPLEX]; // 2 Polarizations; in seconds
typedef  const float (* PhaseOffsetsType)[NR_STATIONS][COMPLEX]; // 2 Polarizations; in radians
typedef  const float (* BandPassFactorsType)[NR_CHANNELS];

/**
 * This kernel perfroms three operations on the input data:
 * - Apply a fine delay by doing a per channel phase correction.
 * - Apply a bandpass correction to compensate for the errors introduced by the
 *   polyphase filter that produced the subbands. This error is deterministic,
 *   hence it can be fully compensated for.
 * - Transpose the data so that the time slices for each channel are placed
 *   consecutively in memory.
 *
 * @param[out] correctedDataPtr    pointer to output data of ::OutputDataType,
 *                                 a 3D array [station][channel][sample][complex]
 *                                 of ::complex (2 complex polarizations)
 * @param[in]  filteredDataPtr     pointer to input data; this can either be a
 *                                 4D array [station][polarization][sample][channel][complex]
 *                                 of ::fcomplex, or a 2D array [station][subband][complex]
 *                                 of ::short_complex2 or ::char_complex2,
 *                                 depending on the value of @c NR_CHANNELS
 * @param[in]  subbandFrequency    center freqency of the subband
 * @param[in]  beam                index number of the beam
 * @param[in]  delaysAtBeginPtr    pointer to delay data of ::DelaysType,
 *                                 a 2D array [beam][station] of float2 (real:
 *                                 2 polarizations), containing delays in
 *                                 seconds at begin of integration period
 * @param[in]  delaysAfterEndPtr   pointer to delay data of ::DelaysType,
 *                                 a 2D array [beam][station] of float2 (real:
 *                                 2 polarizations), containing delays in
 *                                 seconds after end of integration period
 * @param[in]  phaseOffsetsPtr     pointer to phase offset data of
 *                                 ::PhaseOffsetsType, a 1D array [station] of
 *                                 float2 (real: 2 polarizations), containing
 *                                 phase offsets in radians
 * @param[in]  bandPassFactorsPtr  pointer to bandpass correction data of
 *                                 ::BandPassFactorsType, a 1D array [channel] of
 *                                 float, containing bandpass correction factors
 */

#if NR_CHANNELS == 1
#error Support for 1 channel per subband has not yet been implemented due to missing int to float conversion routines.
#endif

extern "C" {
 __global__ void applyDelaysAndCorrectBandPass( complexfloat * correctedDataPtr,
                                                const complexfloat * filteredDataPtr,
                                                float subbandFrequency,
                                                unsigned beam,
                                                const float * delaysAtBeginPtr,
                                                const float * delaysAfterEndPtr,
                                                const float * phaseOffsetsPtr,
                                                const float * bandPassFactorsPtr)
{
  OutputDataType outputData = (OutputDataType) correctedDataPtr;
  InputDataType inputData = (InputDataType) filteredDataPtr;
  DelaysType delaysAtBegin = (DelaysType) delaysAtBeginPtr;
  DelaysType delaysAfterEnd = (DelaysType) delaysAfterEndPtr;
  PhaseOffsetsType phaseOffsets = (PhaseOffsetsType) phaseOffsetsPtr;
#if NR_CHANNELS > 1
  BandPassFactorsType bandPassFactors = (BandPassFactorsType) bandPassFactorsPtr;

  complexfloat tmp[16][17][2]; // one too wide to allow coalesced reads

  unsigned major = (blockIdx.x * blockDim.x + threadIdx.x) / 16;
  unsigned minor = (blockIdx.x * blockDim.x + threadIdx.x) % 16;
  unsigned channel = (blockIdx.y * blockDim.y + threadIdx.y) * 16;
#endif
  unsigned station = blockIdx.z * blockDim.z + threadIdx.z;

#if defined DELAY_COMPENSATION
#if NR_CHANNELS == 1
  float frequency = subbandFrequency;
#else
  float frequency = subbandFrequency - .5f * SUBBAND_BANDWIDTH + (channel + minor) * (SUBBAND_BANDWIDTH / NR_CHANNELS);
#endif
  float2 delayAtBegin = make_float2((*delaysAtBegin)[beam][station][0], (*delaysAtBegin)[beam][station][1]);
  float2 delayAfterEnd = make_float2((*delaysAfterEnd)[beam][station][0], (*delaysAfterEnd)[beam][station][1]);


  // Convert the fraction of sample duration (delayAtBegin/delayAfterEnd) to fractions of a circle.
  // Because we `undo' the delay, we need to rotate BACK.
  float pi2 = -2 * 3.1415926535f;
  float2 phiBegin = make_float2(pi2 * delayAtBegin.x, pi2 * delayAtBegin.y) ;
  float2 phiEnd = make_float2(pi2 * delayAfterEnd.x, pi2 * delayAfterEnd.y) ;

  float2 deltaPhi = make_float2((phiEnd.x - phiBegin.x) / NR_SAMPLES_PER_CHANNEL,
                                (phiEnd.y - phiBegin.y) / NR_SAMPLES_PER_CHANNEL);   
  
#if NR_CHANNELS == 1
  float2 myPhiBegin = make_float2(
                        (phiBegin.x + float(threadIdx.x) * deltaPhi.x) * frequency + (*phaseOffsets)[station][0],
                        (phiBegin.y + float(threadIdx.x) * deltaPhi.y) * frequency + (*phaseOffsets)[station][1]);
  float2 myPhiDelta = make_float2(
                         float(blockDim.x) * deltaPhi.x * frequency.x,
                         float(blockDim.x) * deltaPhi.y * frequency.y);
#else
  float2 myPhiBegin = make_float2(
                          (phiBegin.x + float(major) * deltaPhi.x) * frequency + (*phaseOffsets)[station][0],
                          (phiBegin.y + float(major) * deltaPhi.y) * frequency + (*phaseOffsets)[station][1]);
  float2 myPhiDelta = make_float2(16.0f * deltaPhi.x * frequency,
                                  16.0f * deltaPhi.y * frequency); // Magic constant: 16 is the time step we take in the samples
#endif

  complexfloat vX = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiBegin.x));  // This cast might be costly
  complexfloat vY = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiBegin.y));
  complexfloat dvX = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiDelta.x));
  complexfloat dvY = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiDelta.y));
#endif

#if defined BANDPASS_CORRECTION
  complexfloat weight((*bandPassFactors)[channel + minor]);
#endif

#if defined DELAY_COMPENSATION && defined BANDPASS_CORRECTION
  vX *= weight;
  vY *= weight;
#endif

#if NR_CHANNELS == 1
  for (unsigned time = threadIdx.x; time < NR_SAMPLES_PER_SUBBAND; time += blockDim.x) 
  {
    complexfloat sampleX = (*inputData)[station][time][0];
    complexfloat sampleY = (*inputData)[station][time][1];
#else
  for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time += 16) 
  {
    complexfloat sampleX = (*inputData)[station][0][time + major][channel + minor];    
    complexfloat sampleY = (*inputData)[station][1][time + major][channel + minor];   
#endif

#if defined DELAY_COMPENSATION    
    sampleX = sampleX * vX;
    sampleY = sampleY * vY;
    vX = vX * dvX;    // The calculation are with exponentional complex for: multiplication for correct phase shift
    vY = vY * dvY;
#elif defined BANDPASS_CORRECTION
    sampleX *= weight;
    sampleY *= weight;
#endif

#if NR_CHANNELS == 1
    (*outputData)[station][0][time][0] = sampleX;
    (*outputData)[station][0][time][1] = sampleY;
#else
    tmp[major][minor][0] = sampleX;
    tmp[major][minor][1] = sampleY;
    __syncthreads(); // Wait till all threads are here: we do a transform of the data
    (*outputData)[station][channel + major][time + minor][0] = tmp[minor][major][0];   
    (*outputData)[station][channel + major][time + minor][1] = tmp[minor][major][1];
    __syncthreads();

#endif
  }
}
}

