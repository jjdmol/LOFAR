
#include <hip/hip_runtime.h>
//# tKernelOccupancy.in_.cu: simple function to test occupancy predictions
//# Copyright (C) 2014  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

extern "C" {

  struct s {
    float f1, f2, f3, f4, f5, f6, f7, f8;
  };

  __global__ void blkLimit(struct s *buf, unsigned size)
  {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
      buf[i].f1 = buf[i].f1 + 1.0f;
    }
  }

  __device__ struct s helper(struct s s, struct s t) {
    s.f1 += s.f8 - s.f5 * t.f1;
    s.f2 *= s.f1 + s.f8 - t.f2;
    s.f3 -= s.f2 * s.f4 + t.f3;
    s.f4 += s.f3 + t.f4 / 2.0f;
    s.f5 *= s.f4 * t.f5 / 5.0f;
    s.f6 -= s.f5 / s.f3 + t.f6 / 11.0f;
    s.f7 += s.f6 + s.f1 * sqrt(t.f7);
    s.f8 *= s.f7 - s.f2 - exp(t.f8);
    return s;
  }

  __global__ void regsLimit(struct s *buf, unsigned size)
  {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
      // Try to use quite some reg.
      struct s l, m, n, o, p, q, r;
      do { 
        l = buf[i];
        m = helper(l, l);
        n = helper(m, l);
        o = helper(n, m);
        p = helper(o, n);
        q = helper(p, o);
        r = helper(q, p);
      } while (n.f3 < 0.0f);
      buf[i] = r;
    }
  }

  __global__ void shmemLimit(struct s *buf, unsigned size)
  {
    // Use quite some shmem per block (2 structs * 8 floats * 128 sz * 4 B = 8 kB).
    const unsigned SZ = 128;
    __shared__ struct s s1[SZ];
    __shared__ struct s s2[SZ];

    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
      struct s l = buf[i];
      s1[threadIdx.x] = l;
      struct s m = helper(l, l);
      s2[threadIdx.x] = m;
      __syncthreads();
      l = s1[threadIdx.x + 3 % SZ];
      m = s2[threadIdx.x + 7 % SZ]; 
      buf[i] = helper(m, l);
    }
  }

}

