// cuda-test.cu
// nvcc -o cuda-test cuda-test.cu

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstring>
#include <iostream>

using std::exit;
using std::memset;
using std::cout;
using std::cerr;
using std::endl;

__global__ void kfunc(float* data) {
  float v = data[0];
  //float v = data[1024*1024*1024]; // out of bounds
  data[0] = v + 1.0f;
}

int main() {
  hipError_t err;
  float *dptr;
  size_t len = 1024*1024;
  err = hipMalloc((void **)&dptr, len * sizeof(float));
  if (err != hipSuccess) { cerr << "hipMalloc failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }

  float *hptr = new float[32*len];
  memset(hptr, 0, len * sizeof(float));
  err = hipMemcpy(dptr, hptr, len * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) { cerr << "hipMemcpy (H2D) failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }

  // launch bad kernel
  int block_dim = 1;
  int grid_dim = 1;
  kfunc<<<grid_dim, block_dim>>>(dptr);
  err = hipGetLastError();
  if (err != hipSuccess) { cerr << "kernel launch failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }
  err = hipDeviceSynchronize();
  if (err != hipSuccess) { cerr << "hipDeviceSynchronize failed (launch) (expected): " << hipGetErrorString(err) << " (" << err << ")" << endl; }
  if (err != hipSuccess) {
    err = hipGetLastError();
    if (err != hipSuccess) { cerr << "resetting last error, which was (expected): " << hipGetErrorString(err) << " (" << err << ")" << endl; }
    err = hipGetLastError();
    if (err != hipSuccess) { cerr << "reset failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; }
  } 

  err = hipMemcpy(hptr, dptr, len * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess) { cerr << "hipMemcpy (D2H) failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; }

  for (unsigned i = 0; i < 16; i++) {
    cout << hptr[i] << " ";
  }
  cout << endl;

  delete[] hptr;
  err = hipFree(dptr);
  if (err != hipSuccess) { cerr << "hipFree failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }

  return 0;
}

