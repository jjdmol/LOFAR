#include "hip/hip_runtime.h"
// cu-test.cu
// nvcc -ptx cu-test.cu && nvcc -o cu-test cu-test.cu -lcuda
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>

extern "C" {
__global__ void kfunc(float* data) {
  float v = data[0];
  //float v = data[1024*1024*1024]; // out of bounds
  data[0] = v + 1.0f;
}
}

using std::exit;
using std::memset;
using std::cout;
using std::cerr;
using std::endl;

int main() {
  hipError_t r;
  r = hipInit(0);
  if (r != hipSuccess) { cerr << "hipInit failed: " << r << endl; exit(1); }

  hipDevice_t dev;
  int ordinal = 0;
  r = hipDeviceGet(&dev, ordinal);
  if (r != hipSuccess) { cerr << "hipDeviceGet failed: " << r << endl; exit(1); }

  hipCtx_t ctx;
  unsigned int flags = hipDeviceScheduleAuto;
  r = hipCtxCreate(&ctx, flags, dev);
  if (r != hipSuccess) { cerr << "hipCtxCreate failed: " << r << endl; exit(1); }

  hipDeviceptr_t dptr;
  size_t len = 1024*1024;
  r = hipMalloc(&dptr, len * sizeof(float));
  if (r != hipSuccess) { cerr << "hipMalloc failed:" << r << endl; }

  hipDeviceptr_t dptr2;
  len = 1024ULL*1024*1024*1024; // too large
  r = hipMalloc(&dptr2, len * sizeof(float));
  if (r != hipSuccess) { cerr << "hipMalloc failed (2) (expected): " << r << endl; }

  hipDeviceptr_t dptr3;
  len = 1024*1024; // works again after previous erroneous alloc
  r = hipMalloc(&dptr3, len * sizeof(float));
  if (r != hipSuccess) { cerr << "hipMalloc failed (3): " << r << endl; }

  float *hptr = new float[32*len];
  memset(hptr, 0, len * sizeof(float));

  hipStream_t stream;
  r = hipStreamCreateWithFlags(&stream, 0);
  if (r != hipSuccess) { cerr << "hipStreamCreateWithFlags failed: " << r << endl; exit(1); }

  r = hipMemcpyHtoDAsync(dptr, hptr, len * sizeof(float), stream);
  //r = hipMemcpyHtoDAsync(dptr, hptr, 32*len * sizeof(float), stream); // GPU buffer overflow
  if (r != hipSuccess) { cerr << "hipMemcpyHtoDAsync failed: " << r << endl; }

  r = hipStreamSynchronize(stream);
  if (r != hipSuccess) { cerr << "hipStreamSynchronize failed (HtoD): " << r << endl; }

  hipModule_t kmodule;
  r = hipModuleLoad(&kmodule, "cu-test.ptx");
  if (r != hipSuccess) { cerr << "hipModuleLoad failed: " << r << endl; exit(1); }

  hipFunction_t kfunc;
  r = hipModuleGetFunction(&kfunc, kmodule, "kfunc");
  if (r != hipSuccess) { cerr << "hipModuleGetFunction failed: " << r << endl; exit(1); }

  // async launch bad kernel
  void *args = &dptr;
  r = hipModuleLaunchKernel(kfunc, /*gridDim: */1, 1, 1, /*blockDim: */1, 1, 1,
                     /*dynShmemBytes: */0, stream, &args, NULL);
  if (r != hipSuccess) { cerr << "hipModuleLaunchKernel failed: " << r << endl; exit(1); }

  r = hipStreamSynchronize(stream);
  if (r != hipSuccess) { cerr << "hipStreamSynchronize failed (launch) (expected): " << r << endl; }

  r = hipMemcpyDtoHAsync(hptr, dptr, len * sizeof(float), stream);
  if (r != hipSuccess) { cerr << "hipMemcpyDtoHAsync failed: " << r << endl; }

  r = hipStreamSynchronize(stream);
  if (r != hipSuccess) { cerr << "hipStreamSynchronize failed (DtoH): " << r << endl; }

  for (unsigned i = 0; i < 16; i++) {
    cout << hptr[i] << " ";
  }
  cout << endl;


  delete[] hptr;
  r = hipFree(dptr);
  if (r != hipSuccess) { cerr << "hipFree failed: " << r << endl; exit(1); }
  // delete stream, context, ...

  return 0;
}

