#include "hip/hip_runtime.h"
//# BandPass.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/** @file
 * This file contains a CUDA implementation of the GPU kernel for the 
 bandpass correction. It can also transpose the data (pol to dim 0).
 *
 * Usually, this kernel will be run after the polyphase filter kernel FIR.cl. In
 * that case, the input data for this kernel is already in floating point format
 * (@c NR_CHANNELS > 1). However, if this kernel is the first in row, then the
 * input data is still in integer format (@c NR_CHANNELS == 1), and this kernel
 * needs to do the integer-to-float conversion. If we do BANDPASS_CORRECTION
 * (implies NR_CHANNELS > 1), then we typically also want to transpose the pol
 * dim to the stride 1 dim (@c DO_TRANSPOSE).
 *
 * @attention The following pre-processor variables must be supplied when
 * compiling this program. Please take the pre-conditions for these variables
 * into account:
 * - @c NR_POLARIZATIONS: 2
  *
 */

#include "gpu_math.cuh"
#include "IntToFloat.cuh"


typedef  fcomplex (* OutputDataType)[NR_STATIONS][NR_CHANNELS_1 * NR_CHANNELS_2][NR_SAMPLES_PER_CHANNEL][NR_POLARIZATIONS];
typedef  fcomplex (* InputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_CHANNELS_1][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS_2];
typedef  const float (* BandPassFactorsType)[NR_CHANNELS_1 * NR_CHANNELS_2];

/**
 * This kernel performs (up to) three operations on the input data:
 * - Apply a fine delay by doing a per channel phase correction.
 * - Apply a bandpass correction to compensate for the errors introduced by the
 *   polyphase filter that produced the subbands. This error is deterministic,
 *   hence it can be fully compensated for.
 * - Transpose the data so that the time slices for each channel are placed
 *   consecutively in memory.
 *
 * @param[out] correctedDataPtr    pointer to output data of ::OutputDataType,
 *                                 a 3D array [station][channel][sample][complex]
 *                                 of ::complex (2 complex polarizations)
 * @param[in]  filteredDataPtr     pointer to input data; this can either be a
 *                                 4D array [station][polarization][sample][channel][complex]
 *                                 of ::fcomplex, or a 2D array [station][subband][complex]
 *                                 of ::short_complex2 or ::char_complex2,
 *                                 depending on the value of @c NR_CHANNELS
 * @param[in]  bandPassFactorsPtr  pointer to bandpass correction data of
 *                                 ::BandPassFactorsType, a 1D array [channel] of
 *                                 float, containing bandpass correction factors
 */
#define SHARED
extern "C" {
 __global__ void correctBandPass( fcomplex * correctedDataPtr,
                                  const fcomplex * filteredDataPtr,
                                  const float * bandPassFactorsPtr)
{
  
  OutputDataType outputData = (OutputDataType) correctedDataPtr;
  InputDataType inputData   = (InputDataType)  filteredDataPtr;

  
  BandPassFactorsType bandPassFactors = (BandPassFactorsType) bandPassFactorsPtr;
  
  // fasted dims
  unsigned chan2        = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned sample       = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned station      = blockIdx.z * blockDim.z + threadIdx.z;



  for (unsigned idx_channel1 = 0; idx_channel1 < NR_CHANNELS_1; ++idx_channel1)
  {
    
    unsigned combined_channel = idx_channel1 * NR_CHANNELS_2 + chan2;
    float weight((*bandPassFactors)[combined_channel]);
    // Read from memory in the quickest dimension (optimal)
    fcomplex sampleX = (*inputData)[station][0][idx_channel1][sample][chan2];
    fcomplex sampleY = (*inputData)[station][1][idx_channel1][sample][chan2];
    
    sampleX.x *= weight;
    sampleX.y *= weight;
    sampleY.x *= weight;
    sampleY.y *= weight;

#if defined SHARED
    //  4.6 ms
    // Write blocks of memory 16 by 16 in chared
    __shared__ fcomplex tmp[16][16 + 1][2]; // one too wide to avoid bank-conflicts on read
    // 
    tmp[threadIdx.y][threadIdx.x][0] = sampleX;
    tmp[threadIdx.y][threadIdx.x][1] = sampleY;
    __syncthreads();
    // Write data to global with the sample moving the 
    // Use correct coallesced writes: 2 ms
    (*outputData)[station][idx_channel1 * NR_CHANNELS_2 + blockIdx.x * blockDim.x + threadIdx.y][blockIdx.y * blockDim.y + threadIdx.x][0] = tmp[threadIdx.x][threadIdx.y][0];
    (*outputData)[station][idx_channel1 * NR_CHANNELS_2 + blockIdx.x * blockDim.x + threadIdx.y][blockIdx.y * blockDim.y + threadIdx.x][1] = tmp[threadIdx.x][threadIdx.y][1];
    __syncthreads();
#else
    // 5.5 ms
    (*outputData)[station][combined_channel][sample][0] = sampleX; //tmp[minor][major][0];
    (*outputData)[station][combined_channel][sample][1] = sampleY; //tmp[minor][major][1];
#endif
  }
}
}

