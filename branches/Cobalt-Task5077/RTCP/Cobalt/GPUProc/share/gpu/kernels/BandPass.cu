#include "hip/hip_runtime.h"
//# BandPass.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/** @file
 * This file contains a CUDA implementation of the GPU kernel for the 
 bandpass correction. It can also transpose the data (pol to dim 0).
 *
 * Usually, this kernel will be run after the polyphase filter kernel FIR.cl. In
 * that case, the input data for this kernel is already in floating point format
 * (@c NR_CHANNELS > 1). However, if this kernel is the first in row, then the
 * input data is still in integer format (@c NR_CHANNELS == 1), and this kernel
 * needs to do the integer-to-float conversion. If we do BANDPASS_CORRECTION
 * (implies NR_CHANNELS > 1), then we typically also want to transpose the pol
 * dim to the stride 1 dim (@c DO_TRANSPOSE).
 *
 * @attention The following pre-processor variables must be supplied when
 * compiling this program. Please take the pre-conditions for these variables
 * into account:
 * - @c NR_POLARIZATIONS: 2
  *
 */

#include "gpu_math.cuh"
#include "IntToFloat.cuh"


typedef  fcomplex (* OutputDataType)[NR_STATIONS][NR_CHANNELS_1 * NR_CHANNELS_2][NR_SAMPLES_PER_CHANNEL][NR_POLARIZATIONS];
typedef  fcomplex (* InputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_CHANNELS_1][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS_2];
typedef  const float (* BandPassFactorsType)[NR_CHANNELS_1 * NR_CHANNELS_2];

/**
 * This kernel performs (up to) three operations on the input data:
 * - Apply a fine delay by doing a per channel phase correction.
 * - Apply a bandpass correction to compensate for the errors introduced by the
 *   polyphase filter that produced the subbands. This error is deterministic,
 *   hence it can be fully compensated for.
 * - Transpose the data so that the time slices for each channel are placed
 *   consecutively in memory.
 *
 * @param[out] correctedDataPtr    pointer to output data of ::OutputDataType,
 *                                 a 3D array [station][channel][sample][complex]
 *                                 of ::complex (2 complex polarizations)
 * @param[in]  filteredDataPtr     pointer to input data; this can either be a
 *                                 4D array [station][polarization][sample][channel][complex]
 *                                 of ::fcomplex, or a 2D array [station][subband][complex]
 *                                 of ::short_complex2 or ::char_complex2,
 *                                 depending on the value of @c NR_CHANNELS
 * @param[in]  bandPassFactorsPtr  pointer to bandpass correction data of
 *                                 ::BandPassFactorsType, a 1D array [channel] of
 *                                 float, containing bandpass correction factors
 */

extern "C" {
 __global__ void correctBandPass( fcomplex * correctedDataPtr,
                                  const fcomplex * filteredDataPtr,
                                  const float * bandPassFactorsPtr)
{
  
  OutputDataType outputData = (OutputDataType) correctedDataPtr;
  InputDataType inputData   = (InputDataType)  filteredDataPtr;

  
  BandPassFactorsType bandPassFactors = (BandPassFactorsType) bandPassFactorsPtr;
  
  // fasted dims
  unsigned chan2        = (blockIdx.x * blockDim.x + threadIdx.x) % NR_CHANNELS_2  ;
  unsigned sample       = (blockIdx.x * blockDim.x + threadIdx.x) / NR_CHANNELS_2;
  
  // second dim
  unsigned station      = blockIdx.y * blockDim.y + threadIdx.y;

  for (unsigned idx_channel1 = 0; idx_channel1 < NR_CHANNELS_1; ++idx_channel1)
  {
    
    unsigned combined_channel = idx_channel1 * NR_CHANNELS_2 + chan2;
    float weight((*bandPassFactors)[combined_channel]);
    fcomplex sampleX = (*inputData)[station][0][idx_channel1][sample][chan2];
    fcomplex sampleY = (*inputData)[station][1][idx_channel1][sample][chan2];
    
    sampleX.x *= weight;
    sampleX.y *= weight;
    sampleY.x *= weight;
    sampleY.y *= weight;

//// Support all variants of NR_CHANNELS and DO_TRANSPOSE for testing etc.
//// Transpose: data order is [station][channel][time][pol]
//    __shared__ fcomplex tmp[NR_SAMPLES_PER_CHANNEL][NR_CHANNELS_2 + 1][2]; // one too wide to avoid bank-conflicts on read
//
//    tmp[sample][chan2][0] = sampleX;
//    tmp[sample][chan2][1] = sampleY;
//    __syncthreads();

//    (*outputData)[station][combined_channel][sample][0] = tmp[chan2][sample][0];
//    (*outputData)[station][combined_channel][sample][1] = tmp[chan2][sample][1];
    
    // No use of shared mem
    (*outputData)[station][combined_channel][sample][0] = sampleX; //tmp[minor][major][0];
    (*outputData)[station][combined_channel][sample][1] = sampleY; //tmp[minor][major][1];
    //__syncthreads();
  }
}
}

