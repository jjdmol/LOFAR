#include "hip/hip_runtime.h"
//# FFTShift.cu: multyply odd samples with -1 for correct fft functionality
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include "gpu_math.cuh"


#if !(NR_SAMPLES_PER_CHANNEL >= 1)
#error Precondition violated: NR_SAMPLES_PER_CHANNEL >= 1
#endif

#if !(NR_STATIONS >= 1)
#error Precondition violated: NR_STATIONS >= 1
#endif

#if !(NR_POLARIZATIONS == 2)
#error Precondition violated: NR_POLARIZATIONS == 2
#endif

#if !(NR_CHANNELS >= 1)
#error Precondition violated: NR_CHANNELS >= 1
#endif

typedef float2(*DataType)[NR_STATIONS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];

/**
 * Shift the zero-frequency component to the center of the spectrum.
 * This kernel swaps the half-spaces of the channel dimension 
 * so that the negative frequencies are placed to
 * the left of the positive frequencies. We do this by modulating the samples
 * with exp(-j*pi), which results in a shift over pi in the frequency
 * domain. More information can be found in any decent book on digital signal
 * processing. 
 * @param[data] a 4-D array
 *              [station][polarizations][nr_channels][n_samples_channel]
 *              of complex floats.
 *
 * Required preprocessor symbols:
 * - NR_SAMPLES_PER_CHANNEL: > 0
 * - NR_STATIONS           : > 0
 * - NR_POLARIZATIONS      : ==2
 * - NR_CHANNELS           : > 0
 *
 * Execution configuration:
 * - Use a 3D thread block. (sample, station, channel) size < 1024.
 * - Use a 3D grid dim. (sample, station, channel) channel < 64
 */

extern "C" {
__global__ void FFTShift(DataType data)
{
  unsigned sample  = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned station = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned channel = blockIdx.z * blockDim.z + threadIdx.z;

  // Set the odd samples 
  signed factor = 1 - 2 * (sample % 2);  //multiplication that results in -1 or
              // odd samples (faster then an if statement)
  (*data)[station][0][channel][sample] = 
                            (*data)[station][0][channel][sample] * factor;
  (*data)[station][1][channel][sample] = 
                            (*data)[station][1][channel][sample] * factor;

}
}
