#include "hip/hip_runtime.h"
//# FFTShift.cu: multyply odd samples with -1 for correct fft functionality
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include "gpu_math.cuh"

typedef float2 (*OuputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_SUBBAND];
typedef float2 (*InputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_SUBBAND];

/**
 * This kernel performs a conversion of the integer valued input to floats and
 * transposes the data to get per station: first all samples with polX, then polY.
 * - It supports 8 and 16 bit (char and short) input, which is selectable using
 *   the define NR_BITS_PER_SAMPLE
 * - In 8 bit mode the converted samples with value -128 are clamped to -127.0f
 *
 * @param[out] convertedDataPtr    pointer to output data of ConvertedDataType,
 *                                 a 4D array [station][polarizations][n_samples_subband][complex]
 *                                 of floats (2 complex polarizations).
 * @param[in]  sampledDataPtr      pointer to input data; this can either be a
 *                                 4D array [station][n_samples_subband][polarizations][complex]
 *                                 of shorts or chars, depending on NR_BITS_PER_SAMPLE.
 *
 * Required preprocessor symbols:
 * - NR_SAMPLES_PER_CHANNEL: > 0
 * - NR_BITS_PER_SAMPLE: 8 or 16
 *
 * Execution configuration:
 * - Use a 1D thread block. No restrictions.
 * - Use a 2D grid dim, where the x dim has 1 block and the y dim represents the
 *   number of stations (i.e. antenna fields).
 */

extern "C" {
__global__ void FFTShift(void *outputDataPtr,
                           const void *inputDataPtr)
{
  InputDataType input = (InputDataType)inputDataPtr;
  OuputDataType output = (OuputDataType)outputDataPtr;

  // fasted dims
  unsigned sample        = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned station       = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned channel      = blockIdx.z * blockDim.z + threadIdx.z;

  //if (false)
  //if (sample % 2 != 0) // if an odd sample
  //{
  //  float2 pol0 = (*input)[station][0][channel][sample];
  //  float2 pol1 = (*input)[station][1][channel][sample];
  //  (*output)[station][0][channel][sample] = make_float2(0,0); // *-1.0f;
  //  (*output)[station][1][channel][sample] = make_float2(0, 0); // *-1.0f;
  //}



}

}

