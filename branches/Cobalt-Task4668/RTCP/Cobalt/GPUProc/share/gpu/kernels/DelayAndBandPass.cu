#include "hip/hip_runtime.h"
//# DelayAndBandPass.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/** @file
 * This file contains an Cuda implementation of the GPU kernel for the delay
 * and bandpass correction.
 *
 * Usually, this kernel will be run after the polyphase filter kernel FIR.cl. In
 * that case, the input data for this kernel is already in floating point format
 * (@c NR_CHANNELS > 1). However, if this kernel is the first in row, then the
 * input data is still in integer format (@c NR_CHANNELS == 1), and this kernel
 * needs to do the integer-to-float conversion. If we do BANDPASS_CORRECTION
 * (implies NR_CHANNELS > 1), then we also transpose the pol dim to stride-1.
 *
 * @attention The following pre-processor variables must be supplied when
 * compiling this program. Please take the pre-conditions for these variables
 * into account:
 * - @c NR_CHANNELS: 1 or a multiple of 16
 * - if @c NR_CHANNELS == 1 (input data is in integer format):
 *   - @c NR_BITS_PER_SAMPLE: 8 or 16
 *   - @c NR_SAMPLES_PER_SUBBAND: a multiple of 16
 * - if @c NR_CHANNELS > 1 (input data is in floating point format):
 *   - @c NR_SAMPLES_PER_CHANNEL: a multiple of 16
 * - @c NR_POLARIZATIONS: 2
 * - @c SUBBAND_WIDTH: a multiple of @c NR_CHANNELS
 */

#include "complex.cuh" // TODO: get rid of this: causes warning that is probably not a bug, but does point to a lot of unneeded inits in our __shared__ decl
#include "IntToFloat.cuh"

#if NR_CHANNELS == 1
   // #chnl==1 && BANDPASS_CORRECTION is rejected on the CPU early, (TODO)
   // but once here, don't do difficult and adjust cleanly here.
#  undef BANDPASS_CORRECTION
#endif

// We need to transpose the pol dim to stride-1 iff we have BANDPASS_CORRECTION,
// both for our correlation and (for the 2 instantiations of this kernel) for our
// beamforming pipelines. (BANDPASS_CORRECTION implies >1 channel per subband.)
#if defined BANDPASS_CORRECTION
#  define DO_TRANSPOSE
#endif

typedef LOFAR::Cobalt::gpu::complex<float> complexfloat;
typedef LOFAR::Cobalt::gpu::complex<short> complexshort;
typedef LOFAR::Cobalt::gpu::complex<char> complexchar;

#if defined DO_TRANSPOSE
typedef  complexfloat (* OutputDataType)[NR_STATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_POLARIZATIONS];
#else
typedef  complexfloat (* OutputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];
#endif

#if NR_CHANNELS == 1
#  if NR_BITS_PER_SAMPLE == 16
typedef  complexshort (* InputDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
#  elif NR_BITS_PER_SAMPLE == 8
typedef  complexchar (* InputDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
#  else
#    error unsupported NR_BITS_PER_SAMPLE
#  endif
#else
typedef  complexfloat (* InputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS];
#endif
typedef  const float (* DelaysType)[NR_SAPS][NR_STATIONS][NR_POLARIZATIONS]; // 2 Polarizations; in seconds
typedef  const float (* PhaseOffsetsType)[NR_STATIONS][NR_POLARIZATIONS]; // 2 Polarizations; in radians
typedef  const float (* BandPassFactorsType)[NR_CHANNELS];

/**
 * This kernel performs (up to) three operations on the input data:
 * - Apply a fine delay by doing a per channel phase correction.
 * - Apply a bandpass correction to compensate for the errors introduced by the
 *   polyphase filter that produced the subbands. This error is deterministic,
 *   hence it can be fully compensated for.
 * - Transpose the data so that the time slices for each channel are placed
 *   consecutively in memory.
 *
 * @param[out] correctedDataPtr    pointer to output data of ::OutputDataType,
 *                                 a 3D array [station][channel][sample][complex]
 *                                 of ::complex (2 complex polarizations)
 * @param[in]  filteredDataPtr     pointer to input data; this can either be a
 *                                 4D array [station][polarization][sample][channel][complex]
 *                                 of ::fcomplex, or a 2D array [station][subband][complex]
 *                                 of ::short_complex2 or ::char_complex2,
 *                                 depending on the value of @c NR_CHANNELS
 * @param[in]  subbandFrequency    center freqency of the subband
 * @param[in]  beam                index number of the beam
 * @param[in]  delaysAtBeginPtr    pointer to delay data of ::DelaysType,
 *                                 a 2D array [beam][station] of float2 (real:
 *                                 2 polarizations), containing delays in
 *                                 seconds at begin of integration period
 * @param[in]  delaysAfterEndPtr   pointer to delay data of ::DelaysType,
 *                                 a 2D array [beam][station] of float2 (real:
 *                                 2 polarizations), containing delays in
 *                                 seconds after end of integration period
 * @param[in]  phaseOffsetsPtr     pointer to phase offset data of
 *                                 ::PhaseOffsetsType, a 1D array [station] of
 *                                 float2 (real: 2 polarizations), containing
 *                                 phase offsets in radians
 * @param[in]  bandPassFactorsPtr  pointer to bandpass correction data of
 *                                 ::BandPassFactorsType, a 1D array [channel] of
 *                                 float, containing bandpass correction factors
 */

extern "C" {
 __global__ void applyDelaysAndCorrectBandPass( complexfloat * correctedDataPtr,
                                                const complexfloat * filteredDataPtr,
                                                float subbandFrequency,
                                                unsigned beam,
                                                const float * delaysAtBeginPtr,
                                                const float * delaysAfterEndPtr,
                                                const float * phaseOffsetsPtr,
                                                const float * bandPassFactorsPtr)
{
  OutputDataType outputData = (OutputDataType) correctedDataPtr;
  InputDataType inputData = (InputDataType) filteredDataPtr;
#if defined DELAY_COMPENSATION
  DelaysType delaysAtBegin = (DelaysType) delaysAtBeginPtr;
  DelaysType delaysAfterEnd = (DelaysType) delaysAfterEndPtr;
  PhaseOffsetsType phaseOffsets = (PhaseOffsetsType) phaseOffsetsPtr;
#endif
#if NR_CHANNELS > 1
  BandPassFactorsType bandPassFactors = (BandPassFactorsType) bandPassFactorsPtr;

  unsigned major = (blockIdx.x * blockDim.x + threadIdx.x) / 16;
  unsigned minor = (blockIdx.x * blockDim.x + threadIdx.x) % 16;
  unsigned channel = (blockIdx.y * blockDim.y + threadIdx.y) * 16;
#endif
  unsigned station = blockIdx.z * blockDim.z + threadIdx.z;

#if defined DELAY_COMPENSATION
#if NR_CHANNELS == 1
  float frequency = subbandFrequency;
#else
  float frequency = subbandFrequency - .5f * SUBBAND_BANDWIDTH + (channel + minor) * (SUBBAND_BANDWIDTH / NR_CHANNELS);
#endif
  float2 delayAtBegin  = make_float2((*delaysAtBegin) [beam][station][0], (*delaysAtBegin) [beam][station][1]);
  float2 delayAfterEnd = make_float2((*delaysAfterEnd)[beam][station][0], (*delaysAfterEnd)[beam][station][1]);


  // Convert the fraction of sample duration (delayAtBegin/delayAfterEnd) to fractions of a circle.
  // Because we `undo' the delay, we need to rotate BACK.
  float pi2 = -2.0f * 3.1415926535f;
  float2 phiBegin = make_float2(pi2 * delayAtBegin.x, pi2 * delayAtBegin.y);
  float2 phiEnd   = make_float2(pi2 * delayAfterEnd.x, pi2 * delayAfterEnd.y);

  float2 deltaPhi = make_float2((phiEnd.x - phiBegin.x) / NR_SAMPLES_PER_CHANNEL,
                                (phiEnd.y - phiBegin.y) / NR_SAMPLES_PER_CHANNEL);   
  
#if NR_CHANNELS == 1
  float2 myPhiBegin = make_float2(
                        (phiBegin.x + float(threadIdx.x) * deltaPhi.x) * frequency + (*phaseOffsets)[station][0],
                        (phiBegin.y + float(threadIdx.x) * deltaPhi.y) * frequency + (*phaseOffsets)[station][1]);
  float2 myPhiDelta = make_float2(
                         float(blockDim.x) * deltaPhi.x * frequency.x,
                         float(blockDim.x) * deltaPhi.y * frequency.y);
#else
  float2 myPhiBegin = make_float2(
                          (phiBegin.x + float(major) * deltaPhi.x) * frequency + (*phaseOffsets)[station][0],
                          (phiBegin.y + float(major) * deltaPhi.y) * frequency + (*phaseOffsets)[station][1]);
  // Magic constant 16 is the time step we take in the samples
  float2 myPhiDelta = make_float2(16.0f * deltaPhi.x * frequency,
                                  16.0f * deltaPhi.y * frequency);
#endif

  complexfloat vX = LOFAR::Cobalt::gpu::cosisin(myPhiBegin.x);
  complexfloat vY = LOFAR::Cobalt::gpu::cosisin(myPhiBegin.y);
  complexfloat dvX = LOFAR::Cobalt::gpu::cosisin(myPhiDelta.x);
  complexfloat dvY = LOFAR::Cobalt::gpu::cosisin(myPhiDelta.y);
#endif

#if defined BANDPASS_CORRECTION
  complexfloat weight((*bandPassFactors)[channel + minor]);
#endif

#if defined DELAY_COMPENSATION && defined BANDPASS_CORRECTION
  vX *= weight;
  vY *= weight;
#endif

#if NR_CHANNELS == 1
  for (unsigned time = threadIdx.x; time < NR_SAMPLES_PER_SUBBAND; time += blockDim.x)
  {
    complexfloat sampleX = complexfloat(
                                convertIntToFloat((*inputData)[station][time][0].real()),
                                convertIntToFloat((*inputData)[station][time][0].imag()));
    complexfloat sampleY = complexfloat(
                                convertIntToFloat((*inputData)[station][time][1].real()),
                                convertIntToFloat((*inputData)[station][time][1].imag()));
#else
  for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time += 16)
  {
    complexfloat sampleX = complexfloat(
                        (*inputData)[station][0][time + major][channel + minor].real(),
                        (*inputData)[station][0][time + major][channel + minor].imag());
    complexfloat sampleY = complexfloat(
                        (*inputData)[station][1][time + major][channel + minor].real(),
                        (*inputData)[station][1][time + major][channel + minor].imag());
#endif

#if defined DELAY_COMPENSATION    
    sampleX = sampleX * vX;
    sampleY = sampleY * vY;
    // The calculations are with exponentional complex for: multiplication for correct phase shift
    vX = vX * dvX;
    vY = vY * dvY;
#elif defined BANDPASS_CORRECTION
    sampleX *= weight;
    sampleY *= weight;
#endif

#if defined DO_TRANSPOSE
    __shared__ complexfloat tmp[16][17][2]; // one too wide to avoid bank-conflicts on read

    tmp[major][minor][0] = sampleX;
    tmp[major][minor][1] = sampleY;
    __syncthreads();
    (*outputData)[station][channel + major][time + minor][0] = tmp[minor][major][0];   
    (*outputData)[station][channel + major][time + minor][1] = tmp[minor][major][1];
    __syncthreads();
#else
    (*outputData)[station][0][time][0] = sampleX;
    (*outputData)[station][0][time][1] = sampleY;
#endif
  }
}
}

