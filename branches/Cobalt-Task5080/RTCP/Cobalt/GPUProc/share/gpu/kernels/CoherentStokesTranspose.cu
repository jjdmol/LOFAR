#include "hip/hip_runtime.h"
//# Transpose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/*!
 * Performs data transposition from the output of the beamformer kernel to
 * a data order suitable for an inverse FFT.
 * Parallelisation is performed over the TABs and number of samples (time).
 *
 * We have 4 dimensions, but CUDA thread blocks can be up to three.
 * Mangle the TAB and sample dimension in to dim 0 (x).
 *
 * The kernel needs to determine for each thread whether to read and separately
 * whether to write back a sample, because the number of TABs may not divide by
 * the 16x16 thread arrangement (even though we have a 1D thread block).
 *
 * \param[out] TransposedDataType      4D output array of samples. For each TAB and pol, a spectrum per time step of complex floats.
 * \param[in]  ComplexVoltagesType     3D input array of samples (last dim (pol) is implicit). For each channel, the TABs per time step of two complex floats.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_SAMPLES_PER_CHANNEL  | multiple of 16 and > 0  | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2
 *
 * Execution configuration:
 * - LocalWorkSize = 1 dimensional; (256, 1, 1) is in use. Multiples of (32, 1, 1) may work too.
 * - GlobalWorkSize = 3 dimensional:
 *   + inner dim (x): always 1 block
 *   + middle dim (y): 16 TABs can be processed in a block. Number of blocks required, rounded-up. eg for 17 tabs we need 2 blocks
 *   + outer dim (z): 16 samples per channel can be processed in a block. Number of blocks required (fits exactly). 32 channels is 2 blocks
 */
#include "gpu_math.cuh"

typedef fcomplex (*OutputDataType)[NR_TABS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL]; //last dims of this needs to be swapped

typedef float4 fcomplex2;
// last dim within float4 is NR_POLARIZATIONS
typedef fcomplex2 (*InputDataType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS]; // [NR_POLARIZATIONS];

// fcomplex2 speedup 5%

extern "C"
__global__ void coherentStokesTranspose(void *OutputDataPtr,
                          const void *InputDataPtr)
{ 
  OutputDataType outputData = (OutputDataType) OutputDataPtr;
  InputDataType inputData = (InputDataType) InputDataPtr;
  

//#define BLOCKREORDER 1
#ifdef BLOCKREORDER
  // Reorder the grid block indexing: to prevent global memory access bank conflicts
  // blockIdx.x start at the diagonal
  unsigned block_x;
  unsigned block_y;
  if ( NR_SAMPLES_PER_CHANNEL == NR_TABS)  // if workload is square
  {
     block_x= (blockIdx.y + blockIdx.x) % gridDim.x; 
     block_y = blockIdx.x;
  } 
  else
  {
    unsigned bid = blockIdx.x + gridDim.x * blockIdx.y;
     block_y = bid % gridDim.y;
     block_x = ((bid % gridDim.y) + block_y) % gridDim.x;    

  }
  unsigned tab           = block_x * blockDim.x + threadIdx.x;
  unsigned sample        = block_y * blockDim.y + threadIdx.y;
#else
  unsigned block_x      = blockIdx.x ;
  unsigned block_y      = blockIdx.y ; 
#endif

  unsigned tab           = block_x * blockDim.x + threadIdx.x;
  unsigned sample        = block_y * blockDim.y + threadIdx.y;

  for (unsigned idx = 0; idx < 1; ++idx)  // Do more work in a kernel allows hiding of preparation work
  {
    unsigned channel       = blockIdx.z * blockDim.z + idx;

    // Use shared memory for the transpose
    __shared__ fcomplex2 tmp[16][16 + 1];  // plus one to prevent bank conflicts in shared memory
    
    tmp[threadIdx.y][threadIdx.x] = (*inputData) [channel][sample][tab];
    __syncthreads();  // assures all writes are done

    // Reassign the tab and sample to allow the threadIdx.x to write in the highest dimension
    tab           = block_x * blockDim.x + threadIdx.y;
    sample        = block_y * blockDim.y + threadIdx.x;
    (*outputData)[tab][0][channel][sample] = make_float2(tmp[threadIdx.x][threadIdx.y].x,
                                                         tmp[threadIdx.x][threadIdx.y].y) ;

    (*outputData)[tab][1][channel][sample] = make_float2(tmp[threadIdx.x][threadIdx.y].z,
                                                         tmp[threadIdx.x][threadIdx.y].w) ;

    __syncthreads();  // assures all writes are done
  }
}
    // OPtimal write to global memory
    //  6.43%  757.00us         1  757.00us  757.00us  757.00us  coherentStokesTranspose
    //   6.06%  710.37us         1  710.37us  710.37us  710.37us  coherentStokesTranspose float4
    // 5.51  642.34us       1  642.34us  642.34us  642.34us  coherentStokesTranspose 4 channel in for loop
    // 5.74  671.17us       1  671.17us  671.17us  671.17us  coherentStokesTranspose 8 channel
    // 5.23  607.53us       1  607.53us  607.53us  607.53us  coherentStokesTranspose
    // With blovkr reordering idx = 0  4.70  543.05us       1  543.05us  543.05us  543.05us  coherentStokesTranspose
