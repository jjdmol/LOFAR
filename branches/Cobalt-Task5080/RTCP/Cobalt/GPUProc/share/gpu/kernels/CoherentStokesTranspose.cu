#include "hip/hip_runtime.h"
//# Transpose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/*!
 * Performs data transposition from the output of the beamformer kernel to
 * a data order suitable for an inverse FFT.
 * Parallelisation is performed over the TABs and number of samples (time).
 *
 * We have 4 dimensions, but CUDA thread blocks can be up to three.
 * Mangle the TAB and sample dimension in to dim 0 (x).
 *
 * The kernel needs to determine for each thread whether to read and separately
 * whether to write back a sample, because the number of TABs may not divide by
 * the 16x16 thread arrangement (even though we have a 1D thread block).
 *
 * \param[out] TransposedDataType      4D output array of samples. For each TAB and pol, a spectrum per time step of complex floats.
 * \param[in]  ComplexVoltagesType     3D input array of samples (last dim (pol) is implicit). For each channel, the TABs per time step of two complex floats.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_SAMPLES_PER_CHANNEL  | multiple of 16 and > 0  | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2
 *
 * Execution configuration:
 * - LocalWorkSize = 1 dimensional; (256, 1, 1) is in use. Multiples of (32, 1, 1) may work too.
 * - GlobalWorkSize = 3 dimensional:
 *   + inner dim (x): always 1 block
 *   + middle dim (y): 16 TABs can be processed in a block. Number of blocks required, rounded-up. eg for 17 tabs we need 2 blocks
 *   + outer dim (z): 16 samples per channel can be processed in a block. Number of blocks required (fits exactly). 32 channels is 2 blocks
 */
#include "gpu_math.cuh"

typedef fcomplex (*OutputDataType)[NR_TABS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL]; //last dims of this needs to be swapped

// last dim within float4 is NR_POLARIZATIONS
typedef fcomplex (*InputDataType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS][NR_POLARIZATIONS]; // [NR_POLARIZATIONS];





extern "C"
__global__ void coherentStokesTranspose(void *OutputDataPtr,
                          const void *InputDataPtr)
{

  OutputDataType outputData = (OutputDataType) OutputDataPtr;
  InputDataType inputData = (InputDataType) InputDataPtr;
     
  // fasted dims
  unsigned tab          = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned sample       = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned channel       = blockIdx.z * blockDim.z + threadIdx.z;

  (*outputData)[tab][0][channel][sample] = (*inputData) [channel][sample][tab][0];
  (*outputData)[tab][1][channel][sample] = (*inputData)[channel][sample][tab][1];




  // Shared memory to perform a transpose in shared memory
  //// one too wide to avoid bank-conflicts on read
  //// 16 by 16 limitation for the channels2 and samples per channel are caused by the
  //// dimensions of this array
  //// TODO: Increasing to 32 x 32 allows for a speedup of 13%
  //__shared__ fcomplex tmp[16][16 + 1][2];

  //for (unsigned idx_channel1 = 0; idx_channel1 < NR_CHANNELS_1; ++idx_channel1)
  //{
  //  unsigned combined_channel = idx_channel1 * NR_CHANNELS_2 + chan2;
  //  float weight((*bandPassFactors)[combined_channel]);

  //  // Read from memory in the quickest dimension (optimal)
  //  fcomplex sampleX = (*inputData)[station][0][idx_channel1][sample][chan2];
  //  fcomplex sampleY = (*inputData)[station][1][idx_channel1][sample][chan2];

  //  sampleX.x *= weight;
  //  sampleX.y *= weight;
  //  sampleY.x *= weight;
  //  sampleY.y *= weight;

  //  // Write the data to shared memory

  //  tmp[threadIdx.y][threadIdx.x][0] = sampleX;
  //  tmp[threadIdx.y][threadIdx.x][1] = sampleY;
  //  __syncthreads();  // assures all writes are done

  //  // Now write from shared to global memory.
  //  unsigned chan_index = idx_channel1 * NR_CHANNELS_2 + blockIdx.x * blockDim.x + threadIdx.y;
  //  // Use the threadidx.x for the highest array index: coalesced writes to the global memory
  //  unsigned sample_index = blockIdx.y * blockDim.y + threadIdx.x;

  //  (*outputData)[station][chan_index][sample_index][0] = tmp[threadIdx.x][threadIdx.y][0];  // The threadIdx.y in shared mem is not a problem
  //  (*outputData)[station][chan_index][sample_index][1] = tmp[threadIdx.x][threadIdx.y][1];
  //  __syncthreads();  // assure are writes are done. The next for itteration reuses the array
  //}

#ifdef SKIP
  /*
   41.89%  5.5789ms         1  5.5789ms  5.5789ms  5.5789ms  [CUDA memcpy HtoD]
   38.80%  5.1674ms         1  5.1674ms  5.1674ms  5.1674ms  [CUDA memcpy DtoH]
   19.30%  2.5703ms         1  2.5703ms  2.5703ms  2.5703ms  coherentStokesTranspose
  */
  unsigned tabBase = 16 * blockDim.y * blockIdx.y + threadIdx.y;
  unsigned timeBase = 16 * blockDim.z * blockIdx.z + threadIdx.z;

  unsigned tabOffsetR = threadIdx.x & 15;
  unsigned tabR = tabBase + tabOffsetR;
  unsigned timeOffsetR = threadIdx.x >> 4;
  unsigned timeR = timeBase + timeOffsetR;
  bool doR = NR_TABS % 16 == 0 || tabR < NR_TABS;

  unsigned tabOffsetW = threadIdx.x >> 4;
  unsigned tabW = tabBase + tabOffsetW;
  unsigned timeOffsetW = threadIdx.x & 15;
  unsigned timeW = timeBase + timeOffsetW;
  bool doW = NR_TABS % 16 == 0 || tabW < NR_TABS;

  for (int channel = 0; channel < NR_CHANNELS; channel++) 
  {
    if (doR)
    {
      tmp[tabOffsetR][timeOffsetR] = (*complexVoltages)[channel][timeR][tabR];
    }

    __syncthreads();

    if (doW) {
      float4 sample = tmp[tabOffsetW][timeOffsetW];
      (*transposedData)[tabW][0][channel][timeW] = make_float2(sample.x, sample.y);
      (*transposedData)[tabW][1][channel][timeW] = make_float2(sample.z, sample.w);
    }

    __syncthreads();

  }
  #endif
}
