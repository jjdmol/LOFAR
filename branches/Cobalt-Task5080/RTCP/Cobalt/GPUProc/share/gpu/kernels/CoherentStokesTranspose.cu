#include "hip/hip_runtime.h"
//# CoherentStokesTranspose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/*!
 * Performs data transposition from the output of the beamformer kernel to
 * a data order suitable for an inverse FFT.
 * Parallelisation is performed over the TABs and number of samples (time).
 *
 *
 * \param[out] OutputDataType      4D output array of samples. For each TAB and pol, a spectrum per time step of complex floats.
 * \param[in]  InputDataType       3D input array of samples (last dim (pol) is implicit). For each channel, the TABs per time step of two complex floats.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_SAMPLES_PER_CHANNEL  | multiple of 16 and > 0  | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | multiple of 16 and > 0  | number of Tied Array Beams to create
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2
 *
 * Execution configuration:
 * - LocalWorkSize = 2 dimensional; (16, 16, 1) is in use.
 * - GlobalWorkSize = 3 dimensional:
 *   + inner dim (x): nr tabs (/ 16)
 *   + middle dim (y): nr samples ( /16)
 *   + outer dim (z): number of channels (/1)
 */
#include "gpu_math.cuh"

typedef fcomplex (*OutputDataType)[NR_TABS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL]; //last dims of this needs to be swapped

typedef float4 fcomplex2;
// Allows for better memory access
typedef fcomplex2 (*InputDataType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS]; // [NR_POLARIZATIONS];


extern "C"
__global__ void coherentStokesTranspose(void *OutputDataPtr,
                          const void *InputDataPtr)
{ 
  OutputDataType outputData = (OutputDataType) OutputDataPtr;
  InputDataType inputData = (InputDataType) InputDataPtr;
  
  // Reorder the grid block indexing: to prevent global memory access bank conflicts
  // blockIdx.x starts at the diagonal
  unsigned block_x;
  unsigned block_y;
  if ( NR_SAMPLES_PER_CHANNEL == NR_TABS)  // if workload is square
  {
     block_x= (blockIdx.y + blockIdx.x) % gridDim.x; 
     block_y = blockIdx.x;
  } 
  else
  {
    unsigned bid = blockIdx.x + gridDim.x * blockIdx.y;
    block_y = bid % gridDim.y;
    block_x = ((bid % gridDim.y) + block_y) % gridDim.x;    
  }

  unsigned tab           = block_x * blockDim.x + threadIdx.x;
  unsigned sample        = block_y * blockDim.y + threadIdx.y;
  unsigned channel       = blockIdx.z * blockDim.z ;

  // Use shared memory for the transpose
  __shared__ fcomplex2 tmp[16][16 + 1];  // plus one to prevent bank conflicts in shared memory

  tmp[threadIdx.y][threadIdx.x] = (*inputData) [channel][sample][tab];
  __syncthreads();  // assures all writes are done

  // Reassign the tab and sample to allow the threadIdx.x to write in the highest dimension
  tab           = block_x * blockDim.x + threadIdx.y;
  sample        = block_y * blockDim.y + threadIdx.x;

  // Do the write to global mem
  (*outputData)[tab][0][channel][sample] = make_float2(tmp[threadIdx.x][threadIdx.y].x,
    tmp[threadIdx.x][threadIdx.y].y) ;
  (*outputData)[tab][1][channel][sample] = make_float2(tmp[threadIdx.x][threadIdx.y].z,
    tmp[threadIdx.x][threadIdx.y].w) ;

  __syncthreads();  // assures all writes are done
}
