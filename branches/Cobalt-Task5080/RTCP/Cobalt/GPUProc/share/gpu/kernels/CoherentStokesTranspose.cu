#include "hip/hip_runtime.h"
//# Transpose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/*!
 * Performs data transposition from the output of the beamformer kernel to
 * a data order suitable for an inverse FFT.
 * Parallelisation is performed over the TABs and number of samples (time).
 *
 * We have 4 dimensions, but CUDA thread blocks can be up to three.
 * Mangle the TAB and sample dimension in to dim 0 (x).
 *
 * The kernel needs to determine for each thread whether to read and separately
 * whether to write back a sample, because the number of TABs may not divide by
 * the 16x16 thread arrangement (even though we have a 1D thread block).
 *
 * \param[out] TransposedDataType      4D output array of samples. For each TAB and pol, a spectrum per time step of complex floats.
 * \param[in]  ComplexVoltagesType     3D input array of samples (last dim (pol) is implicit). For each channel, the TABs per time step of two complex floats.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_SAMPLES_PER_CHANNEL  | multiple of 16 and > 0  | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2
 *
 * Execution configuration:
 * - LocalWorkSize = 1 dimensional; (256, 1, 1) is in use. Multiples of (32, 1, 1) may work too.
 * - GlobalWorkSize = 3 dimensional:
 *   + inner dim (x): always 1 block
 *   + middle dim (y): 16 TABs can be processed in a block. Number of blocks required, rounded-up. eg for 17 tabs we need 2 blocks
 *   + outer dim (z): 16 samples per channel can be processed in a block. Number of blocks required (fits exactly). 32 channels is 2 blocks
 */
#include "gpu_math.cuh"

typedef fcomplex (*OutputDataType)[NR_TABS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL]; //last dims of this needs to be swapped

typedef float4 fcomplex2;
// last dim within float4 is NR_POLARIZATIONS
typedef fcomplex2 (*InputDataType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS]; // [NR_POLARIZATIONS];

// fcomplex2 speedup 5%

extern "C"
__global__ void coherentStokesTranspose(void *OutputDataPtr,
                          const void *InputDataPtr)
{

  OutputDataType outputData = (OutputDataType) OutputDataPtr;
  InputDataType inputData = (InputDataType) InputDataPtr;
     
  // fasted dims
  unsigned tab           = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned sample        = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned channel       = blockIdx.z * blockDim.z + threadIdx.z;

  __shared__ fcomplex2 tmp[16][16 + 1];

  tmp[threadIdx.y][threadIdx.x] = (*inputData) [channel][sample][tab];
  __syncthreads();  // assures all writes are done

  // OPtimal write to global memory
  //  6.43%  757.00us         1  757.00us  757.00us  757.00us  coherentStokesTranspose
  //   6.06%  710.37us         1  710.37us  710.37us  710.37us  coherentStokesTranspose float4

  tab           = blockIdx.x * blockDim.x + threadIdx.y;
  sample        = blockIdx.y * blockDim.y + threadIdx.x;
  (*outputData)[tab][0][channel][sample] = make_float2(tmp[threadIdx.x][threadIdx.y].x,
                                                       tmp[threadIdx.x][threadIdx.y].y) ;

  (*outputData)[tab][1][channel][sample] = make_float2(tmp[threadIdx.x][threadIdx.y].z,
                                                       tmp[threadIdx.x][threadIdx.y].w) ;

  __syncthreads();  // assures all writes are done

}
