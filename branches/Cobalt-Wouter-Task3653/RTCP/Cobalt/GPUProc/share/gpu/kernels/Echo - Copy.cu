#include "hip/hip_runtime.h"
//# Correlator.cu
//#
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id: Echo.cu 25342 2013-06-14 12:11:09Z klijn $

// \file
// This file contains a CUDA implementation of the GPU kernel for the
// correlator. It computes correlations between all pairs of stations
// (baselines) and X,Y polarizations, including auto-correlations.

#include <hip/hip_runtime.h>

#include "gpu_math.cuh"
#include "complex.cuh"

typedef LOFAR::Cobalt::gpu::complex<float> fcomplex;

// The external input: Currently input is connected to all neurons,
typedef fcomplex(*ExternalInputsDataType)[NR_INPUTS][NR_TIMESTEPS];
typedef fcomplex(*ExternalInputsWeightsDataType)[NR_INPUTS][NR_NEURONS];
typedef fcomplex(*BaseWeightsDataType)[NR_BASENETWORKS][NR_NEURONS][NR_NEURONS];

// The output: only the activations: The output can be calculated (cheaply??)
typedef fcomplex(*NeuronActivationsDataType)[NR_BASENETWORKS][NR_TIMESTEPS][NR_NEURONS];

extern "C"  __global__ void echo(void *neuronActivationsPtr,
                     const void *externalInputsPtr, 
                     const void *externalInputsWeightsPtr,
                     const void *baseWeightsTypePtr) 
{
  NeuronActivationsDataType neuronActivations = (NeuronActivationsDataType) neuronActivationsPtr;
  ExternalInputsDataType externalInputs = (ExternalInputsDataType) externalInputsPtr;
  ExternalInputsWeightsDataType externalInputsWeights = (ExternalInputsWeightsDataType) externalInputsWeightsPtr;
  BaseWeightsDataType baseWeights = (BaseWeightsDataType) baseWeightsTypePtr;
  // Local array for activations
  fcomplex neuron_activations[NR_NEURONS]; 
  fcomplex neuron_activations_pref[NR_NEURONS]; 

  for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
  {
    neuron_activations[idx_neuron] = fcomplex(0,0);
  }

  for (unsigned idx_timestep = 1; idx_timestep < NR_TIMESTEPS; ++ idx_timestep)
  {
    for (unsigned idx_synaps = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
    {
      // Get the correct neuron activation, this is candidate for the y dimension block
      //fcomplex activation = neuron_activations_pref[idx_neuron]; 
      // external input
      // NEXT STEP: Add the inputs and validate the activations
      fcomplex neuron_activation_deltasum = 0;
      fcomplex neuron_activation_pref = =0;
      for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
      {
        //neuron_activation_pref = neuron_activation[idx_neuron]; // deze bevat dus 
        neuron_activation_delta = (*baseWeights)[0][idx_neuron][idx_synaps] * 
                                    neuron_activations[idx_synaps];
        ////neuron_activation_pref = neuron_activation[idx_neuron]; // deze bevat dus 
        neuron_activation_deltasum += neuron_activation_delta
        neuron_activation[idx_synaps] += neuron_activation_delta;

        // kan ik hier 

        //neuron_activation[idx_neuron] -= neuron_activation_pref; // kan ik met deze de huidige activatie uitrekenen?
        //// Wat als ik hier loop: Dit heb ik ooit eens op papier getekend.
      } 
        // ******************************************************************
        fcomplex neuron_activation_delta = (*baseWeights)[0][idx_neuron][idx_synaps] * 
                                    neuron_activations[idx_synaps];
        //neuron_activation_pref = neuron_activation[idx_neuron]; // deze bevat dus 
        neuron_activation[idx_synaps] += neuron_activation_delta;


        neuron_activation[idx_neuron] -= neuron_activation_pref; // kan ik met deze de huidige activatie uitrekenen?
      }

      // No output function!!
      neuron_activations[idx_neuron] =
          neuron_activation;
    }
    //michien diagonaal over de neuron input heen om de copy van de activiteit te voorkomen?
    for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
    {
      neuron_activations_pref[idx_neuron] = neuron_activations[idx_neuron];
    }
    for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
    {
      (*neuronActivations)[0][idx_timestep][idx_neuron] = neuron_activations[idx_neuron];
    }
    __syncthreads();
  //// Local array for activations
  //fcomplex neuron_activations[NR_NEURONS]; 
  //fcomplex neuron_activations_pref[NR_NEURONS]; 

  //for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
  //{
  //  neuron_activations_pref[idx_neuron] = fcomplex(0,0);
  //}

  //for (unsigned idx_timestep = 1; idx_timestep < NR_TIMESTEPS; ++ idx_timestep)
  //{
  //  for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
  //  {
  //    // Get the correct neuron activation, this is candidate for the y dimension block
  //    fcomplex neuron_activation = neuron_activations_pref[idx_neuron]; 
  //    // external input
  //    // NEXT STEP: Add the inputs and validate the activations
  //    for (unsigned idx_synaps = 0; idx_synaps < NR_NEURONS; ++ idx_synaps)
  //    {
  //      neuron_activation += (*baseWeights)[0][idx_neuron][idx_synaps] * 
  //                                  neuron_activations_pref[idx_synaps];

  //    }
  //    // No output function!!
  //    neuron_activations[idx_neuron] =
  //        neuron_activation;
  //  }
  //  //michien diagonaal over de neuron input heen om de copy van de activiteit te voorkomen?
  //  for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
  //  {
  //    neuron_activations_pref[idx_neuron] = neuron_activations[idx_neuron];
  //  }
  //  for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
  //  {
  //    (*neuronActivations)[0][idx_timestep][idx_neuron] = neuron_activations[idx_neuron];
  //  }
  //  __syncthreads();
  }
  //(*neuronActivations)[0][0][0] = fcomplex(20,20);
  //(*neuronActivations)[0][0][0] = (*externalInputs)[0][0];
  return;
}
