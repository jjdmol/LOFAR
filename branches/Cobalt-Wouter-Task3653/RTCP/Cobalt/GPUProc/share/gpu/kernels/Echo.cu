#include "hip/hip_runtime.h"
//# Correlator.cu
//#
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

// \file
// This file contains a CUDA implementation of the GPU kernel for the
// correlator. It computes correlations between all pairs of stations
// (baselines) and X,Y polarizations, including auto-correlations.

#include <hip/hip_runtime.h>

#include "gpu_math.cuh"
#include "complex.cuh"
#include "hiprand/hiprand_kernel.h"

typedef LOFAR::Cobalt::gpu::complex<float> fcomplex;

// The external input: Currently input is connected to all neurons,
typedef fcomplex(*ExternalInputsDataType)[NR_INPUTS][NR_TIMESTEPS];
typedef fcomplex(*ExternalInputsWeightsDataType)[NR_INPUTS][NR_NEURONS];
typedef fcomplex(*BaseWeightsDataType)[NR_BASENETWORKS][NR_NEURONS][NR_NEURONS];

// The output: only the activations: The output can be calculated (cheaply??)
typedef fcomplex(*NeuronActivationsDataType)[NR_BASENETWORKS][NR_TIMESTEPS][NR_SOLUTION_PARALEL][NR_NEURONS];

extern "C"  __global__ void echo(void *neuronActivationsPtr,
                     const void *externalInputsPtr, 
                     const void *externalInputsWeightsPtr,
                     const void *baseWeightsTypePtr,
                     void * globalStatePtr
                     ) 
{

  NeuronActivationsDataType neuronActivations = (NeuronActivationsDataType) neuronActivationsPtr;
  ExternalInputsDataType externalInputs = (ExternalInputsDataType) externalInputsPtr;
  ExternalInputsWeightsDataType externalInputsWeights = (ExternalInputsWeightsDataType) externalInputsWeightsPtr;
  BaseWeightsDataType baseWeights = (BaseWeightsDataType) baseWeightsTypePtr;
  hiprandState* globalState = (hiprandState*)globalStatePtr;

  

  // **************************************************************************
  // Get the weights into local memory
  unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ fcomplex shared_baseWeights[NR_NEURONS][NR_NEURONS];
  // Candidate for loop unrolling:
  for (unsigned idx_copy = 0;
                idx_copy < (NR_NEURONS * NR_NEURONS) / NR_SOLUTION_PARALEL;
                ++idx_copy)
  {
    unsigned idx = idx_copy * NR_SOLUTION_PARALEL + tid;
    
    shared_baseWeights[idx /  NR_NEURONS][idx % NR_NEURONS] = 
       (*baseWeights)[0][idx /  NR_NEURONS][idx % NR_NEURONS];
  }

  //// *******************************************************************************
  // Local array for activations
  fcomplex neuron_activations_pref[NR_NEURONS];   // Candidate for register variable!!
  for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
  {
    neuron_activations_pref[idx_neuron] = fcomplex(0,0);
  }

  // *******************************************************************************
  // Random numbers to be added to the default weight matrix
  // Will contain the random numbers to added to weights
  // Maybee use a const array to shift through the weigths
  // depending on the threads idx.
  // 256 weights and 256 threads //  misschien plus NR_NEURONS om deling in the most inner loop te ontweiken.
  hiprandState_t state = globalState[threadIdx.x];
  unsigned salt = 12;
  hiprand_init(salt, threadIdx.x,0, &state);  //maybee this can be done better: salt in the randomization

  __shared__ fcomplex weightRandom[NR_NEURONS * NR_NEURONS]; // use 512 complex and shift through them. this alles omition of 
  ////candidate unrolling:
  for (unsigned idx_copy = 0; idx_copy < NR_NEURONS * NR_NEURONS / NR_SOLUTION_PARALEL; ++ idx_copy)
  {
    unsigned idx = idx_copy * NR_SOLUTION_PARALEL + tid;
    // Do not go over the bounds of the random weight matrix
    if ( idx > NR_NEURONS * NR_NEURONS )
      weightRandom[idx] = 
          fcomplex(hiprand_normal(&state), hiprand_normal(&state));  //lognormal aslo candidate?
  }
  __syncthreads();

  for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
  {
    neuron_activations_pref[idx_neuron] = (*neuronActivations)[0][0][threadIdx.x][idx_neuron];
  }

  // ************************************************************************
  // From here all code is fully parallel and no sync between threads is needed
  for (unsigned idx_timestep = 1; idx_timestep < NR_TIMESTEPS; ++ idx_timestep)
  {
    // Under the assumption that we have 128 threads running
    // either have the first wave create the random numbers or
    // have each thread create a random number.
  
    for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
    {
      // Get the correct neuron activation, including decay
      fcomplex neuron_activation =  neuron_activations_pref[idx_neuron]; 
      neuron_activation *= ACTIVATION_DECAY;
      // external input
      // NEXT STEP: Add the inputs and validate the activations
      unsigned saved_neuron_idx = (idx_neuron + threadIdx.x) * NR_NEURONS ;
      for (unsigned idx_synaps = 0; idx_synaps < NR_NEURONS; ++ idx_synaps)
      {
        neuron_activation += (shared_baseWeights[idx_neuron][idx_synaps] + 
           weightRandom[(saved_neuron_idx + idx_synaps) % (NR_NEURONS * NR_NEURONS)]) *
                             neuron_activations_pref[idx_synaps];
      }


      // No output function!!
      (*neuronActivations)[0][idx_timestep][threadIdx.x][idx_neuron] = neuron_activation;
    }
    for (unsigned idx_neuron = 0; idx_neuron < NR_NEURONS; ++ idx_neuron)
    {
      neuron_activations_pref[idx_neuron] = (*neuronActivations)[0][idx_timestep][threadIdx.x][idx_neuron];
    }
  }
  __syncthreads();
  
  return;
}
