#include "hip/hip_runtime.h"
//# BandPassCorrection.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/** @file
 * This file contains a CUDA implementation of the GPU kernel for the 
 * BandPassCorrection. It transposes the data: The FFT produces
 * for each sample X channels in the fastest dimension. The channels and samples
 * are transposed to allow faster processing in later stages.
 * The samples will end up in the fastest dimension ( the time line).
 *
 * @attention The following pre-processor variables must be supplied when
 * compiling this program. Please take the pre-conditions for these variables
 * into account:
 * - @c NR_POLARIZATIONS: 2
 * - @c NR_STATIONS: > 0
 * - @c NR_CHANNELS_1: a multiple of 16 
 * - @c NR_CHANNELS_2: > 0 
 * - @c NR_SAMPLES_PER_CHANNEL: > a multiple of 16
 * - @c DO_BANDPASS_CORRECTION: if defined, perform bandpass correction
 */

#include "gpu_math.cuh"

#if !(NR_POLARIZATIONS == 2)
#error Precondition violated: NR_POLARIZATIONS == 2
#endif

#if !(NR_STATIONS > 0)
#error Precondition violated: NR_STATIONS > 0
#endif

#if !(NR_CHANNELS_1 > 0 && NR_CHANNELS_1 % 16 == 0)
#error Precondition violated: NR_CHANNELS_1 > 0 && NR_CHANNELS_1 % 16 == 0
#endif

#if !(NR_CHANNELS_2 > 0)
#error Precondition violated: NR_CHANNELS_2 > 0
#endif

#if !(NR_SAMPLES_PER_CHANNEL > 0)
#error Precondition violated: NR_SAMPLES_PER_CHANNEL > 0
#endif

typedef  fcomplex (* OutputDataType)[NR_STATIONS][NR_CHANNELS_1 * NR_CHANNELS_2][NR_SAMPLES_PER_CHANNEL][NR_POLARIZATIONS];
typedef  fcomplex (* InputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_CHANNELS_1][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS_2];
typedef  const float (* BandPassFactorsType)[NR_CHANNELS_1 * NR_CHANNELS_2];

/**
 * This kernel performs on the input data:
 * - If the preprocessor variable \c DO_BANDPASS_CORRECTION is defined, apply a
 *   bandpass correction to compensate for the errors introduced by the
 *   polyphase filter that produced the subbands. This error is deterministic,
 *   hence it can be fully compensated for.
 * - Transpose the data so that the samples for each channel are placed
 *   consecutively in memory with both polarization next to each other.
 * - Note: This kernel is optimized for performance in dims samples and channel_1
 *   Previous version was optimized for channel_2 (still supported)
 *   
 *
 * @param[out] correctedDataPtr    pointer to output data of ::OutputDataType,
 *                                 a 4D array  [station][channels1 * channels2][samples][pol]
 *                                 of ::complex (2 complex polarizations)
 * @param[in]  intputDataPtr     pointer to input data; 
 *                               5D array  [station][pol][channels1][samples][channels2]
 * @param[in]  bandPassFactorsPtr  pointer to bandpass correction data of
 *                                 ::BandPassFactorsType, a 1D array [channels1 * channels2] of
 *                                 float, containing bandpass correction factors
 */
extern "C" {
__global__ void bandPassCorrection( fcomplex * outputDataPtr,
                                 const fcomplex * inputDataPtr,
                                 const float * bandPassFactorsPtr)
{ 
  OutputDataType outputData = (OutputDataType) outputDataPtr;
  InputDataType inputData   = (InputDataType)  inputDataPtr;

#if defined DO_BANDPASS_CORRECTION
  // Band pass to apply to the channels  
  BandPassFactorsType bandPassFactors = (BandPassFactorsType) bandPassFactorsPtr;
#endif

  // fasted dims
  unsigned sample = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned idx_channel1 = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned chan2 = blockIdx.z * blockDim.z + threadIdx.z;
  
  for (unsigned station = 0; station < NR_STATIONS; ++station)
  {
    // Read from global memory in the quickest dimension (optimal)
    fcomplex sampleX = (*inputData)[station][0][idx_channel1][sample][chan2];
    fcomplex sampleY = (*inputData)[station][1][idx_channel1][sample][chan2];
    unsigned chan_index = idx_channel1 * NR_CHANNELS_2 + chan2;

#if defined DO_BANDPASS_CORRECTION
    float weight((*bandPassFactors)[chan_index]);
    sampleX.x *= weight;
    sampleX.y *= weight;
    sampleY.x *= weight;
    sampleY.y *= weight;
#endif

    (*outputData)[station][chan_index][sample][0] = sampleX; 
    (*outputData)[station][chan_index][sample][1] = sampleY; 
  }
}
}


