#include "hip/hip_runtime.h"
extern "C" {
  __global__ void FIR_filter(void *filteredDataPtr)
  {
    int *filteredData =(int *)filteredDataPtr;
    unsigned cpr = blockIdx.x*blockDim.x+threadIdx.x;
    filteredData[cpr] = NVIDIA_CUDA;  // define will be set on the command line as a parameter to the nvcc compiler
    int test = 20;
    return;
  }
}