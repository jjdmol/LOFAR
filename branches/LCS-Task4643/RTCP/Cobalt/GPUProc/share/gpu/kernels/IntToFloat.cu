#include "hip/hip_runtime.h"
//# IntToFloat.cl
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include "IntToFloat.cuh"

#if NR_BITS_PER_SAMPLE == 16
typedef short2 SampleType;
#elif NR_BITS_PER_SAMPLE == 8
typedef char2 SampleType;
#else
#error unsupport NR_BITS_PER_SAMPLE
#endif

typedef  SampleType (*SampledDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
typedef  float2 (*ConvertedDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_SUBBAND];

/**
 * This kernel performs a conversion of the integer valued input to floats.
 * - It supports both 16 and 8 bits (short and char) input selectable using
 *   the define NR_BITS_PER_SAMPLE
 * - In 8 bit mode the converted chars with value -128 are clamped to a minimum of -127 
 *
 * @param[out] correctedDataPtr    pointer to output data of ::ConvertedDataType,
 *                                 a 4D array [station][polarizations][n_samples_subband][complex]
 *                                 of floats (2 complex polarizations)
 * @param[in]  SampledDataType     pointer to input data; this can either be a
 *                                 4D array [station][n_samples_subband][polarizations][complex]
 *                                 of shorts or chars. depending on NR_BITS_PER_SAMPLE.
 */

extern "C" {
 __global__ void intToFloat( void * convertedDataPtr,
                          const void * sampledDataPtr)
{
  ConvertedDataType convertedData = (ConvertedDataType) convertedDataPtr;
  SampledDataType sampledData = (SampledDataType) sampledDataPtr;
  // Use the y dim for selecting the station. blockDim.y is normally 1
  uint station = blockIdx.y * blockDim.y + threadIdx.y;
  
  // Step data with whole blocks allows for coalesced reads and writes
  for (uint time = threadIdx.x; time < NR_SAMPLES_PER_SUBBAND; time += blockDim.x) {
    // pol 1
    (*convertedData)[station][0][time] = make_float2(
			convertIntToFloat((*sampledData)[station][time][0].x),
            convertIntToFloat((*sampledData)[station][time][0].y));
    // pol 2
    (*convertedData)[station][1][time] = make_float2(
			convertIntToFloat((*sampledData)[station][time][1].x), 
            convertIntToFloat((*sampledData)[station][time][1].y));
    // TODO: Is a sync needed here? Dont think so but..
  }
}
}


