
#include "hip/hip_runtime.h"



#define COMPLEX 2       // do not change
#define NR_BITS_PER_SAMPLE 16
#define NR_STATION_FILTER_TAPS  16
#define USE_NEW_CORRELATOR
#define NR_POLARIZATIONS         2 
#define NR_TAPS                 16

#define NR_STATIONS 20
#define NR_SAMPLES_PER_CHANNEL 100
#define NR_CHANNELS 16

#if NR_BITS_PER_SAMPLE == 16
typedef signed short SampleType;
#elif NR_BITS_PER_SAMPLE == 8
typedef signed char SampleType;
#else
#error unsupported NR_BITS_PER_SAMPLE
#endif


typedef SampleType (*SampledDataType)[NR_STATIONS][NR_TAPS - 1 + NR_SAMPLES_PER_CHANNEL][NR_CHANNELS][NR_POLARIZATIONS * COMPLEX];
typedef float (*FilteredDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS][COMPLEX];
typedef const float (*WeightsType)[NR_CHANNELS][16];


/*!
 * Applies the Finite Input Response filter defined by the weightsPtr array
 * to the sampledDataPtr array. Output is written into the filteredDataPtr
 * array. The filter works on complex numbers. The weights are real values only.
 *
 * Input values are first converted to (complex) float.
 * The kernel also reorders the polarization dimension and expects the weights
 * per channel in reverse order. If an FFT is applied afterwards, the weights
 * of the odd channels are often supplied negated to get the resulting channels
 * in increasing order of frequency.
 *
 * \param[out] filteredDataPtr         4D output array of floats
 * \param[in]  sampledDataPtr          4D input array of signed chars or shorts
 * \param[in]  weightsPtr              2D per-channel FIR filter coefficient array of floats (considering float16 as a dim)
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values                | Description
 * ----------------------- | --------------------------- | -----------
 * NR_STATIONS             | >= 1                        | number of antenna fields
 * NR_TAPS                 | 1--16                       | number of FIR filtering coefficients
 * NR_SAMPLES_PER_CHANNEL  | multiple of NR_TAPS and > 0 | number of input samples per channel
 * NR_BITS_PER_SAMPLE      | 8 or 16                     | number of bits of signed integral value type of sampledDataPtr (TODO: support 4)
 * NR_CHANNELS             | multiple of 16 and > 0      | number of frequency channels per subband
 * NR_POLARIZATIONS        | power of 2                  | number of polarizations
 *
 * Execution configuration: (TODO: enforce using __attribute__ reqd_work_group_size)
 * - Work dim == 2  (can be 1 iff NR_STATIONS == 1)
 *     + Inner dim: the channel, pol, real/imag the thread processes
 *     + Outer dim: the station the thread processes
 * - Work group size: must divide global size, no other kernel restrictions
 * - Global size: (NR_CHANNELS * NR_POLARIZATIONS * 2, NR_STATIONS)
 *
 * TODO: convert complex dim to fcomplex (=float2 in math.cl) in device code and to complex<float> in host code.
 */
__global__ void FIR_filter( void *filteredDataPtr,
                          const void *sampledDataPtr,
                          const void *weightsPtr)
{
  SampledDataType sampledData = (SampledDataType) sampledDataPtr;
  FilteredDataType filteredData = (FilteredDataType) filteredDataPtr;
  WeightsType weightsData = (WeightsType) weightsPtr;

  unsigned cpr = blockIdx.x*blockDim.x+threadIdx.x;
#if 0
  // Straight index calc for NR_CHANNELS == 1
  uint pol_ri = cpr & 3;
  uint channel = cpr >> 2;
  uint ri = cpr & 1;
  uint pol = pol_ri >> 1;
#else
  unsigned ri = cpr & 1;
  unsigned channel = (cpr >> 1) % NR_CHANNELS;
  unsigned pol = (cpr >> 1) / NR_CHANNELS;
  unsigned pol_ri = (pol << 1) | ri;
#endif
  unsigned station = blockIdx.y*blockDim.y+threadIdx.y;

  //const float16 weights = (*weightsData)[channel];
  const float weights_s0 = (*weightsData)[channel][0];
  const float weights_s1 = (*weightsData)[channel][1];
  const float weights_s2 = (*weightsData)[channel][2];
  const float weights_s3 = (*weightsData)[channel][3];
  const float weights_s4 = (*weightsData)[channel][4];
  const float weights_s5 = (*weightsData)[channel][5];
  const float weights_s6 = (*weightsData)[channel][6];
  const float weights_s7 = (*weightsData)[channel][7];
  const float weights_s8 = (*weightsData)[channel][8];
  const float weights_s9 = (*weightsData)[channel][9];
  const float weights_sA = (*weightsData)[channel][10];
  const float weights_sB = (*weightsData)[channel][11];
  const float weights_sC = (*weightsData)[channel][12];
  const float weights_sD = (*weightsData)[channel][13];
  const float weights_sE = (*weightsData)[channel][14];
  const float weights_sF = (*weightsData)[channel][15];

  //float16 delayLine;
  float delayLine_s0, delayLine_s1, delayLine_s2, delayLine_s3, 
        delayLine_s4, delayLine_s5, delayLine_s6, delayLine_s7, 
        delayLine_s8, delayLine_s9, delayLine_sA, delayLine_sB,
        delayLine_sC, delayLine_sD, delayLine_sE, delayLine_sF;
  

  delayLine_s0 = (*sampledData)[station][0][channel][pol_ri];
  delayLine_s1 = (*sampledData)[station][1][channel][pol_ri];
  delayLine_s2 = (*sampledData)[station][2][channel][pol_ri];
  delayLine_s3 = (*sampledData)[station][3][channel][pol_ri];
  delayLine_s4 = (*sampledData)[station][4][channel][pol_ri];
  delayLine_s5 = (*sampledData)[station][5][channel][pol_ri];
  delayLine_s6 = (*sampledData)[station][6][channel][pol_ri];
  delayLine_s7 = (*sampledData)[station][7][channel][pol_ri];
  delayLine_s8 = (*sampledData)[station][8][channel][pol_ri];
  delayLine_s9 = (*sampledData)[station][9][channel][pol_ri];
  delayLine_sA = (*sampledData)[station][10][channel][pol_ri];
  delayLine_sB = (*sampledData)[station][11][channel][pol_ri];
  delayLine_sC = (*sampledData)[station][12][channel][pol_ri];
  delayLine_sD = (*sampledData)[station][13][channel][pol_ri];
  delayLine_sE = (*sampledData)[station][14][channel][pol_ri];
  

  float sum_s0, sum_s1, sum_s2, sum_s3,
        sum_s4, sum_s5, sum_s6, sum_s7,
        sum_s8, sum_s9, sum_sA, sum_sB,
        sum_sC, sum_sD, sum_sE, sum_sF;

  for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time += NR_TAPS) 
  {
    delayLine_sF = (*sampledData)[station][time + NR_TAPS - 1 + 0][channel][pol_ri];
    sum_s0 = weights_sF * delayLine_s0;
    delayLine_s0 = (*sampledData)[station][time + NR_TAPS - 1 + 1][channel][pol_ri];
    sum_s0 += weights_sE * delayLine_s1;
    sum_s0 += weights_sD * delayLine_s2;
    sum_s0 += weights_sC * delayLine_s3;
    sum_s0 += weights_sB * delayLine_s4;
    sum_s0 += weights_sA * delayLine_s5;
    sum_s0 += weights_s9 * delayLine_s6;
    sum_s0 += weights_s8 * delayLine_s7;
    sum_s0 += weights_s7 * delayLine_s8;
    sum_s0 += weights_s6 * delayLine_s9;
    sum_s0 += weights_s5 * delayLine_sA;
    sum_s0 += weights_s4 * delayLine_sB;
    sum_s0 += weights_s3 * delayLine_sC;
    sum_s0 += weights_s2 * delayLine_sD;
    sum_s0 += weights_s1 * delayLine_sE;
    sum_s0 += weights_s0 * delayLine_sF;
    (*filteredData)[station][pol][time + 0][channel][ri] = sum_s0;

    sum_s1 = weights_sF * delayLine_s1;
    delayLine_s1 = (*sampledData)[station][time + NR_TAPS - 1 + 2][channel][pol_ri];
    sum_s1 += weights_sE * delayLine_s2;
    sum_s1 += weights_sD * delayLine_s3;
    sum_s1 += weights_sC * delayLine_s4;
    sum_s1 += weights_sB * delayLine_s5;
    sum_s1 += weights_sA * delayLine_s6;
    sum_s1 += weights_s9 * delayLine_s7;
    sum_s1 += weights_s8 * delayLine_s8;
    sum_s1 += weights_s7 * delayLine_s9;
    sum_s1 += weights_s6 * delayLine_sA;
    sum_s1 += weights_s5 * delayLine_sB;
    sum_s1 += weights_s4 * delayLine_sC;
    sum_s1 += weights_s3 * delayLine_sD;
    sum_s1 += weights_s2 * delayLine_sE;
    sum_s1 += weights_s1 * delayLine_sF;
    sum_s1 += weights_s0 * delayLine_s0;
    (*filteredData)[station][pol][time + 1][channel][ri] = sum_s1;

    sum_s2 = weights_sF * delayLine_s2;
    delayLine_s2 = (*sampledData)[station][time + NR_TAPS - 1 + 3][channel][pol_ri];
    sum_s2 += weights_sE * delayLine_s3;
    sum_s2 += weights_sD * delayLine_s4;
    sum_s2 += weights_sC * delayLine_s5;
    sum_s2 += weights_sB * delayLine_s6;
    sum_s2 += weights_sA * delayLine_s7;
    sum_s2 += weights_s9 * delayLine_s8;
    sum_s2 += weights_s8 * delayLine_s9;
    sum_s2 += weights_s7 * delayLine_sA;
    sum_s2 += weights_s6 * delayLine_sB;
    sum_s2 += weights_s5 * delayLine_sC;
    sum_s2 += weights_s4 * delayLine_sD;
    sum_s2 += weights_s3 * delayLine_sE;
    sum_s2 += weights_s2 * delayLine_sF;
    sum_s2 += weights_s1 * delayLine_s0;
    sum_s2 += weights_s0 * delayLine_s1;
    (*filteredData)[station][pol][time + 2][channel][ri] = sum_s2;

    sum_s3 = weights_sF * delayLine_s3;
    delayLine_s3 = (*sampledData)[station][time + NR_TAPS - 1 + 4][channel][pol_ri];
    sum_s3 += weights_sE * delayLine_s4;
    sum_s3 += weights_sD * delayLine_s5;
    sum_s3 += weights_sC * delayLine_s6;
    sum_s3 += weights_sB * delayLine_s7;
    sum_s3 += weights_sA * delayLine_s8;
    sum_s3 += weights_s9 * delayLine_s9;
    sum_s3 += weights_s8 * delayLine_sA;
    sum_s3 += weights_s7 * delayLine_sB;
    sum_s3 += weights_s6 * delayLine_sC;
    sum_s3 += weights_s5 * delayLine_sD;
    sum_s3 += weights_s4 * delayLine_sE;
    sum_s3 += weights_s3 * delayLine_sF;
    sum_s3 += weights_s2 * delayLine_s0;
    sum_s3 += weights_s1 * delayLine_s1;
    sum_s3 += weights_s0 * delayLine_s2;
    (*filteredData)[station][pol][time + 3][channel][ri] = sum_s3;

    sum_s4 = weights_sF * delayLine_s4;
    delayLine_s4 = (*sampledData)[station][time + NR_TAPS - 1 + 5][channel][pol_ri];
    sum_s4 += weights_sE * delayLine_s5;
    sum_s4 += weights_sD * delayLine_s6;
    sum_s4 += weights_sC * delayLine_s7;
    sum_s4 += weights_sB * delayLine_s8;
    sum_s4 += weights_sA * delayLine_s9;
    sum_s4 += weights_s9 * delayLine_sA;
    sum_s4 += weights_s8 * delayLine_sB;
    sum_s4 += weights_s7 * delayLine_sC;
    sum_s4 += weights_s6 * delayLine_sD;
    sum_s4 += weights_s5 * delayLine_sE;
    sum_s4 += weights_s4 * delayLine_sF;
    sum_s4 += weights_s3 * delayLine_s0;
    sum_s4 += weights_s2 * delayLine_s1;
    sum_s4 += weights_s1 * delayLine_s2;
    sum_s4 += weights_s0 * delayLine_s3;
    (*filteredData)[station][pol][time + 4][channel][ri] = sum_s4;

    sum_s5 = weights_sF * delayLine_s5;
    delayLine_s5 = (*sampledData)[station][time + NR_TAPS - 1 + 6][channel][pol_ri];
    sum_s5 += weights_sE * delayLine_s6;
    sum_s5 += weights_sD * delayLine_s7;
    sum_s5 += weights_sC * delayLine_s8;
    sum_s5 += weights_sB * delayLine_s9;
    sum_s5 += weights_sA * delayLine_sA;
    sum_s5 += weights_s9 * delayLine_sB;
    sum_s5 += weights_s8 * delayLine_sC;
    sum_s5 += weights_s7 * delayLine_sD;
    sum_s5 += weights_s6 * delayLine_sE;
    sum_s5 += weights_s5 * delayLine_sF;
    sum_s5 += weights_s4 * delayLine_s0;
    sum_s5 += weights_s3 * delayLine_s1;
    sum_s5 += weights_s2 * delayLine_s2;
    sum_s5 += weights_s1 * delayLine_s3;
    sum_s5 += weights_s0 * delayLine_s4;
    (*filteredData)[station][pol][time + 5][channel][ri] = sum_s5;

    sum_s6 = weights_sF * delayLine_s6;
    delayLine_s6 = (*sampledData)[station][time + NR_TAPS - 1 + 7][channel][pol_ri];
    sum_s6 += weights_sE * delayLine_s7;
    sum_s6 += weights_sD * delayLine_s8;
    sum_s6 += weights_sC * delayLine_s9;
    sum_s6 += weights_sB * delayLine_sA;
    sum_s6 += weights_sA * delayLine_sB;
    sum_s6 += weights_s9 * delayLine_sC;
    sum_s6 += weights_s8 * delayLine_sD;
    sum_s6 += weights_s7 * delayLine_sE;
    sum_s6 += weights_s6 * delayLine_sF;
    sum_s6 += weights_s5 * delayLine_s0;
    sum_s6 += weights_s4 * delayLine_s1;
    sum_s6 += weights_s3 * delayLine_s2;
    sum_s6 += weights_s2 * delayLine_s3;
    sum_s6 += weights_s1 * delayLine_s4;
    sum_s6 += weights_s0 * delayLine_s5;
    (*filteredData)[station][pol][time + 6][channel][ri] = sum_s6;

    sum_s7 = weights_sF * delayLine_s7;
    delayLine_s7 = (*sampledData)[station][time + NR_TAPS - 1 + 8][channel][pol_ri];
    sum_s7 += weights_sE * delayLine_s8;
    sum_s7 += weights_sD * delayLine_s9;
    sum_s7 += weights_sC * delayLine_sA;
    sum_s7 += weights_sB * delayLine_sB;
    sum_s7 += weights_sA * delayLine_sC;
    sum_s7 += weights_s9 * delayLine_sD;
    sum_s7 += weights_s8 * delayLine_sE;
    sum_s7 += weights_s7 * delayLine_sF;
    sum_s7 += weights_s6 * delayLine_s0;
    sum_s7 += weights_s5 * delayLine_s1;
    sum_s7 += weights_s4 * delayLine_s2;
    sum_s7 += weights_s3 * delayLine_s3;
    sum_s7 += weights_s2 * delayLine_s4;
    sum_s7 += weights_s1 * delayLine_s5;
    sum_s7 += weights_s0 * delayLine_s6;
    (*filteredData)[station][pol][time + 7][channel][ri] = sum_s7;

    sum_s8 = weights_sF * delayLine_s8;
    delayLine_s8 = (*sampledData)[station][time + NR_TAPS - 1 + 9][channel][pol_ri];
    sum_s8 += weights_sE * delayLine_s9;
    sum_s8 += weights_sD * delayLine_sA;
    sum_s8 += weights_sC * delayLine_sB;
    sum_s8 += weights_sB * delayLine_sC;
    sum_s8 += weights_sA * delayLine_sD;
    sum_s8 += weights_s9 * delayLine_sE;
    sum_s8 += weights_s8 * delayLine_sF;
    sum_s8 += weights_s7 * delayLine_s0;
    sum_s8 += weights_s6 * delayLine_s1;
    sum_s8 += weights_s5 * delayLine_s2;
    sum_s8 += weights_s4 * delayLine_s3;
    sum_s8 += weights_s3 * delayLine_s4;
    sum_s8 += weights_s2 * delayLine_s5;
    sum_s8 += weights_s1 * delayLine_s6;
    sum_s8 += weights_s0 * delayLine_s7;
    (*filteredData)[station][pol][time + 8][channel][ri] = sum_s8;

    sum_s9 = weights_sF * delayLine_s9;
    delayLine_s9 = (*sampledData)[station][time + NR_TAPS - 1 + 10][channel][pol_ri];
    sum_s9 += weights_sE * delayLine_sA;
    sum_s9 += weights_sD * delayLine_sB;
    sum_s9 += weights_sC * delayLine_sC;
    sum_s9 += weights_sB * delayLine_sD;
    sum_s9 += weights_sA * delayLine_sE;
    sum_s9 += weights_s9 * delayLine_sF;
    sum_s9 += weights_s8 * delayLine_s0;
    sum_s9 += weights_s7 * delayLine_s1;
    sum_s9 += weights_s6 * delayLine_s2;
    sum_s9 += weights_s5 * delayLine_s3;
    sum_s9 += weights_s4 * delayLine_s4;
    sum_s9 += weights_s3 * delayLine_s5;
    sum_s9 += weights_s2 * delayLine_s6;
    sum_s9 += weights_s1 * delayLine_s7;
    sum_s9 += weights_s0 * delayLine_s8;
    (*filteredData)[station][pol][time + 9][channel][ri] = sum_s9;

    sum_sA = weights_sF * delayLine_sA;
    delayLine_sA = (*sampledData)[station][time + NR_TAPS - 1 + 11][channel][pol_ri];
    sum_sA += weights_sE * delayLine_sB;
    sum_sA += weights_sD * delayLine_sC;
    sum_sA += weights_sC * delayLine_sD;
    sum_sA += weights_sB * delayLine_sE;
    sum_sA += weights_sA * delayLine_sF;
    sum_sA += weights_s9 * delayLine_s0;
    sum_sA += weights_s8 * delayLine_s1;
    sum_sA += weights_s7 * delayLine_s2;
    sum_sA += weights_s6 * delayLine_s3;
    sum_sA += weights_s5 * delayLine_s4;
    sum_sA += weights_s4 * delayLine_s5;
    sum_sA += weights_s3 * delayLine_s6;
    sum_sA += weights_s2 * delayLine_s7;
    sum_sA += weights_s1 * delayLine_s8;
    sum_sA += weights_s0 * delayLine_s9;
    (*filteredData)[station][pol][time + 10][channel][ri] = sum_sA;

    sum_sB = weights_sF * delayLine_sB;
    delayLine_sB = (*sampledData)[station][time + NR_TAPS - 1 + 12][channel][pol_ri];
    sum_sB += weights_sE * delayLine_sC;
    sum_sB += weights_sD * delayLine_sD;
    sum_sB += weights_sC * delayLine_sE;
    sum_sB += weights_sB * delayLine_sF;
    sum_sB += weights_sA * delayLine_s0;
    sum_sB += weights_s9 * delayLine_s1;
    sum_sB += weights_s8 * delayLine_s2;
    sum_sB += weights_s7 * delayLine_s3;
    sum_sB += weights_s6 * delayLine_s4;
    sum_sB += weights_s5 * delayLine_s5;
    sum_sB += weights_s4 * delayLine_s6;
    sum_sB += weights_s3 * delayLine_s7;
    sum_sB += weights_s2 * delayLine_s8;
    sum_sB += weights_s1 * delayLine_s9;
    sum_sB += weights_s0 * delayLine_sA;
    (*filteredData)[station][pol][time + 11][channel][ri] = sum_sB;

    sum_sC = weights_sF * delayLine_sC;
    delayLine_sC = (*sampledData)[station][time + NR_TAPS - 1 + 13][channel][pol_ri];
    sum_sC += weights_sE * delayLine_sD;
    sum_sC += weights_sD * delayLine_sE;
    sum_sC += weights_sC * delayLine_sF;
    sum_sC += weights_sB * delayLine_s0;
    sum_sC += weights_sA * delayLine_s1;
    sum_sC += weights_s9 * delayLine_s2;
    sum_sC += weights_s8 * delayLine_s3;
    sum_sC += weights_s7 * delayLine_s4;
    sum_sC += weights_s6 * delayLine_s5;
    sum_sC += weights_s5 * delayLine_s6;
    sum_sC += weights_s4 * delayLine_s7;
    sum_sC += weights_s3 * delayLine_s8;
    sum_sC += weights_s2 * delayLine_s9;
    sum_sC += weights_s1 * delayLine_sA;
    sum_sC += weights_s0 * delayLine_sB;
    (*filteredData)[station][pol][time + 12][channel][ri] = sum_sC;

    sum_sD = weights_sF * delayLine_sD;
    delayLine_sD = (*sampledData)[station][time + NR_TAPS - 1 + 14][channel][pol_ri];
    sum_sD += weights_sE * delayLine_sE;
    sum_sD += weights_sD * delayLine_sF;
    sum_sD += weights_sC * delayLine_s0;
    sum_sD += weights_sB * delayLine_s1;
    sum_sD += weights_sA * delayLine_s2;
    sum_sD += weights_s9 * delayLine_s3;
    sum_sD += weights_s8 * delayLine_s4;
    sum_sD += weights_s7 * delayLine_s5;
    sum_sD += weights_s6 * delayLine_s6;
    sum_sD += weights_s5 * delayLine_s7;
    sum_sD += weights_s4 * delayLine_s8;
    sum_sD += weights_s3 * delayLine_s9;
    sum_sD += weights_s2 * delayLine_sA;
    sum_sD += weights_s1 * delayLine_sB;
    sum_sD += weights_s0 * delayLine_sC;
    (*filteredData)[station][pol][time + 13][channel][ri] = sum_sD;

    sum_sE = weights_sF * delayLine_sE;
    delayLine_sE = (*sampledData)[station][time + NR_TAPS - 1 + 15][channel][pol_ri];
    sum_sE += weights_sE * delayLine_sF;
    sum_sE += weights_sD * delayLine_s0;
    sum_sE += weights_sC * delayLine_s1;
    sum_sE += weights_sB * delayLine_s2;
    sum_sE += weights_sA * delayLine_s3;
    sum_sE += weights_s9 * delayLine_s4;
    sum_sE += weights_s8 * delayLine_s5;
    sum_sE += weights_s7 * delayLine_s6;
    sum_sE += weights_s6 * delayLine_s7;
    sum_sE += weights_s5 * delayLine_s8;
    sum_sE += weights_s4 * delayLine_s9;
    sum_sE += weights_s3 * delayLine_sA;
    sum_sE += weights_s2 * delayLine_sB;
    sum_sE += weights_s1 * delayLine_sC;
    sum_sE += weights_s0 * delayLine_sD;
    (*filteredData)[station][pol][time + 14][channel][ri] = sum_sE;

    sum_sF = weights_sF * delayLine_sF;
    sum_sF += weights_sE * delayLine_s0;
    sum_sF += weights_sD * delayLine_s1;
    sum_sF += weights_sC * delayLine_s2;
    sum_sF += weights_sB * delayLine_s3;
    sum_sF += weights_sA * delayLine_s4;
    sum_sF += weights_s9 * delayLine_s5;
    sum_sF += weights_s8 * delayLine_s6;
    sum_sF += weights_s7 * delayLine_s7;
    sum_sF += weights_s6 * delayLine_s8;
    sum_sF += weights_s5 * delayLine_s9;
    sum_sF += weights_s4 * delayLine_sA;
    sum_sF += weights_s3 * delayLine_sB;
    sum_sF += weights_s2 * delayLine_sC;
    sum_sF += weights_s1 * delayLine_sD;
    sum_sF += weights_s0 * delayLine_sE;
    (*filteredData)[station][pol][time + 15][channel][ri] = sum_sF;
  }
}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // cudaDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = cudaDeviceReset();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FIR_filter_wrapper(float *DevFilteredData,
    float const *DevSampledData,
    float const *DevWeightsData)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        //fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;;
    }

    // From here copy pasta of opencl code
    int nrChannelsPerSubband = 8;
    int nrStations = 2; 
    unsigned totalNrThreads = nrChannelsPerSubband * NR_POLARIZATIONS * 2; //ps.nrChannelsPerSubband()
    dim3 globalWorkSize(totalNrThreads, nrStations); //ps.nrStations()

    int MAXNRCUDATHREADS = 512;
    size_t maxNrThreads = MAXNRCUDATHREADS;
    unsigned nrPasses = (totalNrThreads + maxNrThreads - 1) / maxNrThreads;
    dim3 localWorkSize(totalNrThreads / nrPasses, 1); 

    //// Create the needed data
    //unsigned sizeFilteredData = NR_STATIONS * NR_POLARIZATIONS * NR_SAMPLES_PER_CHANNEL * NR_CHANNELS * COMPLEX;
    //float* filteredData = new float[sizeFilteredData];
    //for (unsigned idx = 0; idx < sizeFilteredData; ++idx)
    //{
    //  filteredData[idx] = 0;
    //}
    //
    //unsigned sizeSampledData = NR_STATIONS * (NR_TAPS - 1 + NR_SAMPLES_PER_CHANNEL) * NR_CHANNELS * NR_POLARIZATIONS * COMPLEX;
    //float * sampledData = new float[sizeSampledData];
    //for (unsigned idx = 0; idx < sizeSampledData; ++idx)
    //{
    //  sampledData[idx] = 0;
    //}

    //unsigned sizeWeightsData = NR_CHANNELS * 16;
    //float * weightsData = new float[sizeWeightsData];
    //     for (unsigned idx = 0; idx < sizeWeightsData; ++idx)
    //{
    //  weightsData[idx] = 0;
    //}

    //copy to the gpu
    //float *DevFilteredData;
    //float *DevSampledData;
    //float *DevWeightsData;
    // Allocate GPU buffers for three vectors (two input, one output)    .
    //cudaStatus = cudaMalloc((void**)&DevFilteredData, sizeFilteredData * sizeof(float));
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = cudaMalloc((void**)&DevSampledData, sizeSampledData * sizeof(float));
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMalloc failed!");
    //    goto Error;
    //}

    //cudaStatus = cudaMalloc((void**)&DevWeightsData, sizeWeightsData * sizeof(float));
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMalloc failed!");
    //    goto Error;
    //}    

    //// Copy input vectors from host memory to GPU buffers.
    //cudaStatus = cudaMemcpy(DevWeightsData, weightsData,
    //  sizeWeightsData * sizeof(float), cudaMemcpyHostToDevice);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMemcpy failed!");
    //    goto Error;
    //}

    //cudaStatus = cudaMemcpy(DevSampledData, sampledData,
    //  sizeSampledData * sizeof(float), cudaMemcpyHostToDevice);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMemcpy failed!");
    //    goto Error;
    //}


    // Launch a kernel on the GPU with one thread for each element.
    FIR_filter<<<globalWorkSize, localWorkSize>>>(DevFilteredData,
      DevSampledData, DevWeightsData);

    //// cudaDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = cudaDeviceSynchronize();
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    return cudaStatus;;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = cudaMemcpy(filteredData, DevFilteredData,
    //  sizeFilteredData * sizeof(float), cudaMemcpyDeviceToHost);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaMemcpy failed!");
    //    return cudaStatus;;
    //}

}
