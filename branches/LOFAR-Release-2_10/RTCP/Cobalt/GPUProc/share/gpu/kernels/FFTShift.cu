#include "hip/hip_runtime.h"
//# FFTShift.cu: multyply odd samples with -1 for correct fft functionality
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include "gpu_math.cuh"

typedef float2 *DataType;

/**
 * Shift the zero-frequency component to the center of the spectrum.
 * This kernel swaps the half-spaces of the channel dimension 
 * so that the negative frequencies are placed to
 * the left of the positive frequencies. We do this by modulating the samples
 * with exp(-j*pi), which results in a shift over pi in the frequency
 * domain. More information can be found in any decent book on digital signal
 * processing. 
 *
 * @param[data] a multi-dimensional array with time samples of type complex
 * float in the last dimension.
 * @pre @c The number of data samples must be a multiple of the maximum number
 * of threads per block, typically 1024.
 * @note We will squash the multi-dimensional array to one dimension for reasons
 * of flexibility, because the size of the other dimensions (usually @c[channel]
 * and @c[station]) can vary wildly.
 */

extern "C"
{
  __global__ void FFTShift(DataType data)
  {
    unsigned sample  = blockIdx.x * blockDim.x + threadIdx.x;

    // Multiplication factor: 1 for even samples, -1 for odd samples
    signed factor = 1 - 2 * (sample % 2); 
    data[sample] = data[sample] * factor;
  }
}
