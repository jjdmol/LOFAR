//# tcuda-runtime-api.cu: simple CUDA runtime API test
//# Copyright (C) 2014  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

// nvcc -o tcuda-runtime-api tcuda-runtime-api.cu

//#include <lofar_config.h>


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <vector>
#include <iostream>

using namespace std;

__global__ void kfunc(float* data) {
  float v = data[0];
  //float v = data[1024*1024*1024]; // intentionally out of bounds
  data[0] = v + 1.0f;
}

int main() {
  int rv = 0;
  hipError_t err;

  float *dptr;
  size_t len = 1024*1024;
  err = hipMalloc((void **)&dptr, len * sizeof(float));
  if (err != hipSuccess) { cerr << "hipMalloc failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }

  vector<float> hvec(32*len); // some extra space
  err = hipMemcpy(dptr, &hvec[0], len * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) { cerr << "hipMemcpy (H2D) failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }

  // launch bad kernel
  int block_dim = 1;
  int grid_dim = 1;
  kfunc<<<grid_dim, block_dim>>>(dptr);
  err = hipGetLastError();
  if (err != hipSuccess) { cerr << "kernel launch failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }
  err = hipDeviceSynchronize();
  if (err != hipSuccess) { cerr << "hipDeviceSynchronize failed (launch) (expected): " << hipGetErrorString(err) << " (" << err << ")" << endl; rv = 1; }
  if (err != hipSuccess) {
    err = hipGetLastError();
    if (err != hipSuccess) { cerr << "resetting last error, which was (expected): " << hipGetErrorString(err) << " (" << err << ")" << endl; rv = 1; }
    err = hipGetLastError();
    if (err != hipSuccess) { cerr << "reset failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; rv = 1; }
  } 

  err = hipMemcpy(&hvec[0], dptr, len * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess) { cerr << "hipMemcpy (D2H) failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; rv = 1; }

  // only check first 16 output vals
  if (hvec[0] != 1.0f) { cerr << "expected hvec[0] to be 1.0f, but got " << hvec[0] << endl; rv = 1; }
  for (int i = 1; i < 16; i++) {
    if (hvec[i] != 0.0f) { cerr << "expected hvec[" << i << "] to be 0.0f, but got " << hvec[i] << endl; rv = 1; }
  }

  err = hipFree(dptr);
  if (err != hipSuccess) { cerr << "hipFree failed: " << hipGetErrorString(err) << " (" << err << ")" << endl; exit(1); }

  if (rv == 0)
    cout << "Test passed" << endl;
  else
    cout << "Test failed" << endl;
  return rv;
}

