#include "hip/hip_runtime.h"
//# tcuda-driver-api.cu: simple CUDA driver API test
//# Copyright (C) 2014  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

// nvcc -ptx tcuda-driver-api.cu && nvcc -o tcuda-driver-api tcuda-driver-api.cu -lcuda

//#include <lofar_config.h>

#include <cstdlib>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>

#ifndef PTX_FILENAME
#define PTX_FILENAME tcuda-driver-api.ptx  // ease manual compilation
#endif

// Using automatic CMAKE/make compilation, we pass -DPTX_FILENAME=xxx.
// But xxx must be a C-string and nvcc 6.5 gobbles (shell escaped) double quotes in -D opts.
// Work around it by stringifying the result of macro expansion.
// This requires two levels of macros.
#define xstr(s) str(s)
#define str(s) #s

extern "C" {
__global__ void kfunc(float* data) {
  float v = data[0];
  //float v = data[1024*1024*1024]; // intentionally out of bounds
  data[0] = v + 1.0f;
}
}

using namespace std;

int main() {
  int rv = 0;
  hipError_t r;
  r = hipInit(0);
  if (r != hipSuccess) { cerr << "hipInit failed: " << r << endl; exit(1); }

  hipDevice_t dev;
  int ordinal = 0;
  r = hipDeviceGet(&dev, ordinal);
  if (r != hipSuccess) { cerr << "hipDeviceGet failed: " << r << endl; exit(1); }

  hipCtx_t ctx;
  unsigned int flags = hipDeviceScheduleAuto;
  r = hipCtxCreate(&ctx, flags, dev);
  if (r != hipSuccess) { cerr << "hipCtxCreate failed: " << r << endl; exit(1); }

  hipDeviceptr_t dptr;
  size_t len = 1024*1024;
  r = hipMalloc(&dptr, len * sizeof(float));
  if (r != hipSuccess) { cerr << "hipMalloc failed:" << r << endl; rv = 1; }

/*
  hipDeviceptr_t dptr2;
  len = 1024ULL*1024*1024*1024; // too large
  r = hipMalloc(&dptr2, len * sizeof(float));
  if (r != hipSuccess) { cerr << "hipMalloc failed (2) (expected): " << r << endl; rv = 1; }
*/

  hipDeviceptr_t dptr3;
  len = 1024*1024; // works again after previous erroneous alloc (if enabled at all)
  r = hipMalloc(&dptr3, len * sizeof(float));
  if (r != hipSuccess) { cerr << "hipMalloc failed (3): " << r << endl; rv = 1; }

  vector<float> hvec(32*len); // some extra space

  hipStream_t stream;
  r = hipStreamCreateWithFlags(&stream, 0);
  if (r != hipSuccess) { cerr << "hipStreamCreateWithFlags failed: " << r << endl; exit(1); }

  r = hipMemcpyHtoDAsync(dptr, &hvec[0], len * sizeof(float), stream);
  //r = hipMemcpyHtoDAsync(dptr, &hvec[0], 32*len * sizeof(float), stream); // GPU buffer overflow
  if (r != hipSuccess) { cerr << "hipMemcpyHtoDAsync failed: " << r << endl; rv = 1; }

  r = hipStreamSynchronize(stream);
  if (r != hipSuccess) { cerr << "hipStreamSynchronize failed (HtoD): " << r << endl; rv = 1; }

  cout << "Trying to load module file " << xstr(PTX_FILENAME) << endl;
  hipModule_t kmodule;
  r = hipModuleLoad(&kmodule, xstr(PTX_FILENAME)); // should have been precompiled externally
  if (r != hipSuccess) { cerr << "hipModuleLoad failed: " << r << endl; exit(1); }
  cout << "Module load succeeded" << endl;

  hipFunction_t kfunc;
  r = hipModuleGetFunction(&kfunc, kmodule, "kfunc");
  if (r != hipSuccess) { cerr << "hipModuleGetFunction failed: " << r << endl; exit(1); }

  // async launch bad kernel
  void *args = &dptr;
  r = hipModuleLaunchKernel(kfunc, /*gridDim: */1, 1, 1, /*blockDim: */1, 1, 1,
                     /*dynShmemBytes: */0, stream, &args, NULL);
  if (r != hipSuccess) { cerr << "hipModuleLaunchKernel failed: " << r << endl; exit(1); }

  r = hipStreamSynchronize(stream);
  if (r != hipSuccess) { cerr << "hipStreamSynchronize failed (launch) (expected): " << r << endl; rv = 1; }

  r = hipMemcpyDtoHAsync(&hvec[0], dptr, len * sizeof(float), stream);
  if (r != hipSuccess) { cerr << "hipMemcpyDtoHAsync failed: " << r << endl; rv = 1; }

  r = hipStreamSynchronize(stream);
  if (r != hipSuccess) { cerr << "hipStreamSynchronize failed (DtoH): " << r << endl; rv = 1; }

  // only check first 16 output vals
  if (hvec[0] != 1.0f) { cerr << "expected hvec[0] to be 1.0f, but got " << hvec[0] << endl; rv = 1; }
  for (int i = 1; i < 16; i++) {
    if (hvec[i] != 0.0f) { cerr << "expected hvec[" << i << "] to be 0.0f, but got " << hvec[i] << endl; rv = 1; }
  }

  r = hipModuleUnload(kmodule);
  if (r != hipSuccess) { cerr << "hipModuleUnload failed: " << r << endl; exit(1); }
  r = hipStreamDestroy(stream);
  if (r != hipSuccess) { cerr << "hipStreamDestroy failed: " << r << endl; exit(1); }
  r = hipFree(dptr3);
  if (r != hipSuccess) { cerr << "hipFree dptr3 failed: " << r << endl; exit(1); }
  r = hipFree(dptr);
  if (r != hipSuccess) { cerr << "hipFree dptr failed: " << r << endl; exit(1); }
  r = hipCtxDestroy(ctx);
  if (r != hipSuccess) { cerr << "hipCtxDestroy failed: " << r << endl; exit(1); }

  if (rv == 0)
    cout << "Test passed" << endl;
  else
    cout << "Test failed" << endl;
  return rv;
}

