#include "hip/hip_runtime.h"
//# IntToFloat.cl
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#if NR_BITS_PER_SAMPLE == 16
typedef short2 SampleType;
__device__ float convertIntToFloat(short x)
{
	return x;
}
#elif NR_BITS_PER_SAMPLE == 8
__device__ float convertIntToFloat(char x)
{
	return x==-128 ? -127 : x;
}
typedef char2 SampleType;
#else
#error unsupport NR_BITS_PER_SAMPLE
#endif

typedef  SampleType (*SampledDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
typedef  float2 (*ConvertedDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_SUBBAND];



extern "C" {
 __global__ void intToFloat( void * convertedDataPtr,
                          const void * sampledDataPtr)
{
  ConvertedDataType convertedData = (ConvertedDataType) convertedDataPtr;
  SampledDataType sampledData = (SampledDataType) sampledDataPtr;

  uint station = blockIdx.y * blockDim.y + threadIdx.y;
  for (uint time = threadIdx.x; time < NR_SAMPLES_PER_SUBBAND; time += blockDim.x) {
    (*convertedData)[station][0][time] = make_float2(
			convertIntToFloat((*sampledData)[station][time][0].x),
            convertIntToFloat((*sampledData)[station][time][0].y));
    (*convertedData)[station][1][time] = make_float2(
			convertIntToFloat((*sampledData)[station][time][1].x), 
            convertIntToFloat((*sampledData)[station][time][1].y));
    
  }
}
}

