#include "hip/hip_runtime.h"
//# CoherentStokes.cu: Calculate the Stokes parameters
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id: CoherentStokes.cu 24553 2013-04-09 14:21:56Z mol $

#if NR_SAMPLES_PER_CHANNEL % INTEGRATION_SIZE != 0 
  #error  unsupported INTEGRATION_SIZE for NR_SAMPLES_PER_CHANNEL
#elif NR_COHERENT_STOKES != 1
  #if NR_COHERENT_STOKES != 4
     #error  unsupported NR_COHERENT_STOKES
  #endif
#elif NR_CHANNELS % 16 != 0
  #if NR_CHANNELS != 1
    #error unsupported NR_CHANNELS
  #endif
#endif

//4D output array of stokes values. Each sample contains 1 or 4 stokes paramters. For each tab, there are NR_STOKES timeseries of channels 
typedef float2 (*inputDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS]; 

//4D input array of complex samples. For each tab and polarization there are timelines with data for each channel
typedef float (*outputDataType)[NR_TABS][NR_COHERENT_STOKES][NR_SAMPLES_PER_CHANNEL/INTEGRATION_SIZE][NR_CHANNELS];

/*!
 * Computes the first or all 4 stokes parameters.
 * http://www.astron.nl/~romein/papers/EuroPar-11/EuroPar-11.pdf 
 * \code
 * I = X * conj(X) + Y * conj(Y)
 * Q = X * conj(X) - Y * conj(Y)
 * U = 2 * real(X * con(Y))
 * V = 2 * imag(X * con(Y))
 * \endcode
 * This reduces to (validated on paper by Wouter and John):
 * \code
 * Px = real(X) * real(X) + imag(X) * imag(X)
 * Py = real(Y) * real(Y) + imag(Y) * imag(Y)
 * I = Px + Py
 * Q = Px - Py
 * U = 2 * (real(X) * real(Y) + imag(X) * imag(Y))
 * V = 2 * (imag(X) * real(Y) - real(X) * imag(Y))
 * \endcode
 * 
 * The kernel's first parallel dimension is on the channels; the second dimension is in time; the third on the tabs.
 * The thread block size based on these factors could be larger then the hardmare max.
 * Therefore<tt> NR_CHANNELS * NR_TABS * TIME_PARALLEL_FACTOR </tt>should not exceed the hardware maximum of threads (1024 on a K10).
 *
 * \param[out] outputPtr         4D output array of stokes values. Each sample contains 1 or 4 stokes paramters. For each tab, there are NR_STOKES timeseries of channels 
 *                               The dimensions are: NR_TABS by NR_COHERENT_STOKES by (NR_SAMPLES_PER_CHANNEL/INTEGRATION_SIZE)  by NR_CHANNELS.
 * \param[in]  inputPtr          4D input array of complex samples. For each tab and polarization there are timelines with data for each channel
 *                               The dimensions are: NR_TABS by NR_POLARIZATIONS by NR_SAMPLES_PER_CHANNEL by NR_CHANNELS
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | multiple of INTEGRATION_SIZE | number of input samples per channel
 * NR_CHANNELS             | 1 or 16                 | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of tabs to create
 * NR_COHERENT_STOKES      | 1 or 4                  | number of stokes paramters to create
 * INTEGRATION_SIZE        | >= 1                    | amount of samples to integrate to a single output sample
 * TIME_PARALLEL_FACTOR    | >= 1                    | amount of parallel threads to work on a full timerange
 *
 * Note that this kernel assumes and needs NR_POLARIZATIONS == 2 and COMPLEX == 2
 * 
 * The TIME_PARALLEL_FACTOR splits the time range in a number of portions which get worked on by 
 * seperate threads (in parallel).
 *  
 * Execution configuration:
 * - LocalWorkSize = 3 dimensional; (NR_CHANNELS, TIME_PARALLEL_FACTOR, NR_TABS)
 *                   The product of the three should not be larger then max thread size.
 *                   The max thread size depends on the hardware used. 512 For odl hardware. K10 and higher have 1024 threads 
 * - GlobalWorkSize = 3 dimensional; depends on the size of NR_TABS, NR_CHANNELS and the max thread size
 *                   Ideally the work fits in a single block. If not the remainder could
 *                   be computed with a second (differently sized) block
 */
extern "C" __global__ void coherentStokes(void *outputPtr, const void *inputPtr) 
{
  inputDataType input = (inputDataType) inputPtr;
  outputDataType output = (outputDataType) outputPtr;

  // Define the indexes in the data depending on the block and thread idx
  unsigned channel_idx = threadIdx.x;  // If we have channels do the read and write with 16 in parallel
  unsigned time_idx = threadIdx.y;     
  unsigned tab_idx = threadIdx.z;    

  // Step over (part of) the timerange of samples with INTEGRATION_SIZE steps
  // The time_idx determines which part of (or the whole of) the timerange this thread is working on:
  // Work from the start of the time frame (pending your threadIdx.y) untill the next timeframe
  // Step within this timerange with integration size steps. These substeps are done in the inner loop
  for (unsigned idx_stride = time_idx * (NR_SAMPLES_PER_CHANNEL / TIME_PARALLEL_FACTOR) ; 
                   idx_stride < (time_idx + 1) * (NR_SAMPLES_PER_CHANNEL / TIME_PARALLEL_FACTOR)
                && idx_stride < NR_SAMPLES_PER_CHANNEL;
                idx_stride += INTEGRATION_SIZE)
  {
    // We are integrating all values in the current stride
    // local variable
    float stokesI = 0;
#   if NR_COHERENT_STOKES == 4
    float stokesQ = 0;
    float halfStokesU = 0;
    float halfStokesV = 0;
#   endif

    // Do the integration
    for (unsigned idx_step = 0; idx_step < INTEGRATION_SIZE; idx_step++) 
    {
      float2 X = (*input)[tab_idx][0][idx_stride + idx_step][channel_idx];    
      float2 Y = (*input)[tab_idx][1][idx_stride + idx_step][channel_idx];

      // Calculate the partial solutions
      float powerX = X.x * X.x + X.y * X.y;
      float powerY = Y.x * Y.x + Y.y * Y.y;
      stokesI += powerX + powerY;
#     if NR_COHERENT_STOKES == 4
      stokesQ += powerX - powerY;
      halfStokesU += X.x * Y.x + X.y * Y.y;
      halfStokesV += X.y * Y.x - X.x * Y.y;
#     endif
    }
    // We step in the data with INTEGRATION_SIZE
    unsigned write_idx = idx_stride / INTEGRATION_SIZE;

    (*output)[tab_idx][0][write_idx][channel_idx] = stokesI;
#   if NR_COHERENT_STOKES == 4
    (*output)[tab_idx][1][write_idx][channel_idx] = stokesQ;
    (*output)[tab_idx][2][write_idx][channel_idx] = 2 * halfStokesU;
    (*output)[tab_idx][3][write_idx][channel_idx] = 2 * halfStokesV;
#   endif  
    // No baries needed. All computations are fully parallel
  }
}
