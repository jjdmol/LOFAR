#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <complex>
#include "complex.h"

using namespace std;

hipError_t addWithCuda(int *c, std::complex<float> * output_complex, const int *a, const int *b, const std::complex<float> * input_complex, size_t size);

__global__ void addKernel(void *c_ptr, void *in_ptr, const void *a_ptr, const void *b_ptr, const void *out_ptr)
{
    int i = threadIdx.x;
    int *c = (int *) c_ptr;
    int *a = (int *) a_ptr;
    int *b = (int *) b_ptr;

    LOFAR::Cobalt::gpu::complex<float>*in = (LOFAR::Cobalt::gpu::complex<float>*) in_ptr;
    LOFAR::Cobalt::gpu::complex<float>*out = (LOFAR::Cobalt::gpu::complex<float>*) out_ptr;

    c[i] = a[i] + b[i];
    out[i] = in[i] + in[i];
    out[i] -= in[i];
    out[i] /=- in[i];
    out[i] = out[i];
    out[i] *= 10.0;

}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    const complex<float> complex_in[5] = { complex<float>(1.0,1.0),
                                                complex<float>(1,-1),
                                                complex<float>(-1,1),
                                                complex<float>(-1,-1),
                                                complex<float>(4,-4)};
    complex<float> complex_out[5] = { 0 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c,complex_out, a, b, complex_in,arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
    
    cout << "complex numbers: {";
    for (int idx =0; idx < 5 ;++idx)
      cout << complex_out[idx] << ", ";
    cout << " }" << endl;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c,
                        std::complex<float>* output_complex,
                        const int *a, 
                        const int *b,
                        const std::complex<float>* input_complex,
                        size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    std::complex<float> *dev_in = 0;
    std::complex<float> *dev_out = 0;
    hipError_t cudaStatus;


    cout << input_complex[0] << endl;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // allocate the complex buffers
    cudaStatus = hipMalloc((void**)&dev_in, size * sizeof(std::complex<float>));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_out, size * sizeof(std::complex<float>));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_in, input_complex, size * sizeof(std::complex<float>), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>((void *)dev_c,(void *) dev_in, (const void *)dev_a, (const void *)dev_b, (const void *) dev_out);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(output_complex, dev_out, size * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_in);
    hipFree(dev_out);
    
    return cudaStatus;
}
