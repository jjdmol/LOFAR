#include "hip/hip_runtime.h"
#include <GPUProc/complex.h>
#include <GPUProc/cuda/CUDAException.h>
#include <iostream>

using namespace LOFAR::Cobalt;

__global__ void kernel()
{
  /* gpu::complex<float> cf(3.14f, 2.72f); */
  /* cf *= 3.0f; */
  /* gpu::complex<double> cd(1.618, 0.577); */
  /* cd *= 3.0; */
  /* gpu::complex<long double> cl(1.41421356237L, 1.73205080757L); */
  /* cl *= 3.0L; */
  /* std::cout << cf << std::endl; */
  /* std::cout << cd << std::endl; */
  /* std::cout << cl << std::endl; */
  /* cd *= cf; */
  /* cl *= cd; */
  /* cf *= cl; */
  /* std::cout << cf << std::endl; */
  /* std::cout << cd << std::endl; */
  /* std::cout << cl << std::endl; */
}

using namespace std;

int main()
{
  std::cout << "Executing kernel" << std::endl;
  kernel<<<1,1>>>();
  CUDA_CALL(hipGetLastError());
  gpu::complex<float> cf(3.14f, 2.72f);
  cf *= 3.0f;
  gpu::complex<double> cd(1.618, 0.577);
  cd *= 3.0;
  gpu::complex<long double> cl(1.41421356237L, 1.73205080757L);
  cl *= 3.0L;
  std::cout << cf << std::endl;
  std::cout << cd << std::endl;
  std::cout << cl << std::endl;
  cd *= cf;
  cl *= cd;
  cf *= cl;
  std::cout << cf << std::endl;
  std::cout << cd << std::endl;
  std::cout << cl << std::endl;
  std::cout << "Done" << std::endl;
  return 0;
}
