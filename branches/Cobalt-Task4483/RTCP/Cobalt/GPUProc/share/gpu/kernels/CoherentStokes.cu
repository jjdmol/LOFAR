#include "hip/hip_runtime.h"
//# CoherentStokes.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id: CoherentStokes.cu 24553 2013-04-09 14:21:56Z mol $

#include "IntToFloat.cuh"

typedef float2 (*inputDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS]; 
typedef float2 (*outputDataType)[NR_TABS][NR_STOKES][NR_SAMPLES_PER_CHANNEL/STOKES_INTEGRATION_SAMPLES][NR_CHANNELS];

extern "C" {

/*!
 * Computes correlations between all pairs of stations (baselines) and X,Y
 * polarizations. Also computes all station (and pol) auto-correlations.
 # blabla bla
 * \param[out] visibilitiesPtr         2D output array of visibilities. Each visibility contains the 4 polarization pairs, XX, XY, YX, YY, each of complex float type.
 * \param[in]  correctedDataPtr        3D input array of samples. Each sample contains the 2 polarizations X, Y, each of complex float type.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | multiple of BLOCK_SIZE  | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * Note that for > 1 channels, NR_CHANNELS-1 channels are actually processed,
 * because the second PPF has "corrupted" channel 0. (An inverse PPF can disambiguate.) \n
 * Note that if NR_CHANNELS is low (esp. 1), these kernels perform poorly.
 * Note that this kernel assumes (but does not use) NR_POLARIZATIONS == 2.
 *
 * Execution configuration:
 */

__global__ void coherentStokes(void *inputPtr, const void *outputPtr) 
{
  inputDataType input = (inputDataType) inputPtr;
  outputDataType output = (outputDataType) correctedDataPtr;

  // Define the indexes in the data depending on the block and thread idx
  unsigned tab_idx = 0;
  unsigned channel_idx = 0;

  // Step over the complete time line with COHERENT_STOKES_TIME_INTEGRATION_FACTOR steps
  for (unsigned idx_stride = 0; idx_stride < NR_SAMPLES_PER_CHANNEL / COHERENT_STOKES_TIME_INTEGRATION_FACTOR; idx_stride++)
  {
    // We are integrating all values in the current stride, we need local variable to store
    float stokesI = 0;

#   if NR_COHERENT_STOKES == 4
    float stokesQ = 0;
    float halfStokesU = 0;
    float halfStokesV = 0;
#   endif

    for (unsigned idx_step = 0; idx_step < COHERENT_STOKES_TIME_INTEGRATION_FACTOR; idx_step++) 
    {
      float4 sample = (*inputDataType)[tab_idx][idx_stride][idx_step][channel_idx];
      float2 X = make_float2(sample.x, sample.y);
      float2 Y = sample.zw;
      float powerX = X.x * X.x + X.y * X.y;
      float powerY = Y.x * Y.x + Y.y * Y.y;
      stokesI += powerX + powerY;

#     if NR_COHERENT_STOKES == 4
      stokesQ += powerX - powerY;
      halfStokesU += X.x * Y.x + X.y * Y.y;
      halfStokesV += X.y * Y.x - X.x * Y.y;
#     endif*/
    }

    (*output)[tab_idx][0][idx_stride][channel_idx] = stokesI;
#   if NR_COHERENT_STOKES == 4
    (*output)[tab_idx][0][idx_stride][channel_idx] = stokesQ;
    (*output)[tab_idx][0][idx_stride][channel_idx] = 2 * halfStokesU;
    (*output)[tab_idx][0][idx_stride][channel_idx] = 2 * halfStokesV;
#   endif
  }
}
