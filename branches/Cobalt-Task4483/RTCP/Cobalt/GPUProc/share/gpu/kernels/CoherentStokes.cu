#include "hip/hip_runtime.h"
//# CoherentStokes.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id: CoherentStokes.cu 24553 2013-04-09 14:21:56Z mol $

typedef float2 (*inputDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS]; 
typedef float (*outputDataType)[NR_TABS][NR_COHERENT_STOKES][NR_SAMPLES_PER_CHANNEL/INTEGRATION_SIZE][NR_CHANNELS];

/*!
 * Computes correlations between all pairs of stations (baselines) and X,Y
 * polarizations. Also computes all station (and pol) auto-correlations.
 # blabla bla
 * \param[out] visibilitiesPtr         2D output array of visibilities. Each visibility contains the 4 polarization pairs, XX, XY, YX, YY, each of complex float type.
 * \param[in]  correctedDataPtr        3D input array of samples. Each sample contains the 2 polarizations X, Y, each of complex float type.
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | multiple of BLOCK_SIZE  | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * Note that for > 1 channels, NR_CHANNELS-1 channels are actually processed,
 * because the second PPF has "corrupted" channel 0. (An inverse PPF can disambiguate.) \n
 * Note that if NR_CHANNELS is low (esp. 1), these kernels perform poorly.
 * Note that this kernel assumes (but does not use) NR_POLARIZATIONS == 2.
 *
 * Execution configuration:
 */
extern "C" __global__ void coherentStokes(void *outputPtr, const void *inputPtr) 
{
  inputDataType input = (inputDataType) inputPtr;
  outputDataType output = (outputDataType) outputPtr;

  // Define the indexes in the data depending on the block and thread idx
  unsigned channel_idx = threadIdx.x;  // The channels are next to each other in mem 
  unsigned time_idx = threadIdx.y;     //
  unsigned tab_idx = threadIdx.z;   // 

  // Step over the complete time line with INTEGRATION_SIZE steps
  for (unsigned idx_stride = time_idx * (NR_SAMPLES_PER_CHANNEL / TIME_PARALLEL_FACTOR) ; 
                idx_stride < (time_idx + 1) * (NR_SAMPLES_PER_CHANNEL / TIME_PARALLEL_FACTOR);
                idx_stride += INTEGRATION_SIZE)
  {
    // We are integrating all values in the current stride, we need local variable to store
    float stokesI = 0;

#   if NR_COHERENT_STOKES == 4
    float stokesQ = 0;
    float halfStokesU = 0;
    float halfStokesV = 0;
#   endif

    for (unsigned idx_step = 0; idx_step < INTEGRATION_SIZE; idx_step++) 
    {
        //[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS]
      float2 X = (*input)[tab_idx][0][idx_stride + idx_step][channel_idx];    
      float2 Y = (*input)[tab_idx][1][idx_stride + idx_step][channel_idx];

      float powerX = X.x * X.x + X.y * X.y;
      float powerY = Y.x * Y.x + Y.y * Y.y;
      stokesI += powerX + powerY;

#     if NR_COHERENT_STOKES == 4
      stokesQ += powerX - powerY;
      halfStokesU += X.x * Y.x + X.y * Y.y;
      halfStokesV += X.y * Y.x - X.x * Y.y;
#     endif
    }
    unsigned write_idx = idx_stride / INTEGRATION_SIZE;
    (*output)[tab_idx][0][write_idx][channel_idx] = stokesI;

#   if NR_COHERENT_STOKES == 4
    (*output)[tab_idx][1][write_idx][channel_idx] = stokesQ;
    (*output)[tab_idx][2][write_idx][channel_idx] = 2 * halfStokesU;
    (*output)[tab_idx][3][write_idx][channel_idx] = 2 * halfStokesV;
#   endif

    

  }
}
