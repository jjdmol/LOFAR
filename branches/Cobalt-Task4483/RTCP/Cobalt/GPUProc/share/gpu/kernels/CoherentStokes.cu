#include "hip/hip_runtime.h"
//# CoherentStokes.cu: Calculate the Stokes parameters
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id: CoherentStokes.cu 24553 2013-04-09 14:21:56Z mol $

/*!
 * Computes the first or all 4 stokes parameters.
 * http://www.astron.nl/~romein/papers/EuroPar-11/EuroPar-11.pdf 
 *  
 * I = X *  con(X) + Y * con(Y)
 * Q = X *  con(X) - Y * con(Y)
 * U = 2 * RE(X * con(Y))
 * V = 2 * IM(X * con(Y))
 * This reduces to (validate on paper by Wouter and John):
 * PX = RE(X) * RE(X) + IM(X) * IM(X)
 * PY = RE(Y) * RE(Y) + IM(Y) * IM(Y)
 * I = PX + PY
 * Q = PX - PY
 * U = 2 * (RE(X) * RE(Y) + IM(X) * IM(Y))
 * V = 2 * (IM(X) * RE(Y) - RE(X) * IM(Y))
 * 
 * The kernel's first parallel dimension is on the channels
 * Then second dimension is in time. Then on the tabs.
 * The thread block size based on these factors could be larger then the hardmare max.
 * Therefore NR_CHANNELS * NR_TABS * TIME_PARALLEL_FACTOR should no be more than the hardware maximum of threads (K10 == 1024)
 *
 * \param[out] outputPtr         4D output array of stokes values. Each sample contains 1 or 4 stokes paramters. For each tab, there are NR_STOKES timeseries of channels 
 * \param[in]  correctedDataPtr  4D input array of complex samples. For each tab and polarization the are timelines with data for each channel
 *
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | multiple of BLOCK_SIZE  | number of input samples per channel
 * NR_CHANNELS             | 1 or 16                 | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of tabs to create
 * NR_COHERENT_STOKES      | 1 or 4                  | number of stokes paramters to create
 * INTEGRATION_SIZE        | >= 1                    | amount of samples to integrate to a single output sample
 * TIME_PARALLEL_FACTOR    | >= 1                    | amount of paralel threads to work on a full timerange
 * 
 * Note that this kernel assumes and needs NR_POLARIZATIONS == 2 and COMPLEX == 2
 * 
 * Execution configuration:
 * - LocalWorkSize = 3 dimensional; (NR_CHANNELS, TIME_PARALLEL_FACTOR, NR_TABS)
 *                   The product of the three should not be larger then max thread size
 * - GlobalWorkSize = 3 dimensional; depends on the size of NR_TABS, NR_CHANNELS and the max thread size
 *                   Ideally the work fist in a single block. If not the remainder could
 *                   be computed with a second (differently sized) block
 */

typedef float2 (*inputDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS]; 
typedef float (*outputDataType)[NR_TABS][NR_COHERENT_STOKES][NR_SAMPLES_PER_CHANNEL/INTEGRATION_SIZE][NR_CHANNELS];

extern "C" __global__ void coherentStokes(void *outputPtr, const void *inputPtr) 
{
  inputDataType input = (inputDataType) inputPtr;
  outputDataType output = (outputDataType) outputPtr;

  // Define the indexes in the data depending on the block and thread idx
  unsigned channel_idx = threadIdx.x;  // If we have channels do the read and write with 16 in parallel
  unsigned time_idx = threadIdx.y;     
  unsigned tab_idx = threadIdx.z;    

  // Step over the complete time line with INTEGRATION_SIZE steps
  // Do this for for the correct timeframe (time_idx) based on the threadIdx.y
  for (unsigned idx_stride = time_idx * (NR_SAMPLES_PER_CHANNEL / TIME_PARALLEL_FACTOR) ; 
                idx_stride < (time_idx + 1) * (NR_SAMPLES_PER_CHANNEL / TIME_PARALLEL_FACTOR);
                idx_stride += INTEGRATION_SIZE)
  {
    // We are integrating all values in the current stride
    // local variable
    float stokesI = 0;
#   if NR_COHERENT_STOKES == 4
    float stokesQ = 0;
    float halfStokesU = 0;
    float halfStokesV = 0;
#   endif

    // Do the integration
    for (unsigned idx_step = 0; idx_step < INTEGRATION_SIZE; idx_step++) 
    {
      float2 X = (*input)[tab_idx][0][idx_stride + idx_step][channel_idx];    
      float2 Y = (*input)[tab_idx][1][idx_stride + idx_step][channel_idx];

      // Calculate the partial solutions
      float powerX = X.x * X.x + X.y * X.y;
      float powerY = Y.x * Y.x + Y.y * Y.y;
      stokesI += powerX + powerY;
#     if NR_COHERENT_STOKES == 4
      stokesQ += powerX - powerY;
      halfStokesU += X.x * Y.x + X.y * Y.y;
      halfStokesV += X.y * Y.x - X.x * Y.y;
#     endif
    }
    // We step in the data with INTEGRATION_SIZE
    unsigned write_idx = idx_stride / INTEGRATION_SIZE;

    (*output)[tab_idx][0][write_idx][channel_idx] = stokesI;
#   if NR_COHERENT_STOKES == 4
    (*output)[tab_idx][1][write_idx][channel_idx] = stokesQ;
    (*output)[tab_idx][2][write_idx][channel_idx] = 2 * halfStokesU;
    (*output)[tab_idx][3][write_idx][channel_idx] = 2 * halfStokesV;
#   endif  
    // No baries needed. All computations are fully parallel
  }
}
