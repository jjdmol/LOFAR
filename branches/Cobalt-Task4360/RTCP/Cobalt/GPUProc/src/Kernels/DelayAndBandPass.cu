#include "hip/hip_runtime.h"
//# DelayAndBandPass.cl
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/** @file
 * This file contains an Cuda implementation of the GPU kernel for the delay
 * and bandpass correction.
 *
 * Usually, this kernel will be run after the polyphase filter kernel FIR.cl. In
 * that case, the input data for this kernel is already in floating point format
 * (@c NR_CHANNELS > 1). However, if this kernel is the first in row, then the
 * input data is still in integer format (@c NR_CHANNELS == 1), and this kernel
 * needs to do the integer-to-float conversion.
 *
 * @attention The following pre-processor variables must be supplied when
 * compiling this program. Please take the pre-conditions for these variables
 * into account:
 * - @c NR_CHANNELS: 1 or a multiple of 16
 * - if @c NR_CHANNELS == 1 (input data is in integer format):
 *   - @c NR_BITS_PER_SAMPLE: 8 or 16
 *   - @c NR_SAMPLES_PER_SUBBAND: a multiple of 16
 * - if @c NR_CHANNELS > 1 (input data is in floating point format):
 *   - @c NR_SAMPLES_PER_CHANNEL: a multiple of 16
 * - @c NR_POLARIZATIONS: 2
 * - @c SUBBAND_WIDTH: a multiple of @c NR_CHANNELS
 */

//nvcc /home/wklijn/sources/4360/LOFAR/RTCP/Cobalt/GPUProc/src/Kernels/DelayAndBandPass.cu -I /home/wklijn/sources/4360/LOFAR/RTCP/Cobalt/GPUProc/src --ptx --gpu-architecture compute_30 --use_fast_math

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cuda/complex.h>

#define NR_CHANNELS 10
#define NR_STATIONS 10
#define NR_SAMPLES_PER_CHANNEL 1024
#define NR_SAMPLES_PER_SUBBAND 128
#define NR_BITS_PER_SAMPLE 16
#define NR_POLARIZATIONS 2
#define NR_BEAMS 8
#define USE_CUDA 1
#define COMPLEX 2
#define SUBBAND_BANDWIDTH 4
#define BANDPASS_CORRECTION 1
#define DELAY_COMPENSATION 1

#if NR_CHANNELS == 1
#undef BANDPASS_CORRECTION
#endif


typedef LOFAR::Cobalt::gpu::complex<float> complexfloat;
typedef LOFAR::Cobalt::gpu::complex<short> complexshort;
typedef LOFAR::Cobalt::gpu::complex<char> complexchar;
typedef  complexfloat (* OutputDataType)[NR_STATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_POLARIZATIONS];
#if NR_CHANNELS == 1
#if NR_BITS_PER_SAMPLE == 16
typedef  complexshort (* InputDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
#elif NR_BITS_PER_SAMPLE == 8
typedef  complexchar (* InputDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS];
#else
#error unsupport NR_BITS_PER_SAMPLE
#endif
#else
typedef  complexfloat (* InputDataType)[NR_STATIONS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS];
#endif
typedef  const complexfloat (* DelaysType)[NR_BEAMS][NR_STATIONS]; // 2 Polarizations; in seconds
typedef  const complexfloat (* PhaseOffsetsType)[NR_STATIONS]; // 2 Polarizations; in radians
typedef  const float (* BandPassFactorsType)[NR_CHANNELS];


/**
 * This kernel perfroms three operations on the input data:
 * - Apply a fine delay by doing a per channel phase correction.
 * - Apply a bandpass correction to compensate for the errors introduced by the
 *   polyphase filter that produced the subbands. This error is deterministic,
 *   hence it can be fully compensated for.
 * - Transpose the data so that the time slices for each channel are placed
 *   consecutively in memory.
 *
 * @param[out] correctedDataPtr    pointer to output data of ::OutputDataType,
 *                                 a 3D array [station][channel][sample]
 *                                 of ::fcomplex2 (2 complex polarizations)
 * @param[in]  filteredDataPtr     pointer to input data; this can either be a
 *                                 4D array [station][polarization][sample][channel]
 *                                 of ::fcomplex, or a 2D array [station][subband]
 *                                 of ::short_complex2 or ::char_complex2,
 *                                 depending on the value of @c NR_CHANNELS
 * @param[in]  subbandFrequency    center freqency of the subband
 * @param[in]  beam                index number of the beam
 * @param[in]  delaysAtBeginPtr    pointer to delay data of ::DelaysType,
 *                                 a 2D array [beam][station] of float2 (real:
 *                                 2 polarizations), containing delays in
 *                                 seconds at begin of integration period
 * @param[in]  delaysAfterEndPtr   pointer to delay data of ::DelaysType,
 *                                 a 2D array [beam][station] of float2 (real:
 *                                 2 polarizations), containing delays in
 *                                 seconds after end of integration period
 * @param[in]  phaseOffsetsPtr     pointer to phase offset data of
 *                                 ::PhaseOffsetsType, a 1D array [station] of
 *                                 float2 (real: 2 polarizations), containing
 *                                 phase offsets in radians
 * @param[in]  bandPassFactorsPtr  pointer to bandpass correction data of
 *                                 ::BandPassFactorsType, a 1D array [channel] of
 *                                 float, containing bandpass correction factors
 */

//__kernel __attribute__((reqd_work_group_size(16 * 16, 1, 1)))

extern "C" {
 __global__ void applyDelaysAndCorrectBandPass( complexfloat * correctedDataPtr,
                                    const complexfloat * filteredDataPtr,
                                   float subbandFrequency,
                                   unsigned beam,
                                    const float2 * delaysAtBeginPtr,
                                    const float2 * delaysAfterEndPtr,
                                    const float2 * phaseOffsetsPtr,
                                    const float * bandPassFactorsPtr)
{
  OutputDataType outputData = (OutputDataType) correctedDataPtr;
  InputDataType inputData = (InputDataType) filteredDataPtr;
  DelaysType delaysAtBegin = (DelaysType) delaysAtBeginPtr;
  DelaysType delaysAfterEnd = (DelaysType) delaysAfterEndPtr;
  PhaseOffsetsType phaseOffsets = (PhaseOffsetsType) phaseOffsetsPtr;

#if NR_CHANNELS > 1
  BandPassFactorsType bandPassFactors = (BandPassFactorsType) bandPassFactorsPtr;

  complexfloat tmp[16][17][2]; // one too wide to allow coalesced reads

  unsigned major = blockIdx.x*blockDim.x+threadIdx.x / 16;
  unsigned minor = blockIdx.x*blockDim.x+threadIdx.x % 16;
  unsigned channel = blockIdx.y*blockDim.y+threadIdx.y * 16;
#endif
  unsigned station = blockIdx.y;

#if defined DELAY_COMPENSATION
#if NR_CHANNELS == 1
  float frequency = subbandFrequency;
#else
  float frequency = subbandFrequency - .5f * SUBBAND_BANDWIDTH + (channel + minor) * (SUBBAND_BANDWIDTH / NR_CHANNELS);
#endif
  complexfloat delayAtBegin = (*delaysAtBegin)[beam][station];
  complexfloat delayAfterEnd = (*delaysAfterEnd)[beam][station];
  complexfloat phiBegin = -2 * 3.1415926535f * delayAtBegin;
  complexfloat phiEnd = -2 * 3.1415926535f * delayAfterEnd;
  complexfloat deltaPhi = (phiEnd - phiBegin) / float(NR_SAMPLES_PER_CHANNEL); //cast to float first
#if NR_CHANNELS == 1
  complexfloat myPhiBegin = (phiBegin + float(threadIdx.x) * deltaPhi) * frequency + (*phaseOffsets)[station];
  complexfloat myPhiDelta = float(blockDim.x) * deltaPhi * frequency;
#else
  complexfloat myPhiBegin = (phiBegin + float(major) * deltaPhi) * frequency + (*phaseOffsets)[station];
  complexfloat myPhiDelta = 16.0f * deltaPhi * frequency;
#endif
  complexfloat vX = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiBegin.real()));  // This cast might be costly
  complexfloat vY = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiBegin.imag()));
  complexfloat dvX = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiDelta.real()));
  complexfloat dvY = LOFAR::Cobalt::gpu::exp(complexfloat(myPhiDelta.imag()));
#endif

#if defined BANDPASS_CORRECTION
  float weight = (*bandPassFactors)[channel + minor];
#endif

#if defined DELAY_COMPENSATION && defined BANDPASS_CORRECTION
  vX *= weight;
  vY *= weight;
#endif

#if NR_CHANNELS == 1
  for (unsigned time = threadIdx.x; time < NR_SAMPLES_PER_SUBBAND; time += blockDim.x) 
  {
    //fcomplex2 samples = convert_float4((*inputData)[station][time]);  complexshort (* InputDataType)[NR_STATIONS][NR_SAMPLES_PER_SUBBAND][NR_POLARIZATIONS]
    complexfloat sampleX = complexfloat((*inputData)[station][time][0].real(),
                                        (*inputData)[station][time][0].imag()); //samples.s01; // low floats from the float4
    complexfloat sampleY = complexfloat((*inputData)[station][time][1].real(),
                                        (*inputData)[station][time][1].imag()); //samples.s23; //high floats
#else
  for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time += 16) {
    complexfloat sampleX = (*inputData)[station][0][time + major][channel + minor];
    complexfloat sampleY = (*inputData)[station][1][time + major][channel + minor];
#endif

#if defined DELAY_COMPENSATION
    sampleX = sampleX * vX;
    sampleY = sampleY * vY;
    vX = vY * dvX;
    vY = vY * dvY;
#elif defined BANDPASS_CORRECTION
    sampleX *= weight;
    sampleY *= weight;
#endif

#if NR_CHANNELS == 1
    (*outputData)[station][0][time][0] = sampleX;
    (*outputData)[station][0][time][1] = sampleY;
#else
    tmp[major][minor][0] = sampleX;
    tmp[major][minor][1] = sampleY;
    __syncthreads();


    (*outputData)[station][channel + major][time + minor][0] = tmp[minor][major][0];
    (*outputData)[station][channel + major][time + minor][1] = tmp[minor][major][1];
    __syncthreads();

#endif
  }
}
}
