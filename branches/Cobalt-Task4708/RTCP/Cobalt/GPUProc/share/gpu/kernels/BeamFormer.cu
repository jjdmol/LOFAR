#include "hip/hip_runtime.h"
//# BeamFormer.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#include "gpu_math.cuh"

// Some defines used to determine the correct way the process the data
#define MAX(A,B) ((A)>(B) ? (A) : (B))

#define NR_PASSES MAX((NR_STATIONS + 6) / 16, 1) // gives best results on GTX 680

#ifndef NR_STATIONS_PER_PASS  // Allow overriding for testing optimalizations 
  #define NR_STATIONS_PER_PASS ((NR_STATIONS + NR_PASSES - 1) / NR_PASSES)
#endif
#if NR_STATIONS_PER_PASS > 32
#error "need more passes to beam for this number of stations"
#endif

// Typedefs used to map input data on arrays
typedef  float  (*DelaysType)[NR_SAPS][NR_STATIONS][NR_TABS];
typedef  float4 (*BandPassCorrectedType)[NR_STATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];
typedef  float2 (*ComplexVoltagesType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS][NR_POLARIZATIONS];

/*!
 * Performs beamforming to x beam based.
 * The beamformer performs a complex weighted multiply add of the each sample of the
 * provided input data.
 *
 * \param[out] complexVoltagesPtr      4D output array of beams. For each channel a number of Tied Array Beams time serires is created for two polarizations
 * \param[in]  correctedDataPtr        3D input array of samples. A time series for each station and channel pair. Each sample contains the 2 polarizations X, Y, each of complex float type.
 * \param[in]  delaysPtr               3D input array of complex valued delays to be applied to the correctData samples. There is a delay for each station, channel and Tied Array Beam triplet.
 * \param[in]  subbandFrequency        central frequency of the subband

 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | >= 1                    | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create
 * WEIGHT_CORRECTION       | float                   | weighting applied to all weights derived from the delays, primarily used for correcting FFT and iFFT chain multiplication correction
 * ----------------------- | ------------------------| 
 * NR_STATIONS_PER_PASS    | 1 >= && <= 32           | Set to overide default: Parallelization parameter, controls the number stations to beamform in a single pass over the input data. 
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2 and COMPLEX == 2
 *
 * Execution configuration:
 * - LocalWorkSize = (NR_POLARIZATIONS, NR_TABS, NR_CHANNELS) Note that for full utilization NR_TABS * NR_CHANNELS % 16 = 0
 */
extern "C" __global__ void beamFormer( void *complexVoltagesPtr,
                                       const void *samplesPtr,
                                       const void *delaysPtr,
                                       float subbandFrequency)
{
  ComplexVoltagesType complexVoltages = (ComplexVoltagesType) complexVoltagesPtr;
  BandPassCorrectedType samples = (BandPassCorrectedType) samplesPtr;
  DelaysType delays = (DelaysType) delaysPtr;

  unsigned pol = threadIdx.x;
  unsigned tab = threadIdx.y;
  unsigned channel = blockDim.z * blockIdx.z + threadIdx.z; // The parallelization in the channel is controllable with extra blocks

  float2 sample;
  // This union is in shared memory because it is used by all threads in the block
  __shared__ union { // Union: Maps two variables to the same adress space
    float2 samples[NR_STATIONS_PER_PASS][16][NR_POLARIZATIONS];
    float4 samples4[NR_STATIONS_PER_PASS][16];
  } _local;

#if NR_CHANNELS == 1
  float frequency = subbandFrequency;
#else
  float frequency = subbandFrequency - .5f * SUBBAND_BANDWIDTH + channel * (SUBBAND_BANDWIDTH / NR_CHANNELS);
#endif

#pragma unroll
  for (unsigned first_station = 0;  // Step over data with NR_STATIONS_PER_PASS stride
       first_station < NR_STATIONS;
       first_station += NR_STATIONS_PER_PASS) 
  { // this for loop spans the whole file
#if NR_STATIONS_PER_PASS >= 1
    fcomplex weight_00;                     // assign the weights to register variables
    if (first_station + 0 < NR_STATIONS) {  // Number of station might be larger then 32:
                                            // We then do multiple passes to span all stations
      float delay = (*delays)[first_station + 0][channel][tab];
      weight_00 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif
    // Loop onrolling allows usage of registers for weights
#if NR_STATIONS_PER_PASS >= 2
    fcomplex weight_01;

    if (first_station + 1 < NR_STATIONS) {
      float delay = (*delays)[first_station + 1][channel][tab];
      weight_01 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 3
    fcomplex weight_02;

    if (first_station + 2 < NR_STATIONS) {
      float delay = (*delays)[first_station + 2][channel][tab];
      weight_02 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 4
    fcomplex weight_03;

    if (first_station + 3 < NR_STATIONS) {
      float delay = (*delays)[first_station + 3][channel][tab];
      weight_03 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 5
    fcomplex weight_04;

    if (first_station + 4 < NR_STATIONS) {
      float delay = (*delays)[first_station + 4][channel][tab];
      weight_04 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 6
    fcomplex weight_05;

    if (first_station + 5 < NR_STATIONS) {
      float delay = (*delays)[first_station + 5][channel][tab];
      weight_05 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 7
    fcomplex weight_06;

    if (first_station + 6 < NR_STATIONS) {
      float delay = (*delays)[first_station + 6][channel][tab];
      weight_06 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 8
    fcomplex weight_07;

    if (first_station + 7 < NR_STATIONS) {
      float delay = (*delays)[first_station + 7][channel][tab];
      weight_07 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 9
    fcomplex weight_08;

    if (first_station + 8 < NR_STATIONS) {
      float delay = (*delays)[first_station + 8][channel][tab];
      weight_08 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 10
    fcomplex weight_09;

    if (first_station + 9 < NR_STATIONS) {
      float delay = (*delays)[first_station + 9][channel][tab];
      weight_09 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 11
    fcomplex weight_10;

    if (first_station + 10 < NR_STATIONS) {
      float delay = (*delays)[first_station + 10][channel][tab];
      weight_10 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 12
    fcomplex weight_11;

    if (first_station + 11 < NR_STATIONS) {
      float delay = (*delays)[first_station + 11][channel][tab];
      weight_11 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 13
    fcomplex weight_12;

    if (first_station + 12 < NR_STATIONS) {
      float delay = (*delays)[first_station + 12][channel][tab];
      weight_12 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 14
    fcomplex weight_13;

    if (first_station + 13 < NR_STATIONS) {
      float delay = (*delays)[first_station + 13][channel][tab];
      weight_13 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 15
    fcomplex weight_14;

    if (first_station + 14 < NR_STATIONS) {
      float delay = (*delays)[first_station + 14][channel][tab];
      weight_14 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 16
    fcomplex weight_15;

    if (first_station + 15 < NR_STATIONS) {
      float delay = (*delays)[first_station + 15][channel][tab];
      weight_15 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 17
    fcomplex weight_16;

    if (first_station + 16 < NR_STATIONS) {
      float delay = (*delays)[first_station + 16][channel][tab];
      weight_16 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 18
    fcomplex weight_17;

    if (first_station + 17 < NR_STATIONS) {
      float delay = (*delays)[first_station + 17][channel][tab];
      weight_17 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 19
    fcomplex weight_18;

    if (first_station + 18 < NR_STATIONS)
      float delay = (*delays)[first_station + 18][channel][tab];
      weight_18 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 20
    fcomplex weight_19;

    if (first_station + 19 < NR_STATIONS) {
      float delay = (*delays)[first_station + 19][channel][tab];
      weight_19 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 21
    fcomplex weight_20;

    if (first_station + 20 < NR_STATIONS) {
      float delay = (*delays)[first_station + 20][channel][tab];
      weight_20 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 22
    fcomplex weight_21;

    if (first_station + 21 < NR_STATIONS) {
      float delay = (*delays)[first_station + 21][channel][tab];
      weight_21 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 23
    fcomplex weight_22;

    if (first_station + 22 < NR_STATIONS) {
      float delay = (*delays)[first_station + 22][channel][tab];
      weight_22 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 24
    fcomplex weight_23;

    if (first_station + 23 < NR_STATIONS) {
      float delay = (*delays)[first_station + 23][channel][tab];
      weight_23 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 25
    fcomplex weight_24;

    if (first_station + 24 < NR_STATIONS) {
      float delay = (*delays)[first_station + 24][channel][tab];
      weight_24 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 26
    fcomplex weight_25;

    if (first_station + 25 < NR_STATIONS) {
      float delay = (*delays)[first_station + 25][channel][tab];
      weight_25 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 27
    fcomplex weight_26;

    if (first_station + 26 < NR_STATIONS) {
      float delay = (*delays)[first_station + 26][channel][tab];
      weight_26 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 28
    fcomplex weight_27;

    if (first_station + 27 < NR_STATIONS) {
      float delay = (*delays)[first_station + 27][channel][tab];
      weight_27 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 29
    fcomplex weight_28;

    if (first_station + 28 < NR_STATIONS) {
      float delay = (*delays)[first_station + 28][channel][tab];
      weight_28 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 30
    fcomplex weight_29;

    if (first_station + 29 < NR_STATIONS) {
      float delay = (*delays)[first_station + 29][channel][tab];
      weight_29 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 31
    fcomplex weight_30;

    if (first_station + 30 < NR_STATIONS) {
      float delay = (*delays)[first_station + 30][channel][tab];
      weight_30 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

#if NR_STATIONS_PER_PASS >= 32
    fcomplex weight_31;

    if (first_station + 31 < NR_STATIONS) {
      float delay = (*delays)[first_station + 31][channel][tab];
      weight_31 = phaseShift(frequency, delay) * WEIGHT_CORRECTION;
    }
#endif

    // Loop over all the samples in time
    // TODO: This is a candidate to be added as an extra parallelization dim.
    // problem: we already have the x,y and z filled with parallel parameters. Make the polarization implicit?
    for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time += 16)  // Perform the addition for 16 timesteps
    {
      // Optimized memory transfer: Threads load from memory in parallel
      for (unsigned i = threadIdx.x + NR_POLARIZATIONS * threadIdx.y;
                    i < NR_STATIONS_PER_PASS * 16;
                    i += NR_TABS * NR_POLARIZATIONS) 
      {
        unsigned t = i % 16;
        unsigned s = i / 16;

        if (NR_SAMPLES_PER_CHANNEL % 16 == 0 || time + t < NR_SAMPLES_PER_CHANNEL)
          if (NR_STATIONS % NR_STATIONS_PER_PASS == 0 || first_station + s < NR_STATIONS)
            _local.samples4[0][i] = (*samples)[first_station + s][channel][time + t];
      }

       __syncthreads();


      for (unsigned t = 0; 
                    t < (NR_SAMPLES_PER_CHANNEL % 16 == 0 ? 16 : min(16U, NR_SAMPLES_PER_CHANNEL - time));
                    t++) 
      {
        float2 sum = first_station == 0 ? // The first run the sum should be zero, otherwise we need to take the sum of the previous run
                    make_float2(0,0) :
                    (*complexVoltages)[channel][time + t][tab][pol];

        // Calculate the weighted complex sum of the samples
#if NR_STATIONS_PER_PASS >= 1
        if (first_station + 1 <= NR_STATIONS) {  // Remember that the number of stations might not be a multiple of 32. Skip if station does not exist
          sample = _local.samples[ 0][t][pol];
          sum.x += weight_00.x * sample.x;
          sum.y += weight_00.x * sample.y;
          sum.x += weight_00.y * -sample.y;
          sum.y += weight_00.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 2
        if (first_station + 2 <+ NR_STATIONS) {
          sample = _local.samples[ 1][t][pol];
          sum.x += weight_01.x * sample.x;
          sum.y += weight_01.x * sample.y;
          sum.x += weight_01.y * -sample.y;
          sum.y += weight_01.y * sample.x;
        }
#endif


#if NR_STATIONS_PER_PASS >= 3
        if (first_station + 3 <= NR_STATIONS) {
          sample = _local.samples[ 2][t][pol];
          sum.x += weight_02.x * sample.x;
          sum.y += weight_02.x * sample.y;
          sum.x += weight_02.y * -sample.y;
          sum.y += weight_02.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 4
        if (first_station + 4 <= NR_STATIONS) {
          sample = _local.samples[ 3][t][pol];
          sum.x += weight_03.x * sample.x;
          sum.y += weight_03.x * sample.y;
          sum.x += weight_03.y * -sample.y;
          sum.y += weight_03.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 5
        if (first_station + 5 <= NR_STATIONS) {
          sample = _local.samples[ 4][t][pol];
          sum.x += weight_04.x * sample.x;
          sum.y += weight_04.x * sample.y;
          sum.x += weight_04.y * -sample.y;
          sum.y += weight_04.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 6
        if (first_station + 6 <= NR_STATIONS) {
          sample = _local.samples[ 5][t][pol];
          sum.x += weight_05.x * sample.x;
          sum.y += weight_05.x * sample.y;
          sum.x += weight_05.y * -sample.y;
          sum.y += weight_05.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 7
        if (first_station + 7 <= NR_STATIONS) {
          sample = _local.samples[ 6][t][pol];
          sum.x += weight_06.x * sample.x;
          sum.y += weight_06.x * sample.y;
          sum.x += weight_06.y * -sample.y;
          sum.y += weight_06.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 8
        if (first_station + 8 <= NR_STATIONS) {
          sample = _local.samples[ 7][t][pol];
          sum.x += weight_07.x * sample.x;
          sum.y += weight_07.x * sample.y;
          sum.x += weight_07.y * -sample.y;
          sum.y += weight_07.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 9
        if (first_station + 9 <= NR_STATIONS) {
          sample = _local.samples[ 8][t][pol];
          sum.x += weight_08.x * sample.x;
          sum.y += weight_08.x * sample.y;
          sum.x += weight_08.y * -sample.y;
          sum.y += weight_08.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 10
        if (first_station + 10 <= NR_STATIONS) {
          sample = _local.samples[ 9][t][pol];
          sum.x += weight_09.x * sample.x;
          sum.y += weight_09.x * sample.y;
          sum.x += weight_09.y * -sample.y;
          sum.y += weight_09.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 11
        if (first_station + 11 <= NR_STATIONS) {
          sample = _local.samples[10][t][pol];
          sum.x += weight_10.x * sample.x;
          sum.y += weight_10.x * sample.y;
          sum.x += weight_10.y * -sample.y;
          sum.y += weight_10.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 12
        if (first_station + 12 <= NR_STATIONS) {
          sample = _local.samples[11][t][pol];
          sum.x += weight_11.x * sample.x;
          sum.y += weight_11.x * sample.y;
          sum.x += weight_11.y * -sample.y;
          sum.y += weight_11.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 13
        if (first_station + 13 <= NR_STATIONS) {
          sample = _local.samples[12][t][pol];
          sum.x += weight_12.x * sample.x;
          sum.y += weight_12.x * sample.y;
          sum.x += weight_12.y * -sample.y;
          sum.y += weight_12.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 14
        if (first_station + 14 <= NR_STATIONS) {
          sample = _local.samples[13][t][pol];
          sum.x += weight_13.x * sample.x;
          sum.y += weight_13.x * sample.y;
          sum.x += weight_13.y * -sample.y;
          sum.y += weight_13.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 15
        if (first_station + 15 <= NR_STATIONS) {
          sample = _local.samples[14][t][pol];
          sum.x += weight_14.x * sample.x;
          sum.y += weight_14.x * sample.y;
          sum.x += weight_14.y * -sample.y;
          sum.y += weight_14.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 16
        if (first_station + 16 <= NR_STATIONS) {
          sample = _local.samples[15][t][pol];
          sum.x += weight_15.x * sample.x;
          sum.y += weight_15.x * sample.y;
          sum.x += weight_15.y * -sample.y;
          sum.y += weight_15.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 17
        if (first_station + 17 <= NR_STATIONS) {
          sample = _local.samples[16][t][pol];
          sum.x += weight_16.x * sample.x;
          sum.y += weight_16.x * sample.y;
          sum.x += weight_16.y * -sample.y;
          sum.y += weight_16.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 18
        if (first_station + 18 <= NR_STATIONS) {
          sample = _local.samples[17][t][pol];
          sum.x += weight_17.x * sample.x;
          sum.y += weight_17.x * sample.y;
          sum.x += weight_17.y * -sample.y;
          sum.y += weight_17.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 19
        if (first_station + 19 <= NR_STATIONS) {
          sample = _local.samples[18][t][pol];
          sum.x += weight_18.x * sample.x;
          sum.y += weight_18.x * sample.y;
          sum.x += weight_18.y * -sample.y;
          sum.y += weight_18.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 20
        if (first_station + 20 <= NR_STATIONS) {
          sample = _local.samples[19][t][pol];
          sum.x += weight_19.x * sample.x;
          sum.y += weight_19.x * sample.y;
          sum.x += weight_19.y * -sample.y;
          sum.y += weight_19.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 21
        if (first_station + 21 <= NR_STATIONS) {
          sample = _local.samples[20][t][pol];
          sum.x += weight_20.x * sample.x;
          sum.y += weight_20.x * sample.y;
          sum.x += weight_20.y * -sample.y;
          sum.y += weight_20.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 22
        if (first_station + 22 <= NR_STATIONS) {
          sample = _local.samples[21][t][pol];
          sum.x += weight_21.x * sample.x;
          sum.y += weight_21.x * sample.y;
          sum.x += weight_21.y * -sample.y;
          sum.y += weight_21.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 23
        if (first_station + 23 <= NR_STATIONS) {
          sample = _local.samples[22][t][pol];
          sum.x += weight_22.x * sample.x;
          sum.y += weight_22.x * sample.y;
          sum.x += weight_22.y * -sample.y;
          sum.y += weight_22.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 24
        if (first_station + 24 <= NR_STATIONS) {
          sample = _local.samples[23][t][pol];
          sum.x += weight_23.x * sample.x;
          sum.y += weight_23.x * sample.y;
          sum.x += weight_23.y * -sample.y;
          sum.y += weight_23.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 25
        if (first_station + 25 <= NR_STATIONS) {
          sample = _local.samples[24][t][pol];
          sum.x += weight_24.x * sample.x;
          sum.y += weight_24.x * sample.y;
          sum.x += weight_24.y * -sample.y;
          sum.y += weight_24.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 26
        if (first_station + 26 <= NR_STATIONS) {
          sample = _local.samples[25][t][pol];
          sum.x += weight_25.x * sample.x;
          sum.y += weight_25.x * sample.y;
          sum.x += weight_25.y * -sample.y;
          sum.y += weight_25.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 27
        if (first_station + 27 <= NR_STATIONS) {
          sample = _local.samples[26][t][pol];
          sum.x += weight_26.x * sample.x;
          sum.y += weight_26.x * sample.y;
          sum.x += weight_26.y * -sample.y;
          sum.y += weight_26.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 28
        if (first_station + 28 <= NR_STATIONS) {
          sample = _local.samples[27][t][pol];
          sum.x += weight_27.x * sample.x;
          sum.y += weight_27.x * sample.y;
          sum.x += weight_27.y * -sample.y;
          sum.y += weight_27.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 29
        if (first_station + 29 <= NR_STATIONS) {
          sample = _local.samples[28][t][pol];
          sum.x += weight_28.x * sample.x;
          sum.y += weight_28.x * sample.y;
          sum.x += weight_28.y * -sample.y;
          sum.y += weight_28.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 30
        if (first_station + 30 <= NR_STATIONS) {
          sample = _local.samples[29][t][pol];
          sum.x += weight_29.x * sample.x;
          sum.y += weight_29.x * sample.y;
          sum.x += weight_29.y * -sample.y;
          sum.y += weight_29.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 31
        if (first_station + 31 <= NR_STATIONS) {
          sample = _local.samples[30][t][pol];
          sum.x += weight_30.x * sample.x;
          sum.y += weight_30.x * sample.y;
          sum.x += weight_30.y * -sample.y;
          sum.y += weight_30.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 32
        if (first_station + 32 <= NR_STATIONS) {
          sample = _local.samples[31][t][pol];
          sum.x += weight_31.x * sample.x;
          sum.y += weight_31.x * sample.y;
          sum.x += weight_31.y * -sample.y;
          sum.y += weight_31.y * sample.x;
        }
#endif
        // Write data to global mem
        (*complexVoltages)[channel][time + t][tab][pol] = sum;
      }

      __syncthreads();
    }
  }
}

