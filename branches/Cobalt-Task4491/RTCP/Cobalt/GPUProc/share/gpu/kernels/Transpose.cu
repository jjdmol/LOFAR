#include "hip/hip_runtime.h"
//# Transepose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

/*!
 *
 * COPY PASTA OF BEAMFORMER DOCUMENTATION 
 *
 *
 *
 * Performs beamforming to x beam based.
 * The beamformer performs a complex weighted multiply add of the each sample of the
 * provided input data.
 *
 * \param[out] transposedDataPtr      4D output array of beams. For each channel a number of Tied Array Beams time serires is created for two polarizations
 * \param[in]  complexVoltagesPtr        3D input array of samples. A time series for each station and channel pair. Each sample contains the 2 polarizations X, Y, each of complex float type.
 * \param[in]  weightsPtr              3d input array of complex valued weights to be applied to the correctData samples. THere is a weight for each station, channel and Tied Array Beam triplet.
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | >= 1                    | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create
 * ----------------------- | ------------------------| 
 * NR_STATIONS_PER_PASS    | 1 >= && <= 32           | Set to overide default: Parallelization parameter, controls the number stations to beamform in a single pass over the input data. 
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2 and COMPLEX == 2
 *
 * Execution configuration:
 * - LocalWorkSize = (NR_POLARIZATIONS, NR_TABS, NR_CHANNELS) Note that for full utilization NR_TABS * NR_CHANNELS % 16 = 0
 */

#ifdef CHANNEL_PARALLEL 
typedef float2 (*ComplexVoltagesType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS][NR_POLARIZATIONS];
typedef float2 (*TransposedDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS];

extern "C" __global__  void transpose( 
                    void * transposedDataPtr,
                    const void * complexVoltagesPtr)
{
  TransposedDataType transposedData = (TransposedDataType) transposedDataPtr;
  ComplexVoltagesType complexVoltages = (ComplexVoltagesType) complexVoltagesPtr;

  __shared__ float2 tmp[16][17][2]; // add one to get coaliesced reads?

  unsigned tabBase = 16 * blockIdx.y; // No use of the block size!!!
  unsigned chBase = 16 * blockIdx.z;

  unsigned tabOffsetR = threadIdx.x & 15;
  unsigned tabR = tabBase + tabOffsetR;
  unsigned chOffsetR = threadIdx.x >> 4;
  unsigned channelR = chBase + chOffsetR;
  bool doR = NR_TABS % 16 == 0 || tabR < NR_TABS;

  unsigned tabOffsetW = threadIdx.x >> 4;
  unsigned tabW = tabBase + tabOffsetW;
  unsigned chOffsetW = threadIdx.x & 15;
  unsigned channelW = chBase + chOffsetW;
  bool doW = NR_TABS % 16 == 0 || tabW < NR_TABS;

  for (int time = 0; time < NR_SAMPLES_PER_CHANNEL; time++) 
  {
    if (doR)  // only do a read and write if we are within our bounds
    {    
      tmp[tabOffsetR][chOffsetR][0] = (*complexVoltages)[channelR][time][tabR][0];
      tmp[tabOffsetR][chOffsetR][1] = (*complexVoltages)[channelR][time][tabR][1];
    }
    __syncthreads();
    if (doW) 
    {
      float2 sample = tmp[tabOffsetW][chOffsetW][0];
      float2 sample2 = tmp[tabOffsetW][chOffsetW][1];
      (*transposedData)[tabW][0][time][channelW] = sample;
      (*transposedData)[tabW][1][time][channelW] = sample2;
    }

    __syncthreads();
  }
}

#else

typedef  float2 (*ComplexVoltagesType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS][NR_POLARIZATIONS]; 
typedef  float2 (*TransposedDataType)[NR_TABS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];

extern "C" __global__  void transpose( void * transposedDataPtr,
                                        const void * complexVoltagesPtr)
{
  TransposedDataType transposedData = (TransposedDataType) transposedDataPtr;
  ComplexVoltagesType complexVoltages = (ComplexVoltagesType) complexVoltagesPtr;

  __shared__ float2 tmp[16][17][2];

  unsigned tabBase = 16 * blockDim.y * blockIdx.y + threadIdx.y;
  unsigned timeBase = 16 * blockDim.z * blockIdx.z + threadIdx.z;

  unsigned tabOffsetR = threadIdx.x & 15;   // use and to get module 16
  unsigned tabR = tabBase + tabOffsetR;
  unsigned timeOffsetR = threadIdx.x >> 4;  // use bitshift to get devision
  unsigned timeR = timeBase + timeOffsetR;
  bool doR = NR_TABS % 16 == 0 || tabR < NR_TABS;

  unsigned tabOffsetW = threadIdx.x >> 4;
  unsigned tabW = tabBase + tabOffsetW;
  unsigned timeOffsetW = threadIdx.x & 15;
  unsigned timeW = timeBase + timeOffsetW;
  bool doW = NR_TABS % 16 == 0 || tabW < NR_TABS;

  for (int channel = 0; channel < NR_CHANNELS; channel++) 
  {
    if (doR)
    {
      tmp[tabOffsetR][timeOffsetR][0] = (*complexVoltages)[channel][timeR][tabR][0];
      tmp[tabOffsetR][timeOffsetR][1] = (*complexVoltages)[channel][timeR][tabR][1];
    }

    __syncthreads();
    if (doW) {
      float2 sample = tmp[tabOffsetW][timeOffsetW][0];
      float2 sample2 = tmp[tabOffsetW][timeOffsetW][1];
      (*transposedData)[tabW][0][channel][timeW] = sample;
      (*transposedData)[tabW][1][channel][timeW] = sample2;
    }

    __syncthreads();
  }
}

#endif
