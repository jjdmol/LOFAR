#include "hip/hip_runtime.h"
//# Transepose.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

#if 0
typedef float2 (*TransposedDataType)[NR_TABS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_CHANNELS];
typedef float4 (*ComplexVoltagesType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS];

/*!
 * Performs beamforming to x beam based.
 * The beamformer performs a complex weighted multiply add of the each sample of the
 * provided input data.
 *
 * \param[out] transposedDataPtr      4D output array of beams. For each channel a number of Tied Array Beams time serires is created for two polarizations
 * \param[in]  complexVoltagesPtr        3D input array of samples. A time series for each station and channel pair. Each sample contains the 2 polarizations X, Y, each of complex float type.
 * \param[in]  weightsPtr              3d input array of complex valued weights to be applied to the correctData samples. THere is a weight for each station, channel and Tied Array Beam triplet.
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | >= 1                    | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create
 * ----------------------- | ------------------------| 
 * NR_STATIONS_PER_PASS    | 1 >= && <= 32           | Set to overide default: Parallelization parameter, controls the number stations to beamform in a single pass over the input data. 
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2 and COMPLEX == 2
 *
 * Execution configuration:
 * - LocalWorkSize = (NR_POLARIZATIONS, NR_TABS, NR_CHANNELS) Note that for full utilization NR_TABS * NR_CHANNELS % 16 = 0
 */
extern "C" __global__  void transposeComplexVoltages( void *restrict transposedDataPtr,
                                        const void *restrict complexVoltagesPtr)
{
  TransposedDataType transposedData = (TransposedDataType) transposedDataPtr;
  ComplexVoltagesType complexVoltages = (ComplexVoltagesType) complexVoltagesPtr;

  __shared__ float4 tmp[16][17]; // add one to get coaliesced reads?

  unsigned tabBase = 16 * blockDim.y * blockIdx.y + threadIdx.y;
  unsigned chBase = 16 * blockDim.z * blockIdx.z + threadIdx.z;

  unsigned tabOffsetR = threadIdx.x & 15;
  unsigned tabR = tabBase + tabOffsetR;
  unsigned chOffsetR = threadIdx.x >> 4;
  unsigned channelR = chBase + chOffsetR;
  bool doR = NR_TABS % 16 == 0 || tabR < NR_TABS;

  unsigned tabOffsetW = threadIdx.x >> 4;
  unsigned tabW = tabBase + tabOffsetW;
  unsigned chOffsetW = threadIdx.x & 15;
  unsigned channelW = chBase + chOffsetW;
  bool doW = NR_TABS % 16 == 0 || tabW < NR_TABS;

  for (int time = 0; time < NR_SAMPLES_PER_CHANNEL; time++) {
    if (doR)
      tmp[tabOffsetR][chOffsetR] = (*complexVoltages)[channelR][time][tabR];

    __syncthreads();

    if (doW) {
      float4 sample = tmp[tabOffsetW][chOffsetW];
      (*transposedData)[tabW][0][time][channelW] = sample.xy;
      (*transposedData)[tabW][1][time][channelW] = sample.zw;
    }

    __syncthreads();
  }
}

#else

typedef  float2 (*TransposedDataType)[NR_TABS][NR_POLARIZATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];
typedef  float4 (*ComplexVoltagesType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS];

extern "C" __global__  void transposeComplexVoltages( void *restrict transposedDataPtr,
                                        const void *restrict complexVoltagesPtr)
{
  TransposedDataType transposedData = (TransposedDataType) transposedDataPtr;
  ComplexVoltagesType complexVoltages = (ComplexVoltagesType) complexVoltagesPtr;

  __shared__ float4 tmp[16][17];

  unsigned tabBase = 16 * blockDim.y * blockIdx.y + threadIdx.y;
  unsigned timeBase = 16 * blockDim.z * blockIdx.z + threadIdx.z;

  unsigned tabOffsetR = threadIdx.x & 15;
  unsigned tabR = tabBase + tabOffsetR;
  unsigned timeOffsetR = threadIdx.x >> 4;
  unsigned timeR = timeBase + timeOffsetR;
  bool doR = NR_TABS % 16 == 0 || tabR < NR_TABS;

  unsigned tabOffsetW = threadIdx.x >> 4;
  unsigned tabW = tabBase + tabOffsetW;
  unsigned timeOffsetW = threadIdx.x & 15;
  unsigned timeW = timeBase + timeOffsetW;
  bool doW = NR_TABS % 16 == 0 || tabW < NR_TABS;

  for (int channel = 0; channel < NR_CHANNELS; channel++) {
    if (doR)
      tmp[tabOffsetR][timeOffsetR] = (*complexVoltages)[timeR][channel][tabR];

    __syncthreads();

    if (doW) {
      float4 sample = tmp[tabOffsetW][timeOffsetW];
      (*transposedData)[tabW][0][channel][timeW] = sample.xy;
      (*transposedData)[tabW][1][channel][timeW] = sample.zw;
    }

    __syncthreads();
  }
}

#endif

