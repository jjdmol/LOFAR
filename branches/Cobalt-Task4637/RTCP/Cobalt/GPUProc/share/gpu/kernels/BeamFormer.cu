#include "hip/hip_runtime.h"
//# BeamFormer.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$
#include "gpu_math.cuh"

// Some defines used to determine the correct way the process the data
#define MAX(A,B) ((A)>(B) ? (A) : (B))

#define NR_PASSES MAX((NR_STATIONS + 6) / 16, 1) // gives best results on GTX 680

#ifndef NR_STATIONS_PER_PASS  // Allow overriding for testing optimalizations 
  #define NR_STATIONS_PER_PASS ((NR_STATIONS + NR_PASSES - 1) / NR_PASSES)
#endif
#if NR_STATIONS_PER_PASS > 32
#error "need more passes to beam for this number of stations"
#endif

// Defines applying additional weighting. Currently used for correcting constant multiplications
// introduced by FFT and iFFT operations: 
#ifndef GLOBAL_WEIGHT_CORRECTION
  #define TIMES_WEIGHT_CORRECTION //empty define: nop
#else 
  #define TIMES_THE_WEIGHT_CORRECTION * GLOBAL_WEIGHT_CORRECTION
#endif
  

// Typedefs used to map input data on arrays
typedef  float2 (*WeightsType)[NR_STATIONS][NR_CHANNELS][NR_TABS];
typedef  float4 (*BandPassCorrectedType)[NR_STATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];
typedef  float2 (*ComplexVoltagesType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS][NR_POLARIZATIONS];

/*!
 * Performs beamforming to x beam based.
 * The beamformer performs a complex weighted multiply add of the each sample of the
 * provided input data.
 *
 * \param[out] complexVoltagesPtr      4D output array of beams. For each channel a number of Tied Array Beams time serires is created for two polarizations
 * \param[in]  correctedDataPtr        3D input array of samples. A time series for each station and channel pair. Each sample contains the 2 polarizations X, Y, each of complex float type.
 * \param[in]  weightsPtr              3d input array of complex valued weights to be applied to the correctData samples. THere is a weight for each station, channel and Tied Array Beam triplet.
 * Pre-processor input symbols (some are tied to the execution configuration)
 * Symbol                  | Valid Values            | Description
 * ----------------------- | ----------------------- | -----------
 * NR_STATIONS             | >= 1                    | number of antenna fields
 * NR_SAMPLES_PER_CHANNEL  | >= 1                    | number of input samples per channel
 * NR_CHANNELS             | >= 1                    | number of frequency channels per subband
 * NR_TABS                 | >= 1                    | number of Tied Array Beams to create
 * ----------------------- | ------------------------| 
 * NR_STATIONS_PER_PASS    | 1 >= && <= 32           | Set to overide default: Parallelization parameter, controls the number stations to beamform in a single pass over the input data. 
 *
 * Note that this kernel assumes  NR_POLARIZATIONS == 2 and COMPLEX == 2
 *
 * Execution configuration:
 * - LocalWorkSize = (NR_POLARIZATIONS, NR_TABS, NR_CHANNELS) Note that for full utilization NR_TABS * NR_CHANNELS % 16 = 0
 */
extern "C" __global__ void beamFormer( void *complexVoltagesPtr,
                                       const void *samplesPtr,
                                       const void *weightsPtr)
{
  ComplexVoltagesType complexVoltages = (ComplexVoltagesType) complexVoltagesPtr;
  BandPassCorrectedType samples = (BandPassCorrectedType) samplesPtr;
  WeightsType weights = (WeightsType) weightsPtr;

  unsigned pol = threadIdx.x;
  unsigned tab = threadIdx.y;
  unsigned channel =  blockDim.z * blockIdx.z + threadIdx.z;  // The paralellization in the channel is controllable with extra blocks

  float2 sample;
  // This union is in shared memory because it is used by all threads in the block
  __shared__ union { // Union: Maps two variables to the same adress space
    float2 samples[NR_STATIONS_PER_PASS][16][NR_POLARIZATIONS];
    float4 samples4[NR_STATIONS_PER_PASS][16];
  } _local;

 

#pragma unroll
  for (unsigned first_station = 0;  // Step over data with NR_STATIONS_PER_PASS stride
       first_station < NR_STATIONS;
       first_station += NR_STATIONS_PER_PASS) 
  { // this for loop spand the whole file
#if NR_STATIONS_PER_PASS >= 1
    float2 weight_00;                     // assign the weights to register variables
    if (first_station + 0 < NR_STATIONS)  // Number of station might be larger then 32: We 
                                          // the do multiple passes to span all stations
      weight_00 = (*weights)[first_station + 0][channel][tab] TIMES_THE_WEIGHT_CORRECTION; // Get data from global mem
#endif
    // Loop onrolling allows usage of registers for weights
#if NR_STATIONS_PER_PASS >= 2
    float2 weight_01;

    if (first_station + 1 < NR_STATIONS)
      weight_01 = (*weights)[first_station + 1][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 3
    float2 weight_02;

    if (first_station + 2 < NR_STATIONS)
      weight_02 = (*weights)[first_station + 2][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 4
    float2 weight_03;

    if (first_station + 3 < NR_STATIONS)
      weight_03 = (*weights)[first_station + 3][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 5
    float2 weight_04;

    if (first_station + 4 < NR_STATIONS)
      weight_04 = (*weights)[first_station + 4][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 6
    float2 weight_05;

    if (first_station + 5 < NR_STATIONS)
      weight_05 = (*weights)[first_station + 5][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 7
    float2 weight_06;

    if (first_station + 6 < NR_STATIONS)
      weight_06 = (*weights)[first_station + 6][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 8
    float2 weight_07;

    if (first_station + 7 < NR_STATIONS)
      weight_07 = (*weights)[first_station + 7][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 9
    float2 weight_08;

    if (first_station + 8 < NR_STATIONS)
      weight_08 = (*weights)[first_station + 8][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 10
    float2 weight_09;

    if (first_station + 9 < NR_STATIONS)
      weight_09 = (*weights)[first_station + 9][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 11
    float2 weight_10;

    if (first_station + 10 < NR_STATIONS)
      weight_10 = (*weights)[first_station + 10][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 12
    float2 weight_11;

    if (first_station + 11 < NR_STATIONS)
      weight_11 = (*weights)[first_station + 11][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 13
    float2 weight_12;

    if (first_station + 12 < NR_STATIONS)
      weight_12 = (*weights)[first_station + 12][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 14
    float2 weight_13;

    if (first_station + 13 < NR_STATIONS)
      weight_13 = (*weights)[first_station + 13][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 15
    float2 weight_14;

    if (first_station + 14 < NR_STATIONS)
      weight_14 = (*weights)[first_station + 14][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 16
    float2 weight_15;

    if (first_station + 15 < NR_STATIONS)
      weight_15 = (*weights)[first_station + 15][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 17
    float2 weight_16;

    if (first_station + 16 < NR_STATIONS)
      weight_16 = (*weights)[first_station + 16][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 18
    float2 weight_17;

    if (first_station + 17 < NR_STATIONS)
      weight_17 = (*weights)[first_station + 17][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 19
    float2 weight_18;

    if (first_station + 18 < NR_STATIONS)
      weight_18 = (*weights)[first_station + 18][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 20
    float2 weight_19;

    if (first_station + 19 < NR_STATIONS)
      weight_19 = (*weights)[first_station + 19][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 21
    float2 weight_20;

    if (first_station + 20 < NR_STATIONS)
      weight_20 = (*weights)[first_station + 20][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 22
    float2 weight_21;

    if (first_station + 21 < NR_STATIONS)
      weight_21 = (*weights)[first_station + 21][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 23
    float2 weight_22;

    if (first_station + 22 < NR_STATIONS)
      weight_22 = (*weights)[first_station + 22][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 24
    float2 weight_23;

    if (first_station + 23 < NR_STATIONS)
      weight_23 = (*weights)[first_station + 23][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 25
    float2 weight_24;

    if (first_station + 24 < NR_STATIONS)
      weight_24 = (*weights)[first_station + 24][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 26
    float2 weight_25;

    if (first_station + 25 < NR_STATIONS)
      weight_25 = (*weights)[first_station + 25][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 27
    float2 weight_26;

    if (first_station + 26 < NR_STATIONS)
      weight_26 = (*weights)[first_station + 26][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 28
    float2 weight_27;

    if (first_station + 27 < NR_STATIONS)
      weight_27 = (*weights)[first_station + 27][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 29
    float2 weight_28;

    if (first_station + 28 < NR_STATIONS)
      weight_28 = (*weights)[first_station + 28][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 30
    float2 weight_29;

    if (first_station + 29 < NR_STATIONS)
      weight_29 = (*weights)[first_station + 29][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 31
    float2 weight_30;

    if (first_station + 30 < NR_STATIONS)
      weight_30 = (*weights)[first_station + 30][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

#if NR_STATIONS_PER_PASS >= 32
    float2 weight_31;

    if (first_station + 31 < NR_STATIONS)
      weight_31 = (*weights)[first_station + 31][channel][tab] TIMES_THE_WEIGHT_CORRECTION;
#endif

    // Loop over all the samples in time
    // TODO: This is a candidate to be added as an extra paralellization dim.
    // problem: we already have the x,y and z filled with parallel parameters. Make the polarization implicit?
    for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time += 16)  // Perform the addition for 16 timesteps
    {
      // Optimized memory transver: Threads load paralel memory
      for (unsigned i = threadIdx.x + NR_POLARIZATIONS * threadIdx.y;
                    i < NR_STATIONS_PER_PASS * 16;
                    i += NR_TABS * NR_POLARIZATIONS) 
      {
        unsigned t = i % 16;
        unsigned s = i / 16;

        if (NR_SAMPLES_PER_CHANNEL % 16 == 0 || time + t < NR_SAMPLES_PER_CHANNEL)
          if (NR_STATIONS % NR_STATIONS_PER_PASS == 0 || first_station + s < NR_STATIONS)
            _local.samples4[0][i] = (*samples)[first_station + s][channel][time + t];
      }

       __syncthreads();


      for (unsigned t = 0; 
                    t < (NR_SAMPLES_PER_CHANNEL % 16 == 0 ? 16 : min(16U, NR_SAMPLES_PER_CHANNEL - time));
                    t++) 
      {
        float2 sum = first_station == 0 ? // The first run the sum should be zero, otherwise we need to take the sum of the previous run
                    make_float2(0,0) :
                    (*complexVoltages)[channel][time + t][tab][pol];

        // Calculate the weighted complex sum of the samples
#if NR_STATIONS_PER_PASS >= 1
        if (first_station + 1 <= NR_STATIONS) {  // Remember that the number of stations might not be a multiple of 32. Skip if station does not exist
          sample = _local.samples[ 0][t][pol];
          sum.x += weight_00.x * sample.x;
          sum.y += weight_00.x * sample.y;
          sum.x += weight_00.y * -sample.y;
          sum.y += weight_00.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 2
        if (first_station + 2 <+ NR_STATIONS) {
          sample = _local.samples[ 1][t][pol];
          sum.x += weight_01.x * sample.x;
          sum.y += weight_01.x * sample.y;
          sum.x += weight_01.y * -sample.y;
          sum.y += weight_01.y * sample.x;
        }
#endif


#if NR_STATIONS_PER_PASS >= 3
        if (first_station + 3 <= NR_STATIONS) {
          sample = _local.samples[ 2][t][pol];
          sum.x += weight_02.x * sample.x;
          sum.y += weight_02.x * sample.y;
          sum.x += weight_02.y * -sample.y;
          sum.y += weight_02.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 4
        if (first_station + 4 <= NR_STATIONS) {
          sample = _local.samples[ 3][t][pol];
          sum.x += weight_03.x * sample.x;
          sum.y += weight_03.x * sample.y;
          sum.x += weight_03.y * -sample.y;
          sum.y += weight_03.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 5
        if (first_station + 5 <= NR_STATIONS) {
          sample = _local.samples[ 4][t][pol];
          sum.x += weight_04.x * sample.x;
          sum.y += weight_04.x * sample.y;
          sum.x += weight_04.y * -sample.y;
          sum.y += weight_04.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 6
        if (first_station + 6 <= NR_STATIONS) {
          sample = _local.samples[ 5][t][pol];
          sum.x += weight_05.x * sample.x;
          sum.y += weight_05.x * sample.y;
          sum.x += weight_05.y * -sample.y;
          sum.y += weight_05.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 7
        if (first_station + 7 <= NR_STATIONS) {
          sample = _local.samples[ 6][t][pol];
          sum.x += weight_06.x * sample.x;
          sum.y += weight_06.x * sample.y;
          sum.x += weight_06.y * -sample.y;
          sum.y += weight_06.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 8
        if (first_station + 8 <= NR_STATIONS) {
          sample = _local.samples[ 7][t][pol];
          sum.x += weight_07.x * sample.x;
          sum.y += weight_07.x * sample.y;
          sum.x += weight_07.y * -sample.y;
          sum.y += weight_07.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 9
        if (first_station + 9 <= NR_STATIONS) {
          sample = _local.samples[ 8][t][pol];
          sum.x += weight_08.x * sample.x;
          sum.y += weight_08.x * sample.y;
          sum.x += weight_08.y * -sample.y;
          sum.y += weight_08.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 10
        if (first_station + 10 <= NR_STATIONS) {
          sample = _local.samples[ 9][t][pol];
          sum.x += weight_09.x * sample.x;
          sum.y += weight_09.x * sample.y;
          sum.x += weight_09.y * -sample.y;
          sum.y += weight_09.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 11
        if (first_station + 11 <= NR_STATIONS) {
          sample = _local.samples[10][t][pol];
          sum.x += weight_10.x * sample.x;
          sum.y += weight_10.x * sample.y;
          sum.x += weight_10.y * -sample.y;
          sum.y += weight_10.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 12
        if (first_station + 12 <= NR_STATIONS) {
          sample = _local.samples[11][t][pol];
          sum.x += weight_11.x * sample.x;
          sum.y += weight_11.x * sample.y;
          sum.x += weight_11.y * -sample.y;
          sum.y += weight_11.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 13
        if (first_station + 13 <= NR_STATIONS) {
          sample = _local.samples[12][t][pol];
          sum.x += weight_12.x * sample.x;
          sum.y += weight_12.x * sample.y;
          sum.x += weight_12.y * -sample.y;
          sum.y += weight_12.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 14
        if (first_station + 14 <= NR_STATIONS) {
          sample = _local.samples[13][t][pol];
          sum.x += weight_13.x * sample.x;
          sum.y += weight_13.x * sample.y;
          sum.x += weight_13.y * -sample.y;
          sum.y += weight_13.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 15
        if (first_station + 15 <= NR_STATIONS) {
          sample = _local.samples[14][t][pol];
          sum.x += weight_14.x * sample.x;
          sum.y += weight_14.x * sample.y;
          sum.x += weight_14.y * -sample.y;
          sum.y += weight_14.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 16
        if (first_station + 16 <= NR_STATIONS) {
          sample = _local.samples[15][t][pol];
          sum.x += weight_15.x * sample.x;
          sum.y += weight_15.x * sample.y;
          sum.x += weight_15.y * -sample.y;
          sum.y += weight_15.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 17
        if (first_station + 17 <= NR_STATIONS) {
          sample = _local.samples[16][t][pol];
          sum.x += weight_16.x * sample.x;
          sum.y += weight_16.x * sample.y;
          sum.x += weight_16.y * -sample.y;
          sum.y += weight_16.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 18
        if (first_station + 18 <= NR_STATIONS) {
          sample = _local.samples[17][t][pol];
          sum.x += weight_17.x * sample.x;
          sum.y += weight_17.x * sample.y;
          sum.x += weight_17.y * -sample.y;
          sum.y += weight_17.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 19
        if (first_station + 19 <= NR_STATIONS) {
          sample = _local.samples[18][t][pol];
          sum.x += weight_18.x * sample.x;
          sum.y += weight_18.x * sample.y;
          sum.x += weight_18.y * -sample.y;
          sum.y += weight_18.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 20
        if (first_station + 20 <= NR_STATIONS) {
          sample = _local.samples[19][t][pol];
          sum.x += weight_19.x * sample.x;
          sum.y += weight_19.x * sample.y;
          sum.x += weight_19.y * -sample.y;
          sum.y += weight_19.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 21
        if (first_station + 21 <= NR_STATIONS) {
          sample = _local.samples[20][t][pol];
          sum.x += weight_20.x * sample.x;
          sum.y += weight_20.x * sample.y;
          sum.x += weight_20.y * -sample.y;
          sum.y += weight_20.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 22
        if (first_station + 22 <= NR_STATIONS) {
          sample = _local.samples[21][t][pol];
          sum.x += weight_21.x * sample.x;
          sum.y += weight_21.x * sample.y;
          sum.x += weight_21.y * -sample.y;
          sum.y += weight_21.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 23
        if (first_station + 23 <= NR_STATIONS) {
          sample = _local.samples[22][t][pol];
          sum.x += weight_22.x * sample.x;
          sum.y += weight_22.x * sample.y;
          sum.x += weight_22.y * -sample.y;
          sum.y += weight_22.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 24
        if (first_station + 24 <= NR_STATIONS) {
          sample = _local.samples[23][t][pol];
          sum.x += weight_23.x * sample.x;
          sum.y += weight_23.x * sample.y;
          sum.x += weight_23.y * -sample.y;
          sum.y += weight_23.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 25
        if (first_station + 25 <= NR_STATIONS) {
          sample = _local.samples[24][t][pol];
          sum.x += weight_24.x * sample.x;
          sum.y += weight_24.x * sample.y;
          sum.x += weight_24.y * -sample.y;
          sum.y += weight_24.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 26
        if (first_station + 26 <= NR_STATIONS) {
          sample = _local.samples[25][t][pol];
          sum.x += weight_25.x * sample.x;
          sum.y += weight_25.x * sample.y;
          sum.x += weight_25.y * -sample.y;
          sum.y += weight_25.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 27
        if (first_station + 27 <= NR_STATIONS) {
          sample = _local.samples[26][t][pol];
          sum.x += weight_26.x * sample.x;
          sum.y += weight_26.x * sample.y;
          sum.x += weight_26.y * -sample.y;
          sum.y += weight_26.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 28
        if (first_station + 28 <= NR_STATIONS) {
          sample = _local.samples[27][t][pol];
          sum.x += weight_27.x * sample.x;
          sum.y += weight_27.x * sample.y;
          sum.x += weight_27.y * -sample.y;
          sum.y += weight_27.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 29
        if (first_station + 29 <= NR_STATIONS) {
          sample = _local.samples[28][t][pol];
          sum.x += weight_28.x * sample.x;
          sum.y += weight_28.x * sample.y;
          sum.x += weight_28.y * -sample.y;
          sum.y += weight_28.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 30
        if (first_station + 30 <= NR_STATIONS) {
          sample = _local.samples[29][t][pol];
          sum.x += weight_29.x * sample.x;
          sum.y += weight_29.x * sample.y;
          sum.x += weight_29.y * -sample.y;
          sum.y += weight_29.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 31
        if (first_station + 31 <= NR_STATIONS) {
          sample = _local.samples[30][t][pol];
          sum.x += weight_30.x * sample.x;
          sum.y += weight_30.x * sample.y;
          sum.x += weight_30.y * -sample.y;
          sum.y += weight_30.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 32
        if (first_station + 32 <= NR_STATIONS) {
          sample = _local.samples[31][t][pol];
          sum.x += weight_31.x * sample.x;
          sum.y += weight_31.x * sample.y;
          sum.x += weight_31.y * -sample.y;
          sum.y += weight_31.y * sample.x;
        }
#endif
        // Write data to global mem
        (*complexVoltages)[channel][time + t][tab][pol] = sum;
      }

      __syncthreads();
    }
  }
}

