#include "hip/hip_runtime.h"
//# BeamFormer.cu
//# Copyright (C) 2012-2013  ASTRON (Netherlands Institute for Radio Astronomy)
//# P.O. Box 2, 7990 AA Dwingeloo, The Netherlands
//#
//# This file is part of the LOFAR software suite.
//# The LOFAR software suite is free software: you can redistribute it and/or
//# modify it under the terms of the GNU General Public License as published
//# by the Free Software Foundation, either version 3 of the License, or
//# (at your option) any later version.
//#
//# The LOFAR software suite is distributed in the hope that it will be useful,
//# but WITHOUT ANY WARRANTY; without even the implied warranty of
//# MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//# GNU General Public License for more details.
//#
//# You should have received a copy of the GNU General Public License along
//# with the LOFAR software suite. If not, see <http://www.gnu.org/licenses/>.
//#
//# $Id$

// Some defines used to determine the correct way the process the data
// TODO: Should these be determined outside of the cu file? This is currently black magix
#define MAX(A,B) ((A)>(B) ? (A) : (B))
#define NR_PASSES MAX((NR_STATIONS + 6) / 16, 1) // gives best results on GTX 680
#define NR_STATIONS_PER_PASS ((NR_STATIONS + NR_PASSES - 1) / NR_PASSES)

#if NR_STATIONS_PER_PASS > 32
#error "need more passes to beam for this number of stations"
#endif

// Documentation parts:
// The blockDim.y loops the tabs
// The blockDim.x loops the polarisations

// Typedefs used to map input data on arrays
typedef  float2 (*WeightsType)[NR_STATIONS][NR_CHANNELS][NR_TABS];
typedef  float4 (*BandPassCorrectedType)[NR_STATIONS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL];
typedef  float2 (*ComplexVoltagesType)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL][NR_TABS][NR_POLARIZATIONS];

extern "C" __global__ void beamFormer( void *complexVoltagesPtr,
                                       const void *samplesPtr,
                                       const void *weightsPtr)
{
  ComplexVoltagesType complexVoltages = (ComplexVoltagesType) complexVoltagesPtr;
  BandPassCorrectedType samples = (BandPassCorrectedType) samplesPtr;
  WeightsType weights = (WeightsType) weightsPtr;

  unsigned pol = threadIdx.x;
  unsigned tab = threadIdx.y;
  unsigned channel =  blockDim.z * blockIdx.z + threadIdx.z;  // The paralellization in the channel is controllable with extra blocks

  float2 sample;
  // This union is in shared memory because it is used by all threads in the block
  __shared__ union { // Union: Maps two variables to the same adress space
    float2 samples[NR_STATIONS_PER_PASS][16][NR_POLARIZATIONS];
    float4 samples4[NR_STATIONS_PER_PASS][16];
  } _local;



#pragma unroll
  for (unsigned first_station = 0;  // We loop over the stations: this allows us to get all the weights for a station
       first_station < NR_STATIONS;
       first_station += NR_STATIONS_PER_PASS) 
  { // this for loop spand the whole file
#if NR_STATIONS_PER_PASS >= 1
    float2 weight_00;

    if (first_station + 0 < NR_STATIONS)
      weight_00 = (*weights)[first_station + 0][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 2
    float2 weight_01;

    if (first_station + 1 < NR_STATIONS)
      weight_01 = (*weights)[first_station + 1][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 3
    float2 weight_02;

    if (first_station + 2 < NR_STATIONS)
      weight_02 = (*weights)[first_station + 2][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 4
    float2 weight_03;

    if (first_station + 3 < NR_STATIONS)
      weight_03 = (*weights)[first_station + 3][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 5
    float2 weight_04;

    if (first_station + 4 < NR_STATIONS)
      weight_04 = (*weights)[first_station + 4][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 6
    float2 weight_05;

    if (first_station + 5 < NR_STATIONS)
      weight_05 = (*weights)[first_station + 5][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 7
    float2 weight_06;

    if (first_station + 6 < NR_STATIONS)
      weight_06 = (*weights)[first_station + 6][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 8
    float2 weight_07;

    if (first_station + 7 < NR_STATIONS)
      weight_07 = (*weights)[first_station + 7][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 9
    float2 weight_08;

    if (first_station + 8 < NR_STATIONS)
      weight_08 = (*weights)[first_station + 8][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 10
    float2 weight_09;

    if (first_station + 9 < NR_STATIONS)
      weight_09 = (*weights)[first_station + 9][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 11
    float2 weight_10;

    if (first_station + 10 < NR_STATIONS)
      weight_10 = (*weights)[first_station + 10][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 12
    float2 weight_11;

    if (first_station + 11 < NR_STATIONS)
      weight_11 = (*weights)[first_station + 11][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 13
    float2 weight_12;

    if (first_station + 12 < NR_STATIONS)
      weight_12 = (*weights)[first_station + 12][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 14
    float2 weight_13;

    if (first_station + 13 < NR_STATIONS)
      weight_13 = (*weights)[first_station + 13][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 15
    float2 weight_14;

    if (first_station + 14 < NR_STATIONS)
      weight_14 = (*weights)[first_station + 14][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 16
    float2 weight_15;

    if (first_station + 15 < NR_STATIONS)
      weight_15 = (*weights)[first_station + 15][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 17
    float2 weight_16;

    if (first_station + 16 < NR_STATIONS)
      weight_16 = (*weights)[first_station + 16][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 18
    float2 weight_17;

    if (first_station + 17 < NR_STATIONS)
      weight_17 = (*weights)[first_station + 17][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 19
    float2 weight_18;

    if (first_station + 18 < NR_STATIONS)
      weight_18 = (*weights)[first_station + 18][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 20
    float2 weight_19;

    if (first_station + 19 < NR_STATIONS)
      weight_19 = (*weights)[first_station + 19][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 21
    float2 weight_20;

    if (first_station + 20 < NR_STATIONS)
      weight_20 = (*weights)[first_station + 20][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 22
    float2 weight_21;

    if (first_station + 21 < NR_STATIONS)
      weight_21 = (*weights)[first_station + 21][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 23
    float2 weight_22;

    if (first_station + 22 < NR_STATIONS)
      weight_22 = (*weights)[first_station + 22][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 24
    float2 weight_23;

    if (first_station + 23 < NR_STATIONS)
      weight_23 = (*weights)[first_station + 23][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 25
    float2 weight_24;

    if (first_station + 24 < NR_STATIONS)
      weight_24 = (*weights)[first_station + 24][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 26
    float2 weight_25;

    if (first_station + 25 < NR_STATIONS)
      weight_25 = (*weights)[first_station + 25][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 27
    float2 weight_26;

    if (first_station + 26 < NR_STATIONS)
      weight_26 = (*weights)[first_station + 26][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 28
    float2 weight_27;

    if (first_station + 27 < NR_STATIONS)
      weight_27 = (*weights)[first_station + 27][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 29
    float2 weight_28;

    if (first_station + 28 < NR_STATIONS)
      weight_28 = (*weights)[first_station + 28][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 30
    float2 weight_29;

    if (first_station + 29 < NR_STATIONS)
      weight_29 = (*weights)[first_station + 29][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 31
    float2 weight_30;

    if (first_station + 30 < NR_STATIONS)
      weight_30 = (*weights)[first_station + 30][channel][tab];
#endif

#if NR_STATIONS_PER_PASS >= 32
    float2 weight_31;

    if (first_station + 31 < NR_STATIONS)
      weight_31 = (*weights)[first_station + 31][channel][tab];
#endif

    for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time += 16) 
    {
      for (unsigned i = threadIdx.x + NR_POLARIZATIONS * threadIdx.y;
           i < NR_STATIONS_PER_PASS * 16;
           i += NR_TABS * NR_POLARIZATIONS) 
      {
        unsigned t = i % 16;
        unsigned s = i / 16;

        if (NR_SAMPLES_PER_CHANNEL % 16 == 0 || time + t < NR_SAMPLES_PER_CHANNEL)
          if (NR_STATIONS % NR_STATIONS_PER_PASS == 0 || first_station + s < NR_STATIONS)
            _local.samples4[0][i] = (*samples)[first_station + s][channel][time + t];
      }

       __syncthreads();

      for (unsigned t = 0; 
           t < (NR_SAMPLES_PER_CHANNEL % 16 == 0 ? 
           16 : min(16U, NR_SAMPLES_PER_CHANNEL - time)); t++) 
      {
        // why is the first station zero?
        float2 sum = first_station == 0 ? 
                    make_float2(0,0) :
                    (*complexVoltages)[channel][time + t][tab][pol];


#if NR_STATIONS_PER_PASS >= 1
        if (first_station + 1 <= NR_STATIONS) {
          sample = _local.samples[ 0][t][pol];
          sum.x += weight_00.x * sample.x;
          sum.y += weight_00.x * sample.y;
          sum.x += weight_00.y * -sample.y;
          sum.y += weight_00.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 2
        if (first_station + 2 <+ NR_STATIONS) {
          sample = _local.samples[ 1][t][pol];
          sum.x += weight_01.x * sample.x;
          sum.y += weight_01.x * sample.y;
          sum.x += weight_01.y * -sample.y;
          sum.y += weight_01.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 3
        if (first_station + 3 <= NR_STATIONS) {
          sample = _local.samples[ 2][t][pol];
          sum.x += weight_02.x * sample.x;
          sum.y += weight_02.x * sample.y;
          sum.x += weight_02.y * -sample.y;
          sum.y += weight_02.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 4
        if (first_station + 4 <= NR_STATIONS) {
          sample = _local.samples[ 3][t][pol];
          sum.x += weight_03.x * sample.x;
          sum.y += weight_03.x * sample.y;
          sum.x += weight_03.y * -sample.y;
          sum.y += weight_03.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 5
        if (first_station + 5 <= NR_STATIONS) {
          sample = _local.samples[ 4][t][pol];
          sum.x += weight_04.x * sample.x;
          sum.y += weight_04.x * sample.y;
          sum.x += weight_04.y * -sample.y;
          sum.y += weight_04.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 6
        if (first_station + 6 <= NR_STATIONS) {
          sample = _local.samples[ 5][t][pol];
          sum.x += weight_05.x * sample.x;
          sum.y += weight_05.x * sample.y;
          sum.x += weight_05.y * -sample.y;
          sum.y += weight_05.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 7
        if (first_station + 7 <= NR_STATIONS) {
          sample = _local.samples[ 6][t][pol];
          sum.x += weight_06.x * sample.x;
          sum.y += weight_06.x * sample.y;
          sum.x += weight_06.y * -sample.y;
          sum.y += weight_06.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 8
        if (first_station + 8 <= NR_STATIONS) {
          sample = _local.samples[ 7][t][pol];
          sum.x += weight_07.x * sample.x;
          sum.y += weight_07.x * sample.y;
          sum.x += weight_07.y * -sample.y;
          sum.y += weight_07.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 9
        if (first_station + 9 <= NR_STATIONS) {
          sample = _local.samples[ 8][t][pol];
          sum.x += weight_08.x * sample.x;
          sum.y += weight_08.x * sample.y;
          sum.x += weight_08.y * -sample.y;
          sum.y += weight_08.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 10
        if (first_station + 10 <= NR_STATIONS) {
          sample = _local.samples[ 9][t][pol];
          sum.x += weight_09.x * sample.x;
          sum.y += weight_09.x * sample.y;
          sum.x += weight_09.y * -sample.y;
          sum.y += weight_09.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 11
        if (first_station + 11 <= NR_STATIONS) {
          sample = _local.samples[10][t][pol];
          sum.x += weight_10.x * sample.x;
          sum.y += weight_10.x * sample.y;
          sum.x += weight_10.y * -sample.y;
          sum.y += weight_10.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 12
        if (first_station + 12 <= NR_STATIONS) {
          sample = _local.samples[11][t][pol];
          sum.x += weight_11.x * sample.x;
          sum.y += weight_11.x * sample.y;
          sum.x += weight_11.y * -sample.y;
          sum.y += weight_11.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 13
        if (first_station + 13 <= NR_STATIONS) {
          sample = _local.samples[12][t][pol];
          sum.x += weight_12.x * sample.x;
          sum.y += weight_12.x * sample.y;
          sum.x += weight_12.y * -sample.y;
          sum.y += weight_12.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 14
        if (first_station + 14 <= NR_STATIONS) {
          sample = _local.samples[13][t][pol];
          sum.x += weight_13.x * sample.x;
          sum.y += weight_13.x * sample.y;
          sum.x += weight_13.y * -sample.y;
          sum.y += weight_13.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 15
        if (first_station + 15 <= NR_STATIONS) {
          sample = _local.samples[14][t][pol];
          sum.x += weight_14.x * sample.x;
          sum.y += weight_14.x * sample.y;
          sum.x += weight_14.y * -sample.y;
          sum.y += weight_14.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 16
        if (first_station + 16 <= NR_STATIONS) {
          sample = _local.samples[15][t][pol];
          sum.x += weight_15.x * sample.x;
          sum.y += weight_15.x * sample.y;
          sum.x += weight_15.y * -sample.y;
          sum.y += weight_15.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 17
        if (first_station + 17 <= NR_STATIONS) {
          sample = _local.samples[16][t][pol];
          sum.x += weight_16.x * sample.x;
          sum.y += weight_16.x * sample.y;
          sum.x += weight_16.y * -sample.y;
          sum.y += weight_16.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 18
        if (first_station + 18 <= NR_STATIONS) {
          sample = _local.samples[17][t][pol];
          sum.x += weight_17.x * sample.x;
          sum.y += weight_17.x * sample.y;
          sum.x += weight_17.y * -sample.y;
          sum.y += weight_17.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 19
        if (first_station + 19 <= NR_STATIONS) {
          sample = _local.samples[18][t][pol];
          sum.x += weight_18.x * sample.x;
          sum.y += weight_18.x * sample.y;
          sum.x += weight_18.y * -sample.y;
          sum.y += weight_18.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 20
        if (first_station + 20 <= NR_STATIONS) {
          sample = _local.samples[19][t][pol];
          sum.x += weight_19.x * sample.x;
          sum.y += weight_19.x * sample.y;
          sum.x += weight_19.y * -sample.y;
          sum.y += weight_19.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 21
        if (first_station + 21 <= NR_STATIONS) {
          sample = _local.samples[20][t][pol];
          sum.x += weight_20.x * sample.x;
          sum.y += weight_20.x * sample.y;
          sum.x += weight_20.y * -sample.y;
          sum.y += weight_20.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 22
        if (first_station + 22 <= NR_STATIONS) {
          sample = _local.samples[21][t][pol];
          sum.x += weight_21.x * sample.x;
          sum.y += weight_21.x * sample.y;
          sum.x += weight_21.y * -sample.y;
          sum.y += weight_21.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 23
        if (first_station + 23 <= NR_STATIONS) {
          sample = _local.samples[22][t][pol];
          sum.x += weight_22.x * sample.x;
          sum.y += weight_22.x * sample.y;
          sum.x += weight_22.y * -sample.y;
          sum.y += weight_22.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 24
        if (first_station + 24 <= NR_STATIONS) {
          sample = _local.samples[23][t][pol];
          sum.x += weight_23.x * sample.x;
          sum.y += weight_23.x * sample.y;
          sum.x += weight_23.y * -sample.y;
          sum.y += weight_23.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 25
        if (first_station + 25 <= NR_STATIONS) {
          sample = _local.samples[24][t][pol];
          sum.x += weight_24.x * sample.x;
          sum.y += weight_24.x * sample.y;
          sum.x += weight_24.y * -sample.y;
          sum.y += weight_24.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 26
        if (first_station + 26 <= NR_STATIONS) {
          sample = _local.samples[25][t][pol];
          sum.x += weight_25.x * sample.x;
          sum.y += weight_25.x * sample.y;
          sum.x += weight_25.y * -sample.y;
          sum.y += weight_25.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 27
        if (first_station + 27 <= NR_STATIONS) {
          sample = _local.samples[26][t][pol];
          sum.x += weight_26.x * sample.x;
          sum.y += weight_26.x * sample.y;
          sum.x += weight_26.y * -sample.y;
          sum.y += weight_26.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 28
        if (first_station + 28 <= NR_STATIONS) {
          sample = _local.samples[27][t][pol];
          sum.x += weight_27.x * sample.x;
          sum.y += weight_27.x * sample.y;
          sum.x += weight_27.y * -sample.y;
          sum.y += weight_27.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 29
        if (first_station + 29 <= NR_STATIONS) {
          sample = _local.samples[28][t][pol];
          sum.x += weight_28.x * sample.x;
          sum.y += weight_28.x * sample.y;
          sum.x += weight_28.y * -sample.y;
          sum.y += weight_28.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 30
        if (first_station + 30 <= NR_STATIONS) {
          sample = _local.samples[29][t][pol];
          sum.x += weight_29.x * sample.x;
          sum.y += weight_29.x * sample.y;
          sum.x += weight_29.y * -sample.y;
          sum.y += weight_29.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 31
        if (first_station + 31 <= NR_STATIONS) {
          sample = _local.samples[30][t][pol];
          sum.x += weight_30.x * sample.x;
          sum.y += weight_30.x * sample.y;
          sum.x += weight_30.y * -sample.y;
          sum.y += weight_30.y * sample.x;
        }
#endif

#if NR_STATIONS_PER_PASS >= 32
        if (first_station + 32 <= NR_STATIONS) {
          sample = _local.samples[31][t][pol];
          sum.x += weight_31.x * sample.x;
          sum.y += weight_31.x * sample.y;
          sum.x += weight_31.y * -sample.y;
          sum.y += weight_31.y * sample.x;
        }
#endif

        (*complexVoltages)[channel][time + t][tab][pol] = sum;
      }

      __syncthreads();
    }
  }
}

